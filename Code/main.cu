#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdarg.h>
#include <vector>
#include <algorithm>
#include <stdexcept>
#include <limits>
#include <cmath>
#include <complex>
#include <sstream>
#include <string>
#include <iomanip>
#include <sys/ioctl.h> 
#include <fcntl.h>
#include <time.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <random>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#if defined(_OPENMP)
#include <omp.h>
#else
typedef int omp_int_t;
inline omp_int_t omp_get_thread_num() { return 0;}
inline omp_int_t omp_get_max_threads() { return 1;}
inline omp_int_t omp_get_num_threads() { return 1; }
#endif

#include "basic.h"
#include "vector1.h"
#include "matrix2.h"
#include "matrix2.cpp"
#include "potential.h"
#include "MD.h"
#include "Langevin.h"


// #include "BrownianGel.cpp"
// #include "BrownianGel2.cpp"
// #include "LangevinGel.cpp"
// #include "LangevinGelFixed.cpp"

// #include "NCGasR.h"
//#include "Microtubule.h"


#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include "MDGPU.cu"

using namespace std;



void prdshft(int &i, int max) {
if(i >= max) i=i-max;
else if(i<0) i=i+max;
else {}
}



int main(int argc, char** argv) {
srand (time(NULL));






int N = 10;
float2 *particles = new float2 [N];
int *cells = new int [N];
int *p_indices = new int [N];

for(int i = 0 ; i < N ; i++)
p_indices[i]=i;
// float2 *d_particles = new float2 [N];
// float2 *d_cells = new float2 [N];


for(int i = 0  ; i < N ; i++) {

float2 c;
c.x=5.*(rand()/(double)RAND_MAX);
c.y=5.*(rand()/(double)RAND_MAX);

(particles)[i]=c;
}




// cout << "particles: " << endl;
// for(int i =  0 ; i < N ; i++)
// cout << particles[i].x << " " << particles[i].y <<endl;

float2 *d_particles;
int *d_p_indices;





int size =  N*sizeof(float2);
int size2 = N*sizeof(int);


hipMalloc((void**)&d_particles,size);
hipMalloc((void**)&d_p_indices,size2);

hipMemcpy(d_particles,particles,size,hipMemcpyHostToDevice);
hipMemcpy(d_p_indices,p_indices,size2,hipMemcpyHostToDevice);

// cout << "cells: " << endl;
// for(int i =  0 ; i < N ; i++)
// cout << cells[i] << " " << p_indices_sorted[i] << endl;


int nperl = 4;
int ncells = nperl*nperl;


// cout << "cell counts" << endl;
// for(int i =  0 ; i < ncells ; i++)
// cout << i << " " << cellc2[i] << endl;


//we now have the count of each cell list
int nbpairs = 5*ncells;

int *cells1 = new int [nbpairs];
int *cells2 = new int [nbpairs];


int itery = 0;
for(int i1 = 0 ; i1 < nperl ; i1++) {
	for(int i2 = 0 ; i2 < nperl ; i2++ ) {


		int b1 =  i1*nperl+i2;

		int i3 = i1+0;
		int j3 = i2+0;

		int i4 = i1+1;
		int j4 = i2+0;

		int i5 = i1-1;
		int j5 = i2+1;

		int i6 = i1+0;
		int j6 = i2+1;

		int i7 = i1+1;
		int j7 = i2+1;

		prdshft(i3,nperl);
		prdshft(j3,nperl);

		prdshft(i4,nperl);
		prdshft(j4,nperl);

		prdshft(i5,nperl);
		prdshft(j5,nperl);
		
		prdshft(i6,nperl);
		prdshft(j6,nperl);
		
		prdshft(i7,nperl);
		prdshft(j7,nperl);		

		cells1[itery] =  b1;
		cells2[itery] =  i3*nperl+j3;

		itery++;

		cells1[itery] =  b1;
		cells2[itery] =  i4*nperl+j4;
		
		itery++;
		
		cells1[itery] =  b1;
		cells2[itery] =  i5*nperl+j5;
		
		itery++;
		
		cells1[itery] =  b1;
		cells2[itery] =  i6*nperl+j6;
		
		itery++;
		
		cells1[itery] =  b1;
		cells2[itery] =  i7*nperl+j7;

		itery++;


	}
}
int size4 = nbpairs*sizeof(int);

int *d_cells1;
int *d_cells2;

hipMalloc((void**)&d_cells1,size4);

hipMalloc((void**)&d_cells2,size4);

hipMemcpy(d_cells1,cells1,size4,hipMemcpyHostToDevice);
hipMemcpy(d_cells2,cells2,size4,hipMemcpyHostToDevice);

int *d_indices1;
int *d_indices2;
double *d_close;


int tpp;

construct_possible_pair_list(d_particles,d_p_indices,N,5.,d_cells1,d_cells2,4.,true,d_indices1,d_indices2,d_close,tpp);

// int *indices1 = new int [tpp];
// int *indices2 = new int [tpp];
// double *close = new double [tpp];
// hipMemcpy(indices1,d_indices1,tpp*sizeof(int),hipMemcpyDeviceToHost);
// hipMemcpy(indices2,d_indices2,tpp*sizeof(int),hipMemcpyDeviceToHost);
// hipMemcpy(close,d_close,tpp*sizeof(double),hipMemcpyDeviceToHost);

// for(int i = 0 ; i < tpp ; i++) {
// 	cout << indices1[i] << " " << indices2[i] << " " << close[i] << endl;
// }

// pausel();

//cout << tpp << endl;

int *d_list1;
int *d_list2;
int *d_list3;
int *d_list4;

less_than_condition_NAND cond1(1.75,0,5,5,10);


int th;
pairlist(d_indices1,d_indices2,d_close,cond1,d_list1,d_list2,d_list3,d_list4,tpp, th);





//hipFree(d_list1);
cout << tpp << endl;
cout << th << endl;

int *list1 = new int [th];
int *list2 = new int [th];
int *list3 = new int [th];
int *list4 = new int [th];

hipMemcpy(list1,d_list1,th*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(list2,d_list2,th*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(list3,d_list3,th*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(list4,d_list4,th*sizeof(int),hipMemcpyDeviceToHost);

for(int i = 0 ; i < th ; i++) {
	cout << list1[i] << " " << list2[i] << " " << list3[i] << " " << list4[i] << endl;
}

//pairlist(d_indices1,d_indices2,d_close,cond1,d_list1,d_list2,d_list3,d_list4);

// hipMemcpy(p_indices,d_p_indices,size2,hipMemcpyDeviceToHost);

// cout << "after gold" << endl;

// for(int i = 0 ; i < N ; i++)
// cout << p_indices[i] << endl;


return 0;
}