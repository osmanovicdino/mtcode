#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdarg.h>
#include <vector>
#include <algorithm>
#include <stdexcept>
#include <limits>
#include <cmath>
#include <complex>
#include <sstream>
#include <string>
#include <iomanip>
#include <sys/ioctl.h> 
#include <fcntl.h>
#include <time.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <random>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#if defined(_OPENMP)
#include <omp.h>
#else
typedef int omp_int_t;
inline omp_int_t omp_get_thread_num() { return 0;}
inline omp_int_t omp_get_max_threads() { return 1;}
inline omp_int_t omp_get_num_threads() { return 1; }
#endif

#include "basic.h"
#include "vector1.h"
#include "matrix2.h"
#include "matrix2.cpp"
#include "potential.h"
#include "MD.h"
#include "Langevin.h"


// #include "BrownianGel.cpp"
// #include "BrownianGel2.cpp"
// #include "LangevinGel.cpp"
// #include "LangevinGelFixed.cpp"

// #include "NCGasR.h"
//#include "Microtubule.h"


#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/unique.h>
#include <thrust/device_delete.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "MDGPU.cu"

using namespace std;

void create_list(int *&array) {

	int n = 5;
	hipMalloc((void**)&array,n*sizeof(int));
	int h = 1;
	setstate<<<n,1>>>(array,1,5);

}

int main(int argc, char** argv) {

int *d_list;

create_list(d_list);

print_device_array(d_list,5);

hipFree(d_list);

create_list(d_list);


print_device_array(d_list,5);


// int *d_list;

// int n = 5;
// hipMalloc((void**)&d_list,n*sizeof(int));
// int h = 0;
// hipMemset(d_list,h,n*sizeof(int));
// print_device_array(d_list,n);

hipDeviceSynchronize();

hipError_t error = hipGetLastError();
if(error != hipSuccess) {
	printf("CUDA error: %s\n",hipGetErrorString(error));
	exit(-1);
}

}