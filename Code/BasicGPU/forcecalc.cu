#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdarg.h>
#include <vector>
#include <algorithm>
#include <stdexcept>
#include <limits>
#include <cmath>
#include <complex>
#include <sstream>
#include <string>
#include <iomanip>
#include <sys/ioctl.h> 
#include <fcntl.h>
#include <time.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <random>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#if defined(_OPENMP)
#include <omp.h>
#else
typedef int omp_int_t;
inline omp_int_t omp_get_thread_num() { return 0;}
inline omp_int_t omp_get_max_threads() { return 1;}
inline omp_int_t omp_get_num_threads() { return 1; }
#endif

#include "basic.h"
#include "vector1.h"
#include "matrix2.h"
#include "matrix2.cpp"
#include "potential.h"
#include "MD.h"
#include "Langevin.h"


// #include "BrownianGel.cpp"
// #include "BrownianGel2.cpp"
// #include "LangevinGel.cpp"
// #include "LangevinGelFixed.cpp"

// #include "NCGasR.h"
//#include "Microtubule.h"


#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include "MDGPU.cu"

using namespace std;



void prdshft(int &i, int max) {
if(i >= max) i=i-max;
else if(i<0) i=i+max;
else {}
}



int main(int argc, char** argv) {
srand (time(NULL));






int N = 10;
float2 *particles = new float2 [N];
int *cells = new int [N];
int *p_indices = new int [N];
int *p_indices_sorted = new int [N];

for(int i = 0 ; i < N ; i++)
p_indices[i]=i;
// float2 *d_particles = new float2 [N];
// float2 *d_cells = new float2 [N];


matrix<double> store(N,2);

for(int i = 0  ; i < N ; i++) {

float2 c;
c.x=5.*(rand()/(double)RAND_MAX);
c.y=5.*(rand()/(double)RAND_MAX);

store(i,0)=c.x;
store(i,1)=c.y;

(particles)[i]=c;
}


vector1<bool> pb(2,true);
cube bc(5.,pb,2);
int num = floor(5.0/1.0);
LangevinNVT test(bc);


test.setdat(store);
int ccc;
matrix<int> boxes = (test).getgeo().generate_boxes_relationships(4,ccc);
matrix<int> *froyo1 = test.calculatepairs(boxes,sqrt(1.75));

// cout << "CPU pairs" << endl;
// cout << *froyo1 << endl;

WCAPotential faa(1.0,1.0,1.0); 
matrix<double> F1(test.calculateforces(*froyo1,faa));
cout << "CPU forces" << endl;
cout << F1 << endl;



// cout << "particles: " << endl;
// for(int i =  0 ; i < N ; i++)
// cout << particles[i].x << " " << particles[i].y <<endl;

float2 *d_particles;
int *d_cells;
int *d_p_indices;





int size =  N*sizeof(float2);
int size2 = N*sizeof(int);


hipMalloc((void**)&d_particles,size);
hipMalloc((void**)&d_cells,size2);
hipMalloc((void**)&d_p_indices,size2);

hipMemcpy(d_particles,particles,size,hipMemcpyHostToDevice);
hipMemcpy(d_p_indices,p_indices,size2,hipMemcpyHostToDevice);

assign_cell<<<N,1>>>(d_cells,d_particles,N,5.0,4.);

thrust::device_ptr<int> t_cells(d_cells);
thrust::device_ptr<int> t_indices(d_p_indices);

thrust::sort_by_key(t_cells,t_cells+N,t_indices);

thrust::device_vector<int>::iterator iter = thrust::max_element(t_cells,t_cells+N);

int largest = *iter;
// int largest = t_cells[iter];
// cout << "largest: " << largest << endl;

hipMemcpy(cells,d_cells,size2,hipMemcpyDeviceToHost);

hipMemcpy(p_indices_sorted,d_p_indices,size2,hipMemcpyDeviceToHost);

// cout << "cells: " << endl;
// for(int i =  0 ; i < N ; i++)
// cout << cells[i] << " " << p_indices_sorted[i] << endl;


int nperl = 4;
int ncells = nperl*nperl;
int *cellsc = new int [ncells];
for(int i = 0 ; i < ncells ; i++) {
	cellsc[i]=0;
}

int *d_cellsc;

int size3 = ncells*sizeof(int);
hipMalloc((void**)&d_cellsc,size3);
hipMemcpy(d_cellsc,cellsc,size3,hipMemcpyHostToDevice);


cell_counts<<<N,1>>>(d_cells,d_cellsc,N,largest);


int *cellc2 = new int [ncells];

hipMemcpy(cellc2,d_cellsc,size3,hipMemcpyDeviceToHost);

// cout << "cell counts" << endl;
// for(int i =  0 ; i < ncells ; i++)
// cout << i << " " << cellc2[i] << endl;


//we now have the count of each cell list
int nbpairs = 5*ncells;

int *cells1 = new int [nbpairs];
int *cells2 = new int [nbpairs];
int *npb = new int [nbpairs];

int itery = 0;
for(int i1 = 0 ; i1 < nperl ; i1++) {
	for(int i2 = 0 ; i2 < nperl ; i2++ ) {


		int b1 =  i1*nperl+i2;

		int i3 = i1+0;
		int j3 = i2+0;

		int i4 = i1+1;
		int j4 = i2+0;

		int i5 = i1-1;
		int j5 = i2+1;

		int i6 = i1+0;
		int j6 = i2+1;

		int i7 = i1+1;
		int j7 = i2+1;

		prdshft(i3,nperl);
		prdshft(j3,nperl);

		prdshft(i4,nperl);
		prdshft(j4,nperl);

		prdshft(i5,nperl);
		prdshft(j5,nperl);
		
		prdshft(i6,nperl);
		prdshft(j6,nperl);
		
		prdshft(i7,nperl);
		prdshft(j7,nperl);		

		cells1[itery] =  b1;
		cells2[itery] =  i3*nperl+j3;

		itery++;

		cells1[itery] =  b1;
		cells2[itery] =  i4*nperl+j4;
		
		itery++;
		
		cells1[itery] =  b1;
		cells2[itery] =  i5*nperl+j5;
		
		itery++;
		
		cells1[itery] =  b1;
		cells2[itery] =  i6*nperl+j6;
		
		itery++;
		
		cells1[itery] =  b1;
		cells2[itery] =  i7*nperl+j7;

		itery++;

		//+(0,0)
		//+(1,0)
		//+(-1,1)
		//+(0,1)
		//+(1,1)

	}
}

for(int i = 0 ; i < nbpairs ; i++) {
	npb[i] = 0;
}


// cout << "box checks" << endl;
// for(int i = 0 ; i < nbpairs ; i++) {
// cout << cells1[i] <<  " " << cells2[i] << endl;
// }

int size4 = nbpairs*sizeof(int);

int *d_cells1;
int *d_cells2;
int *d_npb;

hipMalloc((void**)&d_cells1,size4);

hipMalloc((void**)&d_cells2,size4);

hipMalloc((void**)&d_npb,size4);




hipMemcpy(d_cells1,cells1,size4,hipMemcpyHostToDevice);
hipMemcpy(d_cells2,cells2,size4,hipMemcpyHostToDevice);
hipMemcpy(d_npb,npb,size4,hipMemcpyHostToDevice);

neighborlist_number<<<nbpairs,1>>>(d_cells1, d_cells2, d_cellsc,d_npb,nbpairs);


thrust::device_ptr<int> t_npb(d_npb);
//int tp =  thrust::reduce(t_npb,t_npb+nbpairs);

thrust::inclusive_scan(t_npb,t_npb+nbpairs,t_npb);

hipMemcpy(npb,d_npb,size4,hipMemcpyDeviceToHost);

// cout << "no.per box" << endl;
// for(int i = 0  ; i < nbpairs ; i++)
// 	cout << cells1[i] << " " <<cells2[i] << " " << npb[i] << endl;



// cout << "total" << " " << npb[nbpairs-1] << endl;

//int tpp =  npb[nbpairs-1];


int tpp;
hipMemcpy(&tpp,d_npb+nbpairs-1,sizeof(int),hipMemcpyDeviceToHost);
//cout << tpp << endl;
//cout << hTargetVariable << endl;



int *indices1 = new int [tpp];
int *indices2 = new int [tpp];
int *close =  new int [tpp];
for(int i = 0 ; i < tpp ; i++) {
	indices1[i]=0;
	indices2[i]=0;
	close[i]=0;
}

int *d_indices1;
int *d_indices2;
int *d_close;

int size5 = tpp*sizeof(int);

hipMalloc((void**)&d_indices1,size5);

hipMalloc((void**)&d_indices2,size5);

hipMalloc((void**)&d_close,size5);

hipMemcpy(d_indices1,indices1,size5,hipMemcpyHostToDevice);
hipMemcpy(d_indices2,indices2,size5,hipMemcpyHostToDevice);
hipMemcpy(d_close,close,size5,hipMemcpyHostToDevice);


possible_neighborlist<<<nbpairs,1>>>(d_cells1, d_cells2, d_cellsc, d_p_indices, d_npb, nbpairs, d_indices1, d_indices2, d_close, d_particles,5., true,1.75);

hipMemcpy(indices1,d_indices1,size5,hipMemcpyDeviceToHost);
hipMemcpy(indices2,d_indices2,size5,hipMemcpyDeviceToHost);
hipMemcpy(close,d_close,size5,hipMemcpyDeviceToHost);

// for(int i = 0  ; i < tpp ; i++) {
// 	cout << indices1[i] <<  " " << indices2[i] << " " << close[i] << endl;
// }
// cout << "ok boomer" << endl;

thrust::device_ptr<int> t_close(d_close);
//int tp =  thrust::reduce(t_npb,t_npb+nbpairs);

thrust::inclusive_scan(t_close,t_close+tpp,t_close);

hipMemcpy(close,d_close,size5,hipMemcpyDeviceToHost);

// for(int i = 0  ; i < tpp ; i++) {
// 	cout << indices1[i] <<  " " << indices2[i] << " " << close[i] << endl;
// }

int th = close[tpp-1];
int *list1 =  new int [th];
int *list2 =  new int [th];
int *list3 =  new int [th];
int *list4 =  new int [th];

int *d_list1;
int *d_list2;
int *d_list3;
int *d_list4;

hipMalloc((void**)&d_list1,th*sizeof(int));

hipMalloc((void**)&d_list2,th*sizeof(int));

hipMalloc((void**)&d_list3,th*sizeof(int));

hipMalloc((void**)&d_list4,th*sizeof(int));

// hipMemcpy(d_list1,list1,th*sizeof(int),hipMemcpyHostToDevice);
// hipMemcpy(d_list2,list2,th*sizeof(int),hipMemcpyHostToDevice);

neighborlist<<<tpp,2>>>(d_indices1, d_indices2, d_close, tpp, d_list1, d_list2,d_list3,d_list4);

hipMemcpy(list1,d_list1,th*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(list2,d_list2,th*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(list3,d_list3,th*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(list4,d_list4,th*sizeof(int),hipMemcpyDeviceToHost);



cout << "interacting species GPU" << endl;
for(int i = 0  ; i < th ; i++) {
	cout << list1[i] <<  " " << list2[i] << endl;
}

cout << "interacting species GPU" << endl;
for(int i = 0  ; i < th ; i++) {
	cout << list3[i] <<  " " << list4[i] << endl;
}

int yt;

int *d_p_indices2;
hipMalloc((void**)&d_p_indices2,size2);
hipMemcpy(d_p_indices2,p_indices,size2,hipMemcpyHostToDevice);

int *d_list5;
int *d_list6;
int *d_list7;
int *d_list8;

construct_pair_list(d_particles,d_p_indices2,N,5.,d_cells1,d_cells2,4.,true,1.75,d_indices1,d_indices2,d_close);

hipMemcpy(list1,d_list5,th*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(list2,d_list6,th*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(list3,d_list7,th*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(list4,d_list8,th*sizeof(int),hipMemcpyDeviceToHost);



cout << "interacting species GPU from func" << endl;
for(int i = 0  ; i < th ; i++) {
	cout << list1[i] <<  " " << list2[i] << endl;
}

cout << "interacting species GPU from func" << endl;
for(int i = 0  ; i < th ; i++) {
	cout << list3[i] <<  " " << list4[i] << endl;
}


pausel();





double *d_forces1x;
double *d_forces1y;
double *d_forces2x;
double *d_forces2y;



hipMalloc((void**)&d_forces1x,th*sizeof(double));


hipMalloc((void**)&d_forces1y,th*sizeof(double));

hipMalloc((void**)&d_forces2x,th*sizeof(double));


hipMalloc((void**)&d_forces2y,th*sizeof(double));

gpupotential *inj = new gpupotential(1.0,1.0,1.0); 

gpupotential *d_inj;

hipMalloc((void**)&d_inj,sizeof(gpupotential));

hipMemcpy(d_inj,inj,sizeof(gpupotential),hipMemcpyHostToDevice);


calculateforces2D<<<th,1>>>(d_list1,d_list2,d_particles,d_forces1x,d_forces1y,d_forces2x,d_forces2y, d_inj ,th, 5.,true);

double *forces1x = new double [th];

double *forces1y = new double [th];

double *forces2x = new double [th];

double *forces2y = new double [th];


// hipMemcpy(forces1x,d_forces1x,th*sizeof(double),hipMemcpyDeviceToHost);
// hipMemcpy(forces2x,d_forces2x,th*sizeof(double),hipMemcpyDeviceToHost);

// hipMemcpy(forces1y,d_forces1y,th*sizeof(double),hipMemcpyDeviceToHost);
// hipMemcpy(forces2y,d_forces2y,th*sizeof(double),hipMemcpyDeviceToHost);

// for(int i = 0 ; i < th ; i++) {
// 	cout << list1[i] << " " << list2[i] << " " <<forces1x[i] << " " << forces1y[i] << " " << forces2x[i] << " " << forces2y[i] << endl;
// }


thrust::device_ptr<double> t_forces1x(d_forces1x);
thrust::device_ptr<double> t_forces1y(d_forces1y);
thrust::device_ptr<int> t_list1(d_list1);
thrust::device_ptr<int> t_list3(d_list3);

thrust::device_ptr<double> t_forces2x(d_forces2x);
thrust::device_ptr<double> t_forces2y(d_forces2y);
thrust::device_ptr<int> t_list2(d_list2);
thrust::device_ptr<int> t_list4(d_list4);


thrust::sort_by_key(t_list1,t_list1+th,t_forces1x);


thrust::sort_by_key(t_list2,t_list2+th,t_forces2x);

thrust::sort_by_key(t_list3,t_list3+th,t_forces1y);

thrust::sort_by_key(t_list4,t_list4+th,t_forces2y);

// hipMemcpy(list1,d_list1,th*sizeof(int),hipMemcpyDeviceToHost);
// hipMemcpy(list2,d_list2,th*sizeof(int),hipMemcpyDeviceToHost);

// hipMemcpy(forces1x,d_forces1x,th*sizeof(double),hipMemcpyDeviceToHost);
// hipMemcpy(forces2x,d_forces2x,th*sizeof(double),hipMemcpyDeviceToHost);

// hipMemcpy(forces1y,d_forces1y,th*sizeof(double),hipMemcpyDeviceToHost);
// hipMemcpy(forces2y,d_forces2y,th*sizeof(double),hipMemcpyDeviceToHost);

// cout << "sorted" << endl;

// for(int i = 0 ; i < th ; i++) {
// 	cout << list1[i] << " " << list2[i] << " " <<forces1x[i] << " " << forces1y[i] << " " << forces2x[i] << " " << forces2y[i] << endl;
// }

double *d_sumforces1x;
double *d_sumforces1y;
double *d_sumforces2x;
double *d_sumforces2y;


int *d_key_reduce1x;
int *d_key_reduce1y;
int *d_key_reduce2x;
int *d_key_reduce2y;

hipMalloc((void**)&d_sumforces1x,th*sizeof(double));

hipMalloc((void**)&d_sumforces2x,th*sizeof(double));

hipMalloc((void**)&d_sumforces1y,th*sizeof(double));

hipMalloc((void**)&d_sumforces2y,th*sizeof(double));



hipMalloc((void**)&d_key_reduce1x,th*sizeof(int));
hipMalloc((void**)&d_key_reduce2x,th*sizeof(int));
hipMalloc((void**)&d_key_reduce1y,th*sizeof(int));
hipMalloc((void**)&d_key_reduce2y,th*sizeof(int));


thrust::device_ptr<double> t_sumforces1x(d_sumforces1x);
thrust::device_ptr<double> t_sumforces2x(d_sumforces2x);
thrust::device_ptr<double> t_sumforces1y(d_sumforces1y);
thrust::device_ptr<double> t_sumforces2y(d_sumforces2y);

thrust::device_ptr<int> t_key_reduce1x(d_key_reduce1x);
thrust::device_ptr<int> t_key_reduce1y(d_key_reduce1y);
thrust::device_ptr<int> t_key_reduce2x(d_key_reduce2x);
thrust::device_ptr<int> t_key_reduce2y(d_key_reduce2y);


thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end1;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end2;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end3;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end4;

// hipMemcpy(list1,d_list1,th*sizeof(int),hipMemcpyDeviceToHost);
// hipMemcpy(list2,d_list2,th*sizeof(int),hipMemcpyDeviceToHost);


// hipMemcpy(forces1x,d_forces1x,th*sizeof(double),hipMemcpyDeviceToHost);
// hipMemcpy(forces2x,d_forces2x,th*sizeof(double),hipMemcpyDeviceToHost);

// hipMemcpy(forces1y,d_forces1y,th*sizeof(double),hipMemcpyDeviceToHost);
// hipMemcpy(forces2y,d_forces2y,th*sizeof(double),hipMemcpyDeviceToHost);

// for(int i = 0 ; i < th ; i++) {
// 	cout << list1[i] << " " << forces1x[i] << endl;
// }



new_end1 = thrust::reduce_by_key(t_list1,t_list1+th,t_forces1x,t_key_reduce1x,t_sumforces1x);


new_end2 = thrust::reduce_by_key(t_list3,t_list3+th,t_forces1y,t_key_reduce1y,t_sumforces1y);


new_end3 = thrust::reduce_by_key(t_list2,t_list2+th,t_forces2x,t_key_reduce2x,t_sumforces2x);


new_end4 = thrust::reduce_by_key(t_list4,t_list4+th,t_forces2y,t_key_reduce2y,t_sumforces2y);



// cout << "first: " << (new_end1.first) << endl;
// cout << "second: " << (new_end1.second) << endl;

// // int *raw_ptr = thrust::raw_pointer_cast(new_end1.first);
//cout << "hmm: " << thrust::raw_pointer_cast(&new_end1.second[0])-thrust::raw_pointer_cast(&t_sumforces1x[0]) << endl;
int ih1 = thrust::raw_pointer_cast(&new_end1.first[0])-thrust::raw_pointer_cast(&t_key_reduce1x[0]);
//int ih2 = thrust::raw_pointer_cast(&new_end2.first[0])-thrust::raw_pointer_cast(&t_key_reduce1y[0]);
int ih3 = thrust::raw_pointer_cast(&new_end3.first[0])-thrust::raw_pointer_cast(&t_key_reduce2x[0]);
//int ih4 = thrust::raw_pointer_cast(&new_end4.first[0])-thrust::raw_pointer_cast(&t_key_reduce2y[0]);
// cout << &raw_ptr[0] << endl;

//the output of this process will look like 

// hipMemcpy(forces1x,d_sumforces1x,th*sizeof(double),hipMemcpyDeviceToHost);


// for(int i = 0 ; i < th ; i++) {
// 	cout << list1[i] << " "  << forces1x[i] << endl;
// }

double *totalforcex = new double [N];
double *totalforcey = new double [N];

for(int i = 0  ; i < N ; i++) {
	totalforcex[i]=0.0;
	totalforcey[i]=0.0;
}


// cout << "before forces" << endl;
// for(int i = 0 ; i < N ; i++) {
// 	cout << totalforcex[i] << " " << totalforcey[i] << endl;
// }

double *d_totalforcex;
double *d_totalforcey;

hipMalloc((void**)&d_totalforcex,N*sizeof(double));

hipMalloc((void**)&d_totalforcey,N*sizeof(double));

hipMemcpy(d_totalforcex,totalforcex,N*sizeof(double),hipMemcpyHostToDevice);

hipMemcpy(d_totalforcey,totalforcey,N*sizeof(double),hipMemcpyHostToDevice);


addforce<<<ih1,1>>>(d_totalforcex,d_totalforcey, d_key_reduce1x,d_key_reduce1y, d_sumforces1x, d_sumforces1y, ih1);

addforce<<<ih3,1>>>(d_totalforcex,d_totalforcey, d_key_reduce2x,d_key_reduce2y, d_sumforces2x, d_sumforces2y, ih3);


hipMemcpy(totalforcex,d_totalforcex,N*sizeof(double),hipMemcpyDeviceToHost);

hipMemcpy(totalforcey,d_totalforcey,N*sizeof(double),hipMemcpyDeviceToHost);


cout << "reduced" << endl;

cout << "GPU forces" << endl;
for(int i = 0 ; i < N ; i++) {
	cout << totalforcex[i] << " " << totalforcey[i] << endl;
}


free(particles); free(cells);

hipFree(d_particles); hipFree(d_cells);







return 0;
}