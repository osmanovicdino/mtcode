#include "hip/hip_runtime.h"
typedef struct {
	int x,y;
} INT2;


// __global__ void assign_cell(float2 *dev_cell_list, float2 *dev_unc_pos, int nbead, double lcell, double nbox) {
// 	int global_index = blockIdx.x;

// 	if(global_index < nbead) {
// 		dev_cell_list[global_index].x = floorf(nbox*(dev_unc_pos[global_index].x)/lcell);
// 		dev_cell_list[global_index].y = floorf(nbox*(dev_unc_pos[global_index].y)/lcell);
// 	}
// }
template <class Q>
void arracychck(Q *array, int n) {
Q *temparray = new Q [n];
hipMemcpy(temparray,array,n*sizeof(Q),hipMemcpyDeviceToHost);
for(int i = 0 ; i < n ; i++) {
	if(temparray[i]!=temparray[i]) {
		cout << i << endl;
		error("nan array");
	}
}
cout << endl;
delete temparray;

}

template <class Q>
void print_device_array(Q *array, int n) {
Q *temparray = new Q [n];
hipMemcpy(temparray,array,n*sizeof(Q),hipMemcpyDeviceToHost);
for(int i = 0 ; i < n ; i++) {
	cout <<setw(4) << temparray[i] <<  ",";
}
cout << endl;
delete temparray;

}


template <class Q>
void print_device_array_indices(Q *array, int n) {
Q *temparray = new Q [n];
hipMemcpy(temparray,array,n*sizeof(Q),hipMemcpyDeviceToHost);
for(int i = 0 ; i < n ; i++) {
	if(temparray[i]>0) {
	cout <<setw(4) << i <<  ",";
	}
}
cout << endl;
delete temparray;

}


template <class Q>
void print_device_array(Q *array, int n, int m) {
Q *temparray = new Q [n];
hipMemcpy(temparray,array,n*sizeof(Q),hipMemcpyDeviceToHost);
for(int i = 0 ; i < m ; i++) {
	cout << setw(4) << temparray[i] <<  ",";
}
cout << endl;
delete temparray;

}


void print_device_float2(float2 *array, int n) {
float2 *temparray = new float2 [n];
hipMemcpy(temparray,array,n*sizeof(float2),hipMemcpyDeviceToHost);
for(int i = 0 ; i < n ; i++) {
	cout << temparray[i].x << " " << temparray[i].y <<  endl;
}
cout << endl;
delete temparray;

}

void file_print_device_float2(float2 *array, int n, ofstream &s) {
float2 *temparray = new float2 [n];
hipMemcpy(temparray,array,n*sizeof(float2),hipMemcpyDeviceToHost);
for(int i = 0 ; i < n ; i++) {
	s << temparray[i].x << "," << temparray[i].y <<  endl;
}
delete temparray;

}


void print_device_weave_float2(int *array1, int *array2, double *f1, double *f2, float2 *parts, int n, int N) {
float2 *temparray = new float2 [N];
int *temparray1 = new int [n];
int *temparray2 = new int [n];
double *temparray3 = new double [n];
double *temparray4 = new double [n];

hipMemcpy(temparray,parts,N*sizeof(float2),hipMemcpyDeviceToHost);
hipMemcpy(temparray1,array1,n*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(temparray2,array2,n*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(temparray3,f1,n*sizeof(double),hipMemcpyDeviceToHost);
hipMemcpy(temparray4,f2,n*sizeof(double),hipMemcpyDeviceToHost);

for(int i = 0 ; i < n ; i++) {
	int indx1 = temparray1[i];
	int indx2 = temparray2[i];
	double f1 = temparray3[i];
	double f2 = temparray4[i];

	cout <<"p1: " << indx1 << ": "<<temparray[indx1].x << " " << temparray[indx1].y;
	cout <<" p2: " << indx2 << ": "<<temparray[indx2].x << " " << temparray[indx2].y;
	cout << " :forces: " << f1 << " " << f2 << endl;	
}
cout << endl;
delete temparray;
delete temparray1;
delete temparray2;
delete temparray3;
delete temparray4;
}

void print_device_weave_float2(int *array1, int *array2, double *f1, double *f2, double *f3, double *f4, float2 *parts, int n, int N) {
float2 *temparray = new float2 [N];
int *temparray1 = new int [n];
int *temparray2 = new int [n];
double *temparray3 = new double [n];
double *temparray4 = new double [n];
double *temparray5 = new double [n];
double *temparray6 = new double [n];

hipMemcpy(temparray,parts,N*sizeof(float2),hipMemcpyDeviceToHost);
hipMemcpy(temparray1,array1,n*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(temparray2,array2,n*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(temparray3,f1,n*sizeof(double),hipMemcpyDeviceToHost);
hipMemcpy(temparray4,f2,n*sizeof(double),hipMemcpyDeviceToHost);
hipMemcpy(temparray5,f3,n*sizeof(double),hipMemcpyDeviceToHost);
hipMemcpy(temparray6,f4,n*sizeof(double),hipMemcpyDeviceToHost);


for(int i = 0 ; i < n ; i++) {
	int indx1 = temparray1[i];
	int indx2 = temparray2[i];
	double f1 = temparray3[i];
	double f2 = temparray4[i];
	double f3 = temparray5[i];
	double f4 = temparray6[i];

	cout <<"p1: " << indx1 << ": "<<temparray[indx1].x << " " << temparray[indx1].y;
	cout <<" p2: " << indx2 << ": "<<temparray[indx2].x << " " << temparray[indx2].y;
	cout << " :forces: " << f1 << " " << f2 << " " << f3 << " " << f4 << endl;	
}
cout << endl;
delete temparray;
delete temparray1;
delete temparray2;
delete temparray3;
delete temparray4;
delete temparray5;
delete temparray6;
}


template <class Q>
void print_device_array_weave(Q *array1, Q *array2, int n) {
Q *temparray1 = new Q [n];
Q *temparray2 = new Q [n];
hipMemcpy(temparray1,array1,n*sizeof(Q),hipMemcpyDeviceToHost);
hipMemcpy(temparray2,array2,n*sizeof(Q),hipMemcpyDeviceToHost);
for(int i = 0 ; i < n ; i++) {
	cout << temparray1[i] << "," << temparray2[i]<<  "\n";
}
cout << endl;
delete temparray1;
delete temparray2;

}

template <class Q>
void print_device_array_weave(Q *array1, Q *array2, Q *array3, Q * array4, int n) {
Q *temparray1 = new Q [n];
Q *temparray2 = new Q [n];
Q *temparray3 = new Q [n];
Q *temparray4 = new Q [n];

hipMemcpy(temparray1,array1,n*sizeof(Q),hipMemcpyDeviceToHost);
hipMemcpy(temparray2,array2,n*sizeof(Q),hipMemcpyDeviceToHost);
hipMemcpy(temparray3,array3,n*sizeof(Q),hipMemcpyDeviceToHost);
hipMemcpy(temparray4,array4,n*sizeof(Q),hipMemcpyDeviceToHost);

for(int i = 0 ; i < n ; i++) {
	cout << temparray1[i] << "," << temparray2[i]<< ","<< temparray3[i]<<"," <<temparray4[i]<<"\n";
}
cout << endl;
delete temparray1;
delete temparray2;
delete temparray3;
delete temparray4;

}

template <class Q>
__global__ void setstate(Q *a, Q i, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n) {
		a[global_index]=i;
	}
}

__global__ void setstateincr(int *a, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n) {
		int j = global_index;
		//printf("global_index is %d\n", global_index);
		a[global_index]=j;
	}
}
__global__ void setstateincr(int *a, int n, int offset) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n) {
		int j = global_index;
		//printf("global_index is %d\n", global_index);
		a[global_index]=j+offset;
	}
}

__global__ void setstate(int *a, int *b, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n) {
		a[global_index]=b[global_index];
	}	
}

__global__ void setstaterandomGPU(double *a, double lim, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n) {

		hiprandState state;

		hiprand_init(clock64(),1,0,&state);

		double rr1 = hiprand_uniform(&state);

		a[global_index] = 2*lim*rr1-lim;		
	}	
}

void setstaterandom(double *a, double lim, int n) {
	setstaterandomGPU<<<n,1>>>(a,lim,n);
}

__global__ void setstatefrom(int *a, int *b,int start, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n&& global_index >= start) {
		a[global_index]=b[global_index-start];
	}
	else if(global_index < start) {
		a[global_index]=0;
	}
}

template <class Q>
__global__ void setstatefunc(int *a, Q func, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n) {
		a[global_index]=func(global_index);
	}	
}

template <class Q>
__global__ void setstatefunc(double *a, Q func, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n) {
		a[global_index]=func(global_index);
	}	
}

template <class Q>
__global__ void setstatefunc(double *a, int *b,Q func, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n) {
		a[global_index]=func(b[global_index]);
	}	
}

__global__ void normalize(double *forcex, double *forcey, double max_s, double v0, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n) {
		double mag  = SQR(forcex[global_index])+SQR(forcey[global_index]);
		if(mag > 1E-10) {
		double fac = (mag/(max_s*v0*v0))/tanh(mag/(max_s*v0*v0));
		double rescale = 1./fac;
		forcex[global_index] = rescale * forcex[global_index];
		forcey[global_index] = rescale * forcey[global_index];
		}	
	}
// 	double v0 = (v0_a+v0_b)/2.;
// 	for(int i = 0 ; i < number_of_microtubules ; i++) { //normalize for many motors (not fully collective)
// 		if(motorsbound[i]>1E-10) {
// 			for(int j = 0 ; j < L ; j++) {
// 				double mag =0.0;
// 				for(int k = 0 ; k < dimension ; k++) {
// 					mag+=SQR(forces(na+nb+i*L+j,k));
// 				}
// //				cout << "mag: " << mag << endl;
// 				double fac = (mag/(max_s*v0*v0))/tanh(mag/(max_s*v0*v0));
// //				cout << "fac: " << fac << endl;
				
// 				double rescale = 1./fac;
// 				// cout << "rescaled: " << rescale << endl;
// 				// pausel();
// 				for(int k = 0 ; k < dimension ; k++) {
// 					forces(na+nb+i*L+j,k)=rescale*forces(na+nb+i*L+j,k);
// 				}
// 			}
// 		}
// 	}

}