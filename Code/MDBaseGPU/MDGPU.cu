#include "hip/hip_runtime.h"
// typedef struct {
// 	double x,y;
// } float2;
#include "potentialGPU.cu"
#include "potential3GPU.cu"
#include "../BasicGPU/basic.cu"


__device__ void distance_vector2D(float2 h1,float2 h2,float2 &uv, double &d, double l, bool periodic) {
	double dx = h1.x-h2.x;
	double dy = h1.y-h2.y; 
	if(periodic) { 
		double atemp = dx > 0 ? 1. : -1.;
		if(dx*dx > 0.25*l*l) {
			dx = dx - atemp*l;
		}
		double btemp = dy > 0 ? 1. : -1.;
		if(dy*dy > 0.25*l*l) {
			dy = dy - btemp*l;
		}
		// if(dx*dx > l*l ) dx = dx - SIGN(l,dx);
		// if(dy*dy > l*l ) dy = dy - SIGN(l,dy);
	}
	uv.x = dx;
	uv.y = dy;
	d = sqrt(SQR(dx)+SQR(dy));

}

__device__ void correct_position2D(float2 &h1, double l, bool periodic) {
			if(periodic) {
				if(h1.x < 0 ) h1.x = h1.x + l;
				else if(h1.x > l) h1.x = h1.x - l;
				else {
				}
				if(h1.y < 0 ) h1.y = h1.y + l;
				else if(h1.y > l) h1.y = h1.y - l;
				else {
				}				
			}
			else {
				if(h1.x<0) {
					h1.x = -h1.x;
				}
				else if(h1.y>l) {
					h1.y = l-(h1.y-l);
				}
				else{

				}				
			}
}


__global__ void assign_cell(int *dev_cell_list, float2 *dev_unc_pos, int nbead, double lcell, double nbox) {
	//particle positions in dev_unc_pos
	//dev cell list is the list of all particles with their cell
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	if(global_index < nbead) {
		dev_cell_list[global_index] = nbox*floorf(nbox*(dev_unc_pos[global_index].x)/lcell) +  floorf(nbox*(dev_unc_pos[global_index].y)/lcell);
		//dev_cell_list[global_index].y =
	}
}

__global__ void cell_counts(int *cell_id, int *cell_counts, int nbead, int cellmax) {
	int global_index = blockIdx.x;
	if(global_index>0 && global_index < nbead ) {
		if(cell_id[global_index]!=cell_id[global_index-1] && cell_id[global_index]==cellmax ) {
			//cell_counts[cell_id[global_index-1]]=global_index;
			//global_index is the point at which the mismatch occurs;
			cell_counts[cell_id[global_index-1]] = global_index;
			cell_counts[cell_id[global_index]] = nbead;
		}
		else if(cell_id[global_index]!=cell_id[global_index-1]){
			cell_counts[cell_id[global_index-1]] = global_index;
		}
		else{

		}
	}
}

//p1 is the list 
__global__ void neighborlist_number(int *p1, int *p2, int *cell_counts, int *c, int nt) {

	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	if( global_index < nt ) {
		int box1 = p1[global_index];
		int box2 = p2[global_index];



		if(box1 == box2 ) { //if the boxes are the same
			if(box1 == 0 ) { //if it is the first box
				c[global_index] = cell_counts[box1]*(cell_counts[box1]-1)/2;
			}
			else{
				if(cell_counts[box1]==0) { //empty box
					c[global_index]=0;
				}
				else {
					int index1 = cell_counts[box1];
					int box_b = box1 - 1;
					while(box_b >= 0 && cell_counts[box_b] ==0 ) {
						box_b--;
					}
					int pn;
					if(box_b==-1) pn = index1;
					else {

						int index2 = cell_counts[box_b];
						pn = index1 - index2;
					}
					c[global_index] = pn*(pn-1)/2;

				}
			}

		}
		else {
			//box1 is not box2
			if(cell_counts[box1]==0||cell_counts[box2]==0) {
				c[global_index]=0;
			}
			else {
				int index1 = cell_counts[box1];
				int box_b = box1 - 1;
				while(box_b >= 0 && cell_counts[box_b] ==0 ) {
					box_b--;
				}
				int pn1;
				if(box_b==-1) pn1 = index1;
				else {

					int index2 = cell_counts[box_b];
					pn1 = index1 - index2;
				}

				int index3 = cell_counts[box2];
				int box_b2 = box2 - 1;
				while(box_b2 >= 0 && cell_counts[box_b2] ==0 ) {
					box_b2--;
				}
				int pn2;
				if(box_b2==-1) pn2 = index3;
				else {

					int index4 = cell_counts[box_b2];
					pn2 = index3 - index4;
				}
				c[global_index] = pn1*pn2;								
			}

		}
	}

}

__global__ void possible_neighborlist(int *p1, int *p2, int *cell_counts, int *indices_sorted, int *c, int nt, int *i1, int *i2, double *dis2, float2 *positions, double ll, bool periodic) { //store the indices of the possible particles interacting in the list i1,i2
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("Hello from block %d\n",global_index);
	if( global_index < nt ) {
		// int box1 = p1[global_index];
		// int box2 = p2[global_index];
		// if(global_index==0 && c[global_index] ==0 ) {
		// 	printf("this one %d\n",global_index);
		// }
		if(global_index ==0 && c[0] == 0) {
			//printf("THIS ONE %d\n",global_index);
		}
		else if( c[global_index]-c[global_index-1]==0 ) { //don't do anything if there are no pairs
			//printf("no diff is %d\n", global_index);
			//printf("that one %d\n",global_index);
		}
		else { 
			//printf("other one %d\n",global_index);
			int box1 = p1[global_index];
			int box2 = p2[global_index];
			if(box1 == box2) { //BOXES ARE THE SAME
				int index1 = cell_counts[box1];
				int box_b = box1 - 1;
				while(box_b >= 0 && cell_counts[box_b] ==0 ) {
					box_b--;
				}
				int index2;
				if(box_b==-1) index2 = 0;
				else {
					index2 = cell_counts[box_b];
				}
				int iter = 0;
				int start;
				if(global_index==0) start = 0;
				else start = c[global_index-1];

				// int t_particle1 = indices_sorted[index2];
				// int t_particle2 = indices_sorted[index2+1];
				//printf("box1:%d ,box2: %d, index2: %d, part1: %d, part2: %d, start: %d, nt: %d, p1x: %d, p1y: %d, p2x: %d, p2y: %d, ll: %d, per: %d\n",box1,box2,index2,indices_sorted[index2],indices_sorted[index2+1],start,nt,positions[indices_sorted[index2]].x,positions[indices_sorted[index2]].y,positions[indices_sorted[index2+1]].x,positions[indices_sorted[index2+1]].y,ll,periodic);
				// printf("part1 %d, part2 %d, p1x is %d and p2x is %d, p1y is %d and p2y is %d\n", t_particle1, t_particle2, positions[t_particle1].x , positions[t_particle2].x , positions[t_particle1].y ,positions[t_particle2].y);
				//printf("Hello from global_index %d\n",global_index);
				//printf("Hello from global_index %d, value of start is: %d, iter is %d, index2 is %d and index1 is %d\n",global_index,start,iter,index2,index1);
				//printf("box is %d: ",box1);
				//printf("box: %d box same start is %d, index1 %d, index2 %d\n", box1 , start, index1 ,index2);
				for(int i = index2 ;  i < index1 ; i++) {
					for(int j = i+1 ; j < index1 ; j++) {
							int particle1 =  indices_sorted[i];
							int particle2 =  indices_sorted[j];
							

							//printf("particle1 %d, particle2 %d, start %d and iter %d index1 is %d, j is %d, p1x is %d and p2x is %d, p1y is %d and p2y is %d\n",particle1,particle2,start,iter,index1,j, positions[particle1].x , positions[particle2].x , positions[particle1].y ,positions[particle2].y);
							
							i1[start+iter]=particle1;
							i2[start+iter]=particle2;						


							double lx = positions[particle1].x-positions[particle2].x;

							//double sqrlx =  lx*lx;
							if(periodic) {
								double a = lx > 0. ? 1. : -1.;

								if(lx*lx > 0.25*ll*ll) {
								lx = lx - a*ll;
								}
							}

							double ly = positions[particle1].y-positions[particle2].y;

							//double sqrlx =  lx*lx;
							if(periodic) {
								double b = ly > 0 ? 1. : -1.;

								if(ly*ly > 0.25*ll*ll) {
									ly = ly - b*ll;
								}			
							}				

							dis2[start+iter] = lx*lx+ly*ly;
														


							iter++;
					}
				}



			}
			else{ //BOXES ARE DIFFERENT
				int index1 = cell_counts[box1];
				int box_b = box1 - 1;
				while(box_b >= 0 && cell_counts[box_b] ==0 ) {
					box_b--;
				}
				int index2;
				if(box_b==-1) index2 = 0;
				else {
					index2 = cell_counts[box_b];
				}		

				int index3 = cell_counts[box2];
				int box_b2 = box2 - 1;
				while(box_b2 >= 0 && cell_counts[box_b2] ==0 ) {
					box_b2--;
				}
				int index4;
				if(box_b2==-1) index4 = 0;
				else {
					index4 = cell_counts[box_b2];
				}
				int iter = 0;
				int start;
				if(global_index==0) start = 0;
				else start = c[global_index-1];

				// int t_particle1 = indices_sorted[index2];
				// int t_particle2 = indices_sorted[index4];
				//printf("box1:%d ,box2: %d, index2: %d, index4: %d, part1: %d, part2: %d, start: %d, nt: %d, p1x: %d, p1y: %d, p2x: %d, p2y: %d, ll: %d, per: %d\n",box1,box2,index2,index4,indices_sorted[index2],indices_sorted[index4],start,nt,positions[indices_sorted[index2]].x,positions[indices_sorted[index2]].y,positions[indices_sorted[index4]].x,positions[indices_sorted[index4]].y,ll,periodic);
				// printf("part1 %d, part2 %d, p1x is %d and p2x is %d, p1y is %d and p2y is %d\n", t_particle1, t_particle2, positions[t_particle1].x , positions[t_particle2].x , positions[t_particle1].y ,positions[t_particle2].y);
				//printf("boxes diff %d and %d : ", box1 , box2);
				//printf("box %d, box %d, box different start is %d, index1 %d, index2 %d, index3 %d, index4 %d\n", box1, box2, start, index1, index2, index3 ,index4);
				//printf("Hello from global_index %d, value of start is: %d, iter is %d, index2 is %d and index1 is %d, index 3 is %d and index4 is %d\n",global_index,start,iter,index2,index1,index3,index4);
				for(int i = index2 ;  i < index1 ; i++) {
					for(int j = index4 ; j < index3 ; j++) {	
							int particle1 =  indices_sorted[i];
							int particle2 =  indices_sorted[j];
							//printf("block index %d, thread index %d, particle1 %d, particle2 %d, start %d and iter %d i is %d, j is %d\n",blockIdx.x,threadIdx.x,particle1,particle2,start,iter,i,j);
							//printf("particle1 %d, particle2 %d, start %d and iter %d index1 is %d, index3 is %d, p1x is %d and p2x is %d, p1y is %d and p2y is %d\n",particle1,particle2,start,iter,index1,index3, positions[particle1].x , positions[particle2].x , positions[particle1].y ,positions[particle2].y);
							
							i1[start+iter]=particle1;
							i2[start+iter]=particle2;

							double lx = positions[particle1].x-positions[particle2].x;

							//double sqrlx =  lx*lx;
							if(periodic) {
								double a = lx > 0 ? 1 : -1;

								if(lx*lx > 0.25*ll*ll) {
									lx = lx - a*ll;
								}
							}
							double ly = positions[particle1].y-positions[particle2].y;

							//double sqrlx =  lx*lx;
							if(periodic) {
								double b = ly > 0 ? 1. : -1.;

								if(ly*ly > 0.25*ll*ll) {
									ly = ly - b*ll;
								}							
							}

							dis2[start+iter] = lx*lx+ly*ly;


							iter++;
					}
				}						

			}
			// for(int i = c[global_index-1] ;  i < c[global_index] ; i++) {
			// 	i1[i] = 1;
			// 	i2[i] = 1;

			// }
		}

	}

}

void construct_possible_pair_list(float2 *d_particles, int *d_p_indices, int N, double lcell, int *d_cells1, int *d_cells2, double nbox, bool periodic, int *&d_indices1, int *&d_indices2, double *&d_close, int &tpp2, bool show = false) { //Construct a full pair list with only positions as input
	//assign_cell(int *dev_cell_list, float2 *dev_unc_pos, int nbead, double lcell, double nbox)
	
	//d_particles is the storage of all the particles;
	//d_p_indices is list of all the indices;
	//N is the number of the particles
	//lcell is the length of the 
	//d_cells1,d_cells2 is the list of all the boxes that interact with each other
	//nbox is the number of boxes per 
	//periodic is if there are periodic boundary conditions
	//cut_off is the cut off of the distances

	int st = 5*nbox*nbox;


	int *d_cells;

	int size2 = N*sizeof(int);


	hipMalloc((void**)&d_cells,size2);

	assign_cell<<<N,1>>>(d_cells,d_particles,N,lcell,nbox); //assign cells to each of the particles in particles, where l is the length of the box and nbox is the number of the boxes per dimension

	//if(show) cout << "got to here 1" << endl;


	thrust::device_ptr<int> t_cells(d_cells);
	thrust::device_ptr<int> t_indices(d_p_indices); //initiliaze device pointers


	// if(show) {
	// cout << "got to here 2" << endl;
	// print_device_array(d_p_indices,N);
	// print_device_array(d_cells,N);
	// cout << N << endl;
	// pausel();
	// }

	// hipDeviceSynchronize();

	// hipError_t error = hipGetLastError();

	// if( error != hipSuccess) {
	// 	printf("Cuda Error: %s\n",hipGetErrorString(error));
	// 	exit(-1);
	// }

	thrust::sort_by_key(t_cells,t_cells+N,t_indices); //sort the indices by cell


	//if(show) cout << "got to here 3" << endl;

	thrust::device_vector<int>::iterator iter = thrust::max_element(t_cells,t_cells+N);	

	int largest = *iter;

	int ncells = nbox*nbox;

	int size3 = ncells*sizeof(int);
	int *d_cellsc;
	hipMalloc((void**)&d_cellsc,size3);
	hipMemset(d_cellsc,0,size3);


	




	cell_counts<<<N,1>>>(d_cells,d_cellsc,N,largest); //number of particles in each cell

	


	int *d_npb;

	int nbpairs = 5*ncells;
	int size4 = nbpairs*sizeof(int);
	hipMalloc((void**)&d_npb,size4);
	hipMemset(d_npb,0,size4);


	neighborlist_number<<<nbpairs,1>>>(d_cells1, d_cells2, d_cellsc,d_npb,nbpairs); //total number of possible pairs




	thrust::device_ptr<int> t_npb(d_npb);
//int tp =  thrust::reduce(t_npb,t_npb+nbpairs);

	thrust::inclusive_scan(t_npb,t_npb+nbpairs,t_npb); //cumulative binnings




	int tpp;
	hipMemcpy(&tpp,d_npb+nbpairs-1,sizeof(int),hipMemcpyDeviceToHost);
	//cout << tpp << endl;
	tpp2 = tpp;
	// int tpp = *d_tpp;
	int size5 = tpp*sizeof(int);





	hipMalloc((void**)&d_indices1,tpp*sizeof(int));

	hipMalloc((void**)&d_indices2,tpp*sizeof(int));

	hipMalloc((void**)&d_close,tpp*sizeof(double));

	setstate<<<tpp,1>>>(d_indices1,0,tpp);

	setstate<<<tpp,1>>>(d_indices2,0,tpp);

	setstate<<<tpp,1>>>(d_close,1.5,tpp);

	possible_neighborlist<<<nbpairs,1>>>(d_cells1, d_cells2, d_cellsc, d_p_indices, d_npb, nbpairs, d_indices1, d_indices2, d_close, d_particles,lcell,periodic);


	if(show) cout << "device delete" << endl;
	hipFree(d_cellsc); 
	if(show) cout << "cuda free 1" << endl;
	hipFree(d_npb); 
	if(show) cout << "cuda free 2" << endl;
	hipFree(d_cells);
	if(show) cout << "cuda free 3" << endl;



}

struct less_than_condition {
double dis_less;
__host__ __device__ less_than_condition(double dis) : dis_less(dis) {}
__host__ __device__ int operator()(int i1, int i2, double a) { 
	if(a < dis_less) return 1;
	else return 0;
}
};



struct less_than_condition_NAND {
	double dis_less;
	int p1,p2;
	int p3,p4;
	__host__ __device__ less_than_condition_NAND(double dis, int q1, int q2, int q3, int q4) : dis_less(dis),p1(q1),p2(q2),p3(q3),p4(q4) {}
	__host__ __device__ int operator()(int i1, int i2, double a) { 
		int j1,j2;
		if(i1 < i2) {j1 = i1; j2 = i2;}
		else{j1 = i2 ; j2 = i1; }

		if(j1<p2&&j1>=p1) {
			if(j2<p4&&j2>=p3){
				if(a < dis_less) {
					return 1;
				}
				else{
					return 0;
				}
			}
			else{
				return 0; 
			}
		}
		else{
			return 0;
		}

	}
};

struct less_than_condition_AND {
	double dis_less;
	int p1,p2;
	__host__ __device__ less_than_condition_AND(double dis, int q1, int q2) : dis_less(dis),p1(q1),p2(q2) {}
	__host__ __device__ int operator()(int i1, int i2, double a) {
		if(i1<p2&&i1>=p1) {
			if(i2<p2&&i2>=p1){
				if(a < dis_less) {
					return 1;
				}
				else{
					return 0;
				}
			}
			else{
				return 0; 
			}
		}
		else{
			return 0;
		}
	}
};

template <typename cond>
__global__ void applycondition(int *d_indices1, int *d_indices2,double *d_close,cond F,int N, int *res) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < N ) {
		int i1 = d_indices1[global_index];
		int i2 = d_indices2[global_index];
		double dis =  d_close[global_index];

		res[global_index]=F(i1,i2,dis);
	}
}

__global__ void neighborlist(int *p1, int *p2, int *dis, int nt, int *i1, int *i2, int *i3, int *i4) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	if(global_index ==0 ) {
		if(dis[global_index]==0) {

		}
		else{
			i1[0] = p1[global_index];
			i2[0] = p2[global_index];
			i3[0] = p1[global_index];
			i4[0] = p2[global_index];
		}

	}
	else if(global_index < nt ) {
		if(dis[global_index]!=dis[global_index-1]) {
			int fg = dis[global_index-1];
			i1[fg] = p1[global_index];
			i2[fg] = p2[global_index];
			i3[fg] = p1[global_index];
			i4[fg] = p2[global_index];			

		}

	}
	else {

	}
}


template <typename Func>
void pairlist(int *d_indices1,int *d_indices2, double *d_close,Func f,int *&d_list1,int *&d_list2,int *&d_list3,int *&d_list4, int tpp, int &th2, int show = true) {

int *d_close2;
hipMalloc((void**)&d_close2,tpp*sizeof(int));
//hipMemset(d_close2,0,tpp*sizeof(int));


setstate<<<tpp,1>>>(d_close2,0,tpp);

//resetstate<<<tpp,1>>>(d_close2,0,tpp);

//if(show) cout << "mem allocated" << endl;

applycondition<<<tpp,1>>>(d_indices1,d_indices2,d_close,f,tpp,d_close2);

//if(show) cout << "cond applied" << endl;

//if(show) print_device_array(d_close,tpp);


thrust::device_ptr<int> t_close(d_close2);

thrust::inclusive_scan(t_close,t_close+tpp,t_close);

//if(show) cout << "condscan" << endl;

int th;

hipMemcpy(&th,d_close2+tpp-1,sizeof(int),hipMemcpyDeviceToHost);

//if(show) cout << "memcpy" << endl;

th2 = th;

hipMalloc((void**)&d_list1,th*sizeof(int));

hipMalloc((void**)&d_list2,th*sizeof(int));

hipMalloc((void**)&d_list3,th*sizeof(int));

hipMalloc((void**)&d_list4,th*sizeof(int));

neighborlist<<<tpp,1>>>(d_indices1, d_indices2, d_close2, tpp, d_list1, d_list2,d_list3,d_list4);

hipFree(d_close2);


}


// __global__ void neighborlist(bool *list, int *p1, int *p2) {
// 	//p1 and p2 are a list of indices of particles which are in the correct part
// }
//given a list of 


template <typename Q>
__global__ void calculateforces2DGPU(int *i1, int *i2, float2 *positions, double *forces1x, double *forces1y, double *forces2x, double *forces2y, Q iny ,int nt, double ll, bool periodic) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;

	if(global_index < nt) {
		int particle1 = i1[global_index];
		int particle2 = i2[global_index];

		double lx = positions[particle1].x-positions[particle2].x;

		//double sqrlx =  lx*lx;
		if(periodic) {
			double a = lx > 0 ? 1 : -1;
			if(lx*lx > 0.25*ll*ll) {
				lx = lx - a*ll;
			}
		}	

		double ly = positions[particle1].y-positions[particle2].y;

		//double sqrlx =  lx*lx;
		if(periodic) {
			double b = ly > 0 ? 1 : -1;
			if(ly*ly > 0.25*ll*ll) {
				ly = ly - b*ll;
			}
		}


		double dis = sqrt(lx*lx+ly*ly);

		double f = iny(dis);

		forces1x[global_index] = lx*f/dis;
		forces1y[global_index] = ly*f/dis;

		forces2x[global_index] = -lx*f/dis;
		forces2y[global_index] = -ly*f/dis;

	}


}


template <typename Q>
__global__ void calculateforces_threebodyGPU(float2 *particles, int *list1, int *list2, int *list3,double *forcep1x, double *forcep1y, double *forcep2x, double *forcep2y,double *forcep3x, double *forcep3y,Q iny,double ll, bool periodic, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < n) {
		int p1 = list1[global_index];
		int p2 = list2[global_index];
		int p3 = list3[global_index];


		float2 a = particles[p1];

		float2 b = particles[p2];

		float2 c = particles[p3];

		float2 ab;

		float2 bc;

		double lx = b.x-a.x;

		//double sqrlx =  lx*lx;
		if(periodic) {
			double atemp = lx > 0. ? 1. : -1.;

			if(lx*lx > 0.25*ll*ll) {
				lx = lx - atemp*ll;
			}
		}
		double ly = b.y-a.y;

		//double sqrlx =  lx*lx;
		if(periodic) {
			double btemp = ly > 0. ? 1. : -1.;

			if(ly*ly > 0.25*ll*ll) {
				ly = ly - btemp*ll;
			}							
		}		
		ab.x = lx;
		ab.y = ly;

		double lx2 = c.x-b.x;

		//double sqrlx =  lx*lx;
		if(periodic) {
			double atemp = lx2 > 0. ? 1. : -1.;

			if(lx2*lx2 > 0.25*ll*ll) {
				lx2 = lx2 - atemp*ll;
			}
		}
		double ly2 = c.y-b.y;

		//double sqrlx =  lx*lx;
		if(periodic) {
			double btemp = ly2 > 0 ? 1 : -1;

			if(ly2*ly2 > 0.25*ll*ll) {
				ly2 = ly2 - btemp*ll;
			}							
		}		
		bc.x = lx2;
		bc.y = ly2;		


		float2_3 forc = iny(ab,bc);

		//printf("gin: %d, 1: %f 2: %f 3: %f 4: %f 5: %f 6: %f, 7: %f, 8: %f, 9: %f, 10: %f\n",global_index,forc.f1.x,forc.f1.y,forc.f2.x,forc.f2.y,forc.f3.x,forc.f3.y,ab.x,ab.y,bc.x,bc.y);
		//float2_3 force(float2 ab,float2 bc)

		forcep1x[global_index] = forc.f1.x;

		forcep1y[global_index] = forc.f1.y;

		forcep2x[global_index] = forc.f2.x;

		forcep2y[global_index] = forc.f2.y;

		forcep3x[global_index] = forc.f3.x;

		forcep3y[global_index] = forc.f3.y;




	}

}


template <typename Q>
void calculateforces2D(int *i1, int *i2, float2 *positions, double *&forces1x, double *&forces1y, double *&forces2x, double *&forces2y, Q iny ,int nt, double ll, bool periodic) {

calculateforces2DGPU<<<nt,1>>>(i1,i2,positions,forces1x,forces1y,forces2x,forces2y,iny ,nt,ll,periodic);

}

__global__ void addforce(double *forcex, double *forcey, int *i1, int *i2, double *forces1x, double *forces1y, int nt) {
	 int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	 if(global_index < nt) {
	 	int particle1 = i1[global_index];
		int particle2 = i2[global_index];

		forcex[particle1] += forces1x[global_index];
		forcey[particle2] += forces1y[global_index];
	 }
}


//take the forces and sum them

void ReduceForces(int *dd_list1,int *dd_list2,int *dd_list3,int *dd_list4,double *d_forces1x,double *d_forces2x,double *d_forces1y,double *d_forces2y,double *d_totalforcex,double *d_totalforcey, int th) {

int *d_list1;
int *d_list2;
int *d_list3;
int *d_list4;
hipMalloc((void**)&d_list1,th*sizeof(double));
hipMalloc((void**)&d_list2,th*sizeof(double));
hipMalloc((void**)&d_list3,th*sizeof(double));
hipMalloc((void**)&d_list4,th*sizeof(double));

setstate<<<th,1>>>(d_list1, dd_list1, th);
setstate<<<th,1>>>(d_list2, dd_list2, th);
setstate<<<th,1>>>(d_list3, dd_list3, th);
setstate<<<th,1>>>(d_list4, dd_list4, th);



thrust::device_ptr<double> t_forces1x(d_forces1x);
thrust::device_ptr<double> t_forces1y(d_forces1y);
thrust::device_ptr<int> t_list1(d_list1);
thrust::device_ptr<int> t_list3(d_list3);

thrust::device_ptr<double> t_forces2x(d_forces2x);
thrust::device_ptr<double> t_forces2y(d_forces2y);
thrust::device_ptr<int> t_list2(d_list2);
thrust::device_ptr<int> t_list4(d_list4);

thrust::sort_by_key(t_list1,t_list1+th,t_forces1x);

thrust::sort_by_key(t_list2,t_list2+th,t_forces2x);

thrust::sort_by_key(t_list3,t_list3+th,t_forces1y);

thrust::sort_by_key(t_list4,t_list4+th,t_forces2y);

double *d_sumforces1x;
double *d_sumforces1y;
double *d_sumforces2x;
double *d_sumforces2y;


int *d_key_reduce1x;
int *d_key_reduce1y;
int *d_key_reduce2x;
int *d_key_reduce2y;

hipMalloc((void**)&d_sumforces1x,th*sizeof(double));
hipMalloc((void**)&d_sumforces2x,th*sizeof(double));
hipMalloc((void**)&d_sumforces1y,th*sizeof(double));
hipMalloc((void**)&d_sumforces2y,th*sizeof(double));

hipMalloc((void**)&d_key_reduce1x,th*sizeof(int));
hipMalloc((void**)&d_key_reduce2x,th*sizeof(int));
hipMalloc((void**)&d_key_reduce1y,th*sizeof(int));
hipMalloc((void**)&d_key_reduce2y,th*sizeof(int));


thrust::device_ptr<double> t_sumforces1x(d_sumforces1x);
thrust::device_ptr<double> t_sumforces2x(d_sumforces2x);
thrust::device_ptr<double> t_sumforces1y(d_sumforces1y);
thrust::device_ptr<double> t_sumforces2y(d_sumforces2y);

thrust::device_ptr<int> t_key_reduce1x(d_key_reduce1x);
thrust::device_ptr<int> t_key_reduce1y(d_key_reduce1y);
thrust::device_ptr<int> t_key_reduce2x(d_key_reduce2x);
thrust::device_ptr<int> t_key_reduce2y(d_key_reduce2y);


thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end1;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end2;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end3;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end4;

new_end1 = thrust::reduce_by_key(t_list1,t_list1+th,t_forces1x,t_key_reduce1x,t_sumforces1x);


new_end2 = thrust::reduce_by_key(t_list3,t_list3+th,t_forces1y,t_key_reduce1y,t_sumforces1y);


new_end3 = thrust::reduce_by_key(t_list2,t_list2+th,t_forces2x,t_key_reduce2x,t_sumforces2x);


new_end4 = thrust::reduce_by_key(t_list4,t_list4+th,t_forces2y,t_key_reduce2y,t_sumforces2y);




int ih1 = thrust::raw_pointer_cast(&new_end1.first[0])-thrust::raw_pointer_cast(&t_key_reduce1x[0]);

int ih3 = thrust::raw_pointer_cast(&new_end3.first[0])-thrust::raw_pointer_cast(&t_key_reduce2x[0]);

addforce<<<ih1,1>>>(d_totalforcex,d_totalforcey, d_key_reduce1x,d_key_reduce1y, d_sumforces1x, d_sumforces1y, ih1);

addforce<<<ih3,1>>>(d_totalforcex,d_totalforcey, d_key_reduce2x,d_key_reduce2y, d_sumforces2x, d_sumforces2y, ih3);

hipFree(d_sumforces1x);
hipFree(d_sumforces2x);
hipFree(d_sumforces1y);
hipFree(d_sumforces2y);


hipFree(d_list1);
hipFree(d_list2);
hipFree(d_list3);
hipFree(d_list4);

hipFree(d_key_reduce1x);
hipFree(d_key_reduce2x);
hipFree(d_key_reduce1y);
hipFree(d_key_reduce2y);

}

void ReduceForces(int *dd_list1,int *dd_list2,double *d_forces1x,double *d_forces2x,double *d_forces1y,double *d_forces2y,double *d_totalforcex,double *d_totalforcey, int th) {

int *d_list1;
int *d_list2;
int *d_list3;
int *d_list4;
hipMalloc((void**)&d_list1,th*sizeof(double));
hipMalloc((void**)&d_list2,th*sizeof(double));
hipMalloc((void**)&d_list3,th*sizeof(double));
hipMalloc((void**)&d_list4,th*sizeof(double));

setstate<<<th,1>>>(d_list1, dd_list1, th);
setstate<<<th,1>>>(d_list2, dd_list2, th);
setstate<<<th,1>>>(d_list3, dd_list1, th);
setstate<<<th,1>>>(d_list4, dd_list2, th);



thrust::device_ptr<double> t_forces1x(d_forces1x);
thrust::device_ptr<double> t_forces1y(d_forces1y);
thrust::device_ptr<int> t_list1(d_list1);
thrust::device_ptr<int> t_list3(d_list3);

thrust::device_ptr<double> t_forces2x(d_forces2x);
thrust::device_ptr<double> t_forces2y(d_forces2y);
thrust::device_ptr<int> t_list2(d_list2);
thrust::device_ptr<int> t_list4(d_list4);

thrust::sort_by_key(t_list1,t_list1+th,t_forces1x);


thrust::sort_by_key(t_list2,t_list2+th,t_forces2x);

thrust::sort_by_key(t_list3,t_list3+th,t_forces1y);

thrust::sort_by_key(t_list4,t_list4+th,t_forces2y);

double *d_sumforces1x;
double *d_sumforces1y;
double *d_sumforces2x;
double *d_sumforces2y;


int *d_key_reduce1x;
int *d_key_reduce1y;
int *d_key_reduce2x;
int *d_key_reduce2y;

hipMalloc((void**)&d_sumforces1x,th*sizeof(double));
hipMalloc((void**)&d_sumforces2x,th*sizeof(double));
hipMalloc((void**)&d_sumforces1y,th*sizeof(double));
hipMalloc((void**)&d_sumforces2y,th*sizeof(double));



hipMalloc((void**)&d_key_reduce1x,th*sizeof(int));
hipMalloc((void**)&d_key_reduce2x,th*sizeof(int));
hipMalloc((void**)&d_key_reduce1y,th*sizeof(int));
hipMalloc((void**)&d_key_reduce2y,th*sizeof(int));


thrust::device_ptr<double> t_sumforces1x(d_sumforces1x);
thrust::device_ptr<double> t_sumforces2x(d_sumforces2x);
thrust::device_ptr<double> t_sumforces1y(d_sumforces1y);
thrust::device_ptr<double> t_sumforces2y(d_sumforces2y);

thrust::device_ptr<int> t_key_reduce1x(d_key_reduce1x);
thrust::device_ptr<int> t_key_reduce1y(d_key_reduce1y);
thrust::device_ptr<int> t_key_reduce2x(d_key_reduce2x);
thrust::device_ptr<int> t_key_reduce2y(d_key_reduce2y);


thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end1;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end2;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end3;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end4;

new_end1 = thrust::reduce_by_key(t_list1,t_list1+th,t_forces1x,t_key_reduce1x,t_sumforces1x);


new_end2 = thrust::reduce_by_key(t_list3,t_list3+th,t_forces1y,t_key_reduce1y,t_sumforces1y);


new_end3 = thrust::reduce_by_key(t_list2,t_list2+th,t_forces2x,t_key_reduce2x,t_sumforces2x);


new_end4 = thrust::reduce_by_key(t_list4,t_list4+th,t_forces2y,t_key_reduce2y,t_sumforces2y);




int ih1 = thrust::raw_pointer_cast(&new_end1.first[0])-thrust::raw_pointer_cast(&t_key_reduce1x[0]);

int ih3 = thrust::raw_pointer_cast(&new_end3.first[0])-thrust::raw_pointer_cast(&t_key_reduce2x[0]);

addforce<<<ih1,1>>>(d_totalforcex,d_totalforcey, d_key_reduce1x,d_key_reduce1y, d_sumforces1x, d_sumforces1y, ih1);

addforce<<<ih3,1>>>(d_totalforcex,d_totalforcey, d_key_reduce2x,d_key_reduce2y, d_sumforces2x, d_sumforces2y, ih3);

hipFree(d_sumforces1x);
hipFree(d_sumforces2x);
hipFree(d_sumforces1y);
hipFree(d_sumforces2y);

hipFree(d_list1);
hipFree(d_list2);
hipFree(d_list3);
hipFree(d_list4);

hipFree(d_key_reduce1x);
hipFree(d_key_reduce2x);
hipFree(d_key_reduce1y);
hipFree(d_key_reduce2y);

}


void ReduceForcesAndNormalize(int *dd_list1,double *d_forces1x,double *d_forces1y,double *d_totalforcex,double *d_totalforcey, double max_s, double v0, int th) {

int *d_list1;
int *d_list2;
hipMalloc((void**)&d_list1,th*sizeof(double));
hipMalloc((void**)&d_list2,th*sizeof(double));

setstate<<<th,1>>>(d_list1, dd_list1, th);
setstate<<<th,1>>>(d_list2, dd_list1, th);



thrust::device_ptr<double> t_forces1x(d_forces1x);
thrust::device_ptr<double> t_forces1y(d_forces1y);
thrust::device_ptr<int> t_list1(d_list1);
thrust::device_ptr<int> t_list2(d_list2);

// thrust::device_ptr<double> t_forces2x(d_forces2x);
// thrust::device_ptr<double> t_forces2y(d_forces2y);
// thrust::device_ptr<int> t_list2(d_list2);
// thrust::device_ptr<int> t_list4(d_list4);

thrust::sort_by_key(t_list1,t_list1+th,t_forces1x);


thrust::sort_by_key(t_list2,t_list2+th,t_forces1y);


double *d_sumforces1x;
double *d_sumforces1y;

int *d_key_reduce1x;
int *d_key_reduce1y;

hipMalloc((void**)&d_sumforces1x,th*sizeof(double));
hipMalloc((void**)&d_sumforces1y,th*sizeof(double));


hipMalloc((void**)&d_key_reduce1x,th*sizeof(int));
hipMalloc((void**)&d_key_reduce1y,th*sizeof(int));


thrust::device_ptr<double> t_sumforces1x(d_sumforces1x);
thrust::device_ptr<double> t_sumforces1y(d_sumforces1y);

thrust::device_ptr<int> t_key_reduce1x(d_key_reduce1x);
thrust::device_ptr<int> t_key_reduce1y(d_key_reduce1y);



thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end1;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end2;


new_end1 = thrust::reduce_by_key(t_list1,t_list1+th,t_forces1x,t_key_reduce1x,t_sumforces1x);


new_end2 = thrust::reduce_by_key(t_list2,t_list2+th,t_forces1y,t_key_reduce1y,t_sumforces1y);





int ih1 = thrust::raw_pointer_cast(&new_end1.first[0])-thrust::raw_pointer_cast(&t_key_reduce1x[0]);



normalize<<<ih1,1>>>(d_sumforces1x, d_sumforces1y,max_s, v0, ih1);


addforce<<<ih1,1>>>(d_totalforcex,d_totalforcey, d_key_reduce1x,d_key_reduce1y, d_sumforces1x, d_sumforces1y, ih1);


hipFree(d_sumforces1x);
hipFree(d_sumforces1y);

hipFree(d_list1);
hipFree(d_list2);

hipFree(d_key_reduce1x);
hipFree(d_key_reduce1y);


}



void ReduceForces3(int *dd_list1,int *dd_list2,int *dd_list3,double *d_forces1x,double *d_forces2x, double *d_forces3x,double *d_forces1y,double *d_forces2y, double *d_forces3y,double *d_totalforcex,double *d_totalforcey, int th) {

int *d_list1;
int *d_list2;
int *d_list3;
int *d_list4;
int *d_list5;
int *d_list6;
hipMalloc((void**)&d_list1,th*sizeof(double));
hipMalloc((void**)&d_list2,th*sizeof(double));
hipMalloc((void**)&d_list3,th*sizeof(double));
hipMalloc((void**)&d_list4,th*sizeof(double));
hipMalloc((void**)&d_list5,th*sizeof(double));
hipMalloc((void**)&d_list6,th*sizeof(double));



setstate<<<th,1>>>(d_list1, dd_list1, th);
setstate<<<th,1>>>(d_list2, dd_list2, th);
setstate<<<th,1>>>(d_list3, dd_list3, th);
setstate<<<th,1>>>(d_list4, dd_list1, th);
setstate<<<th,1>>>(d_list5, dd_list2, th);
setstate<<<th,1>>>(d_list6, dd_list3, th);




thrust::device_ptr<double> t_forces1x(d_forces1x);
thrust::device_ptr<double> t_forces1y(d_forces1y);
thrust::device_ptr<int> t_list1(d_list1);
thrust::device_ptr<int> t_list4(d_list4);

thrust::device_ptr<double> t_forces2x(d_forces2x);
thrust::device_ptr<double> t_forces2y(d_forces2y);
thrust::device_ptr<int> t_list2(d_list2);
thrust::device_ptr<int> t_list5(d_list5);

thrust::device_ptr<double> t_forces3x(d_forces3x);
thrust::device_ptr<double> t_forces3y(d_forces3y);
thrust::device_ptr<int> t_list3(d_list3);
thrust::device_ptr<int> t_list6(d_list6);

thrust::sort_by_key(t_list1,t_list1+th,t_forces1x);


thrust::sort_by_key(t_list2,t_list2+th,t_forces2x);

thrust::sort_by_key(t_list3,t_list3+th,t_forces3x);

thrust::sort_by_key(t_list4,t_list4+th,t_forces1y);

thrust::sort_by_key(t_list5,t_list5+th,t_forces2y);

thrust::sort_by_key(t_list6,t_list6+th,t_forces3y);

double *d_sumforces1x;
double *d_sumforces1y;
double *d_sumforces2x;
double *d_sumforces2y;
double *d_sumforces3x;
double *d_sumforces3y;


int *d_key_reduce1x;
int *d_key_reduce1y;
int *d_key_reduce2x;
int *d_key_reduce2y;
int *d_key_reduce3x;
int *d_key_reduce3y;

hipMalloc((void**)&d_sumforces1x,th*sizeof(double));
hipMalloc((void**)&d_sumforces2x,th*sizeof(double));
hipMalloc((void**)&d_sumforces1y,th*sizeof(double));
hipMalloc((void**)&d_sumforces2y,th*sizeof(double));
hipMalloc((void**)&d_sumforces3x,th*sizeof(double));
hipMalloc((void**)&d_sumforces3y,th*sizeof(double));



hipMalloc((void**)&d_key_reduce1x,th*sizeof(int));
hipMalloc((void**)&d_key_reduce2x,th*sizeof(int));
hipMalloc((void**)&d_key_reduce1y,th*sizeof(int));
hipMalloc((void**)&d_key_reduce2y,th*sizeof(int));
hipMalloc((void**)&d_key_reduce3x,th*sizeof(int));
hipMalloc((void**)&d_key_reduce3y,th*sizeof(int));

thrust::device_ptr<double> t_sumforces1x(d_sumforces1x);
thrust::device_ptr<double> t_sumforces2x(d_sumforces2x);
thrust::device_ptr<double> t_sumforces1y(d_sumforces1y);
thrust::device_ptr<double> t_sumforces2y(d_sumforces2y);
thrust::device_ptr<double> t_sumforces3x(d_sumforces3x);
thrust::device_ptr<double> t_sumforces3y(d_sumforces3y);

thrust::device_ptr<int> t_key_reduce1x(d_key_reduce1x);
thrust::device_ptr<int> t_key_reduce1y(d_key_reduce1y);
thrust::device_ptr<int> t_key_reduce2x(d_key_reduce2x);
thrust::device_ptr<int> t_key_reduce2y(d_key_reduce2y);
thrust::device_ptr<int> t_key_reduce3x(d_key_reduce3x);
thrust::device_ptr<int> t_key_reduce3y(d_key_reduce3y);


thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end1;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end2;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end3;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end4;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end5;

thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<double> > new_end6;


new_end1 = thrust::reduce_by_key(t_list1,t_list1+th,t_forces1x,t_key_reduce1x,t_sumforces1x);


new_end2 = thrust::reduce_by_key(t_list4,t_list4+th,t_forces1y,t_key_reduce1y,t_sumforces1y);


new_end3 = thrust::reduce_by_key(t_list2,t_list2+th,t_forces2x,t_key_reduce2x,t_sumforces2x);


new_end4 = thrust::reduce_by_key(t_list5,t_list5+th,t_forces2y,t_key_reduce2y,t_sumforces2y);


new_end5 = thrust::reduce_by_key(t_list3,t_list3+th,t_forces3x,t_key_reduce3x,t_sumforces3x);


new_end6 = thrust::reduce_by_key(t_list6,t_list6+th,t_forces3y,t_key_reduce3y,t_sumforces3y);




int ih1 = thrust::raw_pointer_cast(&new_end1.first[0])-thrust::raw_pointer_cast(&t_key_reduce1x[0]);

int ih3 = thrust::raw_pointer_cast(&new_end3.first[0])-thrust::raw_pointer_cast(&t_key_reduce2x[0]);

int ih5 = thrust::raw_pointer_cast(&new_end5.first[0])-thrust::raw_pointer_cast(&t_key_reduce3x[0]);

addforce<<<ih1,1>>>(d_totalforcex,d_totalforcey, d_key_reduce1x,d_key_reduce1y, d_sumforces1x, d_sumforces1y, ih1);

addforce<<<ih3,1>>>(d_totalforcex,d_totalforcey, d_key_reduce2x,d_key_reduce2y, d_sumforces2x, d_sumforces2y, ih3);

addforce<<<ih3,1>>>(d_totalforcex,d_totalforcey, d_key_reduce3x,d_key_reduce3y, d_sumforces3x, d_sumforces3y, ih5);

hipFree(d_sumforces1x);

hipFree(d_sumforces2x);

hipFree(d_sumforces1y);

hipFree(d_sumforces2y);

hipFree(d_sumforces3x);

hipFree(d_sumforces3y);

hipFree(d_list1);
hipFree(d_list2);
hipFree(d_list3);
hipFree(d_list4);
hipFree(d_list5);
hipFree(d_list6);

hipFree(d_key_reduce1x);
hipFree(d_key_reduce2x);
hipFree(d_key_reduce1y);
hipFree(d_key_reduce2y);
hipFree(d_key_reduce3x);
hipFree(d_key_reduce3y);


}
  
__global__ void advmom(double *p, double *F, double *R, int nt, double cons1, double cons2, double cons3) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x; 
	if(global_index < nt ) {
		p[global_index] =  cons1*p[global_index] + cons2*F[global_index]+cons3*R[global_index];
	} 
}

__global__ void advpos(double *x, double *p, int nt, double cons1) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x; 
	if(global_index < nt ) {
		//(*dat)(i,i1) = (*dat)(i,i1)+ c1*(*mom)(i,i1);
		x[global_index] =  x[global_index] + cons1*p[global_index];
	} 
}


__global__ void applypbc(double *x, double *p, double l, bool periodic, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x; 
	if(global_index < n) {
		if(periodic) {
			if(x[global_index] < 0) {
				x[global_index] = x[global_index]+l;
			}
			else if(x[global_index]>l) {
				x[global_index] = x[global_index]-l;
			}
			else{

			}
		}
		else{
			if(x[global_index] < 0) {
				x[global_index] = -x[global_index];
				p[global_index] = -p[global_index];
			}
			else if(x[global_index]>l) {
				x[global_index] = l-(x[global_index]-l);
				p[global_index] = -p[global_index];
			}	
			else{

			}	
		}
	}	
}

__global__ void applypbc2DGPU(float2 *x, float2 *p, double l, bool periodic, int n) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x; 
	if(global_index < n) {
		if(periodic) {
			if(x[global_index].x < 0) {
				x[global_index].x = x[global_index].x+l;
			}
			else if(x[global_index].x>l) {
				x[global_index].x = x[global_index].x-l;
			}
			else{

			}
			if(x[global_index].y < 0) {
				x[global_index].y = x[global_index].y+l;
			}
			else if(x[global_index].y>l) {
				x[global_index].y = x[global_index].y-l;
			}
			else{

			}			
		}
		else{
			if(x[global_index].x < 0) {
				x[global_index].x = -x[global_index].x;
				p[global_index].x = -p[global_index].x;
			}
			else if(x[global_index].x>l) {
				x[global_index].x = l-(x[global_index].x-l);
				p[global_index].x = -p[global_index].x;
			}	
			else{

			}	
			if(x[global_index].y < 0) {
				x[global_index].y = -x[global_index].y;
				p[global_index].y = -p[global_index].y;
			}
			else if(x[global_index].y>l) {
				x[global_index].y = l-(x[global_index].y-l);
				p[global_index].y = -p[global_index].y;
			}	
			else{

			}
		}
	}	
}

void applypbc2D(float2 *x, float2 *p, double l, bool periodic, int n) {
applypbc2DGPU<<<n,1>>>(x,p,l,periodic,n);
}


__global__ void advmom2DGPU(float2 *p, double *Fx, double *Fy, double *Rx, double *Ry, double cons1, double cons2, double cons3, int nt) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x; 
	if(global_index < nt ) {
		p[global_index].x =  cons1*p[global_index].x + cons2*Fx[global_index]+cons3*Rx[global_index];
		p[global_index].y =  cons1*p[global_index].y + cons2*Fy[global_index]+cons3*Ry[global_index];		
	} 
}

template <typename Func>
__global__ void advmom2DGPU_spatialdependence(float2 *p, float2 *x, double *Fx, double *Fy, double *Rx, double *Ry, Func fun, double dt, double kT, double m, int nt){ 
	int global_index = threadIdx.x + blockIdx.x * blockDim.x; 
	if(global_index < nt ) {

		double p_x = x[global_index].x;
		double p_y = x[global_index].y;
		double t_gamma = fun(p_x,p_y);
		double t_d = (t_gamma*dt/2.);
		double t_q = (dt)/2.;
		double t_r = sqrt(0.5*kT*(t_gamma)*(m)*(dt));

		double t_c2 = (1.0/(1.0+(t_d)));
		double t_c3 = (1.0/(1.0+(t_d)))*t_q;
		double t_c4 = (1.0/(1.0+(t_d)))*t_r;
		double t_c5 = (1-(t_d));
		p[global_index].x =  (t_c5*t_c2)*p[global_index].x + (t_c5*(t_c3)+t_q)*Fx[global_index]+(t_c5*(t_c4)+t_r)*Rx[global_index];
		p[global_index].y =  (t_c5*t_c2)*p[global_index].y + (t_c5*(t_c3)+t_q)*Fy[global_index]+(t_c5*(t_c4)+t_r)*Ry[global_index];		
	} 	
}

template <typename Func>
__global__ void advmom2DGPU_particledependence(float2 *p, double *Fx, double *Fy, double *Rx, double *Ry, Func fun, double dt, double kT, double m, int nt){ 
	int global_index = threadIdx.x + blockIdx.x * blockDim.x; 
	if(global_index < nt ) {


		double t_gamma = fun(global_index);//fun(p_x,p_y);
		double t_d = (t_gamma*dt/2.);
		double t_q = (dt)/2.;
		double t_r = sqrt(0.5*kT*(t_gamma)*(m)*(dt));

		double t_c2 = (1.0/(1.0+(t_d)));
		double t_c3 = (1.0/(1.0+(t_d)))*t_q;
		double t_c4 = (1.0/(1.0+(t_d)))*t_r;
		double t_c5 = (1-(t_d));
		p[global_index].x =  (t_c5*t_c2)*p[global_index].x + (t_c5*(t_c3)+t_q)*Fx[global_index]+(t_c5*(t_c4)+t_r)*Rx[global_index];
		p[global_index].y =  (t_c5*t_c2)*p[global_index].y + (t_c5*(t_c3)+t_q)*Fy[global_index]+(t_c5*(t_c4)+t_r)*Ry[global_index];		
	} 	
}


__global__ void advpos2DGPU(float2 *x, float2 *p, double cons1, int nt) {
	int global_index = threadIdx.x + blockIdx.x * blockDim.x; 
	if(global_index < nt ) {
		//(*dat)(i,i1) = (*dat)(i,i1)+ c1*(*mom)(i,i1);
		x[global_index].x =  x[global_index].x + cons1*p[global_index].x;
		x[global_index].y =  x[global_index].y + cons1*p[global_index].y;
	} 
}

template <typename Func>
void advmom2D_spatialdependence(float2 *p, float2 *x, double *Fx, double *Fy, double *Rx, double *Ry, Func fun, double dt, double kT, double m, int nt) { 
advmom2DGPU_spatialdependence<<<nt,1>>>(p,x,Fx,Fy,Rx,Ry,fun,dt,kT,m, nt);
}

template <typename Func>
void advmom2D_particledependence(float2 *p, double *Fx, double *Fy, double *Rx, double *Ry, Func fun, double dt, double kT, double m, int nt) { 
advmom2DGPU_particledependence<<<nt,1>>>(p,Fx,Fy,Rx,Ry,fun,dt,kT,m, nt);
}

void advmom2D(float2 *p, double *Fx, double *Fy, double *Rx, double *Ry, double cons1, double cons2, double cons3, int nt) {
advmom2DGPU<<<nt,1>>>(p,Fx,Fy,Rx,Ry,cons1,cons2,cons3, nt);
}

void advpos2D(float2 *x, float2 *p, double cons1, int nt) {
advpos2DGPU<<<nt,1>>>(x,p,cons1,nt);
}