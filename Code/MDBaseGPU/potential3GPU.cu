#include "hip/hip_runtime.h"
struct float2_3 {
	float2 f1,f2,f3;
};


// struct gpupotential3_2D {
// bool dl;
// double interaction_distance;
// __host__ __device__ virtual float2_3 force(float2,float2)=0;

// };

struct BendingPotentialGPU  {
	double k0;
	double theta0;
	bool dl;
	double interaction_distance;

	__host__ __device__ BendingPotentialGPU(double a, double b) : k0(a) , theta0(b), dl(false), interaction_distance(0.0) { }

	__host__ __device__ float2_3 operator()(float2 ab, float2 bc) {
		double a = ab.x*bc.x+ab.y*bc.y;

		double mabsq = ab.x*ab.x+ab.y*ab.y;

		double mbcsq = bc.x*bc.x+bc.y*bc.y;

		double temp1 = a/(sqrt(mabsq*mbcsq));

		if(temp1>1.) temp1 = 0.999999999;
		else if( temp1<-1 ) temp1 = -0.9999999999;
		else{ }

		double theta = acos(temp1);

		double fac1 = sqrt(1.-SQR(temp1));



		double fa = -k0*(theta-theta0);

		float2_3 res;
		if(abs(theta-theta0)<1E-10) {
			
			res.f1.x =0;
			res.f1.y =0;
			res.f2.x =0;
			res.f2.y =0;
			res.f3.x =0;
			res.f3.y =0;
			return res;
		}

		float2 f1;
		float2 f2;
		float2 f3;


		f1.x = (bc.x*mabsq-ab.x*a)/(mabsq*fac1*sqrt(mabsq)*sqrt(mbcsq));
		f1.y = (bc.y*mabsq-ab.y*a)/(mabsq*fac1*sqrt(mabsq)*sqrt(mbcsq));


		//vector1<double> f2 = ((ab - bc)*mabsq*mbcsq + a*(-(bc*mabsq) + ab*mbcsq))/(fac1*sqrt(mabsq*mbcsq)*mabsq*mbcsq);

		f2.x = ((ab.x - bc.x)*mabsq*mbcsq + a*(-(bc.x*mabsq) + ab.x*mbcsq))/(fac1*sqrt(mabsq*mbcsq)*mabsq*mbcsq);
		f2.y = ((ab.y - bc.y)*mabsq*mbcsq + a*(-(bc.y*mabsq) + ab.y*mbcsq))/(fac1*sqrt(mabsq*mbcsq)*mabsq*mbcsq);
		// f2.x = (-bc.x*mabsq*a+(-bc.x+ab.x)*mabsq+ab.x*a*mbcsq)/(mabsq*mbcsq*fac1*sqrt(mabsq)*sqrt(mbcsq));
		// f2.y = (-bc.y*mabsq*a+(-bc.y+ab.y)*mabsq+ab.y*a*mbcsq)/(mabsq*mbcsq*fac1*sqrt(mabsq)*sqrt(mbcsq));

		f3.x = (bc.x*a-ab.x*mbcsq)/(mbcsq*fac1*sqrt(mabsq)*sqrt(mbcsq));
		f3.y = (bc.y*a-ab.y*mbcsq)/(mbcsq*fac1*sqrt(mabsq)*sqrt(mbcsq));

		//printf("%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g,%.20g\n",ab.x,ab.y,bc.x,bc.y,a,mabsq,mbcsq,temp1,theta,fac1,fa,f1.x,f1.y,f2.x,f2.y,f3.x,f3.y);

		res.f1.x = fa*f1.x;
		res.f1.y = fa*f1.y;
		res.f2.x = fa*f2.x;
		res.f2.y = fa*f2.y;
		res.f3.x = fa*f3.x;
		res.f3.y = fa*f3.y;

		return res;
	}
};