#include "hip/hip_runtime.h"
__global__ void CalculateUnbindingsGPU(float2 *particles, int *bound, double *bound_along, double probunbind, int na, int nb, double L, int N, int *changestate, double l, bool periodic, double disless) { //bindings has binding info, 
	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < N) {
		if(bound[global_index]>0) {
			double tt  =  bound_along[global_index];
			int mt =  bound[global_index];
			double tt2 = tt*(double)(L-1);


			int p2 = global_index;
			int k2 = ceil(tt2);
			int k1 = k2 - 1;
			double t = tt2-k1;
			int p1a = na+nb+(mt-1)*L+k1;
			int p1b = na+nb+(mt-1)*L+k2;
			//vector1<double> a0(dimension),a1(dimension),a2(dimension);

			float2 a0,a1,a2;

			a0.x=particles[p2].x;
			a0.y=particles[p2].y;
			a1.x=particles[p1a].x;
			a1.y=particles[p1a].y;
			a2.x=particles[p1b].x;
			a2.y=particles[p1b].y;


			//vector1<double> uv3(dimension),uv4(dimension);
			double dis3,dis4;
			float2 uv3,uv4;

			distance_vector2D(a2,a1,uv3,dis3,l,periodic);

			float2 tempvector;
			tempvector.x = a1.x + t*uv3.x;
			tempvector.y = a1.y + t*uv3.y;

			correct_position2D(tempvector,l,periodic);

			hiprandState state;

			hiprand_init(clock64(),1,0,&state);

			double rr1 = hiprand_uniform(&state);

			distance_vector2D(a0,tempvector,uv4,dis4,l,periodic);

				if(abs(bound_along[global_index]-0.5)>0.4) { //drop off
					changestate[global_index] = 1;
					bound[global_index]=0;
					bound_along[global_index]=0.;
				}
				else if(rr1<probunbind) { //unbind with rate
					changestate[global_index] = 1;
					bound[global_index]=0;
					bound_along[global_index]=0.;
				}
				else if(dis4>1.2*disless) { // if distance is larger
					changestate[global_index] = 1;
					bound[global_index]=0;
					bound_along[global_index]=0.;				
				}
				else {

				} 
			// vector1<double> tempvector = a1+(t*(uv3));
			// (obj->getgeo()).correct_position(tempvector);
			// (obj->getgeo()).distance_vector(a0,tempvector,uv4,dis4);
		}
	}
}

void Microtubule::callCalculateUnbindingsGPU(float2 *particles, int *bound, double *bound_along, int *changestate) {
	int NaNb = na+nb;
	CalculateUnbindingsGPU<<<NaNb,1>>>(particles,bound,bound_along,probunbind,na,nb,L,totalN,changestate,l,is_periodic,excess_force_distance);
}


__device__ void BindGPU(int p1, int p2, float2 h1, float2 h2, int na, int nb, double L, double l, bool periodic, double disless, int initialbind, int &finalbind, double &fpos, double probbind) { //p1 is the microtubule
		int mt = 1+(int)((double)(p1-na-nb)/L); //which microtubule, starting from 1
	//	cout << 1 << endl;
		
		int LL = (int)L;
		double tt = (double)((p1-na-nb)%LL);
	//	cout << 2 << endl;
		tt = tt/(double)(L-1);
	//	cout << 3 << endl;
		// double dx = h1.x-h2.x;
		// double dy = h1.y-h2.y; 
		// if(periodic) { 
		// 	if(SQR(dx) > l*l ) dx = dx - SIGN(l,dx);
		// 	if(SQR(dy) > l*l ) dy = dy - SIGN(l,dy);
		// }
		// double dis4 = sqrt(SQR(dx)+SQR(dy));		
		float2 uv;
		double dis4;
		distance_vector2D(h1,h2,uv, dis4,l,periodic);
	//	cout << 4 << endl;
		//printf("p1: %d, p2: %d, initialbind: %d, (x1,y1) %d,%d, (x2,y2) %d,%d, tt %d\n",p1,p2,initialbind,h1.x,h1.y,h2.x,h2.y,tt);

		if(initialbind==0 && abs(tt-0.5)<0.4 && dis4 < disless ) { //only if they are close and the particle is close
			hiprandState state;

			hiprand_init(clock64(),1,0,&state);

			double rr1 = hiprand_uniform(&state);

			if(rr1<probbind) { //bind with rate
			finalbind=mt; //p2 is bound to p1;
			fpos = tt;
		//	cout << p1 << " " << p2 << " " << tt << endl;
			}
		}
}


__global__ void CalculateBindingsGPU(int *list1, int *list2, int *list3, int *list4, float2 *particles, int *bound, double *bound_along, int *changestate,int nt1, int nt2, double L,double l,bool periodic, double disless, double na, double nb, double probbind) {
	//bound is a vector of bindings between particles and the microtubule they are attached too.
	//matrix<double> 


	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index < nt1 ) {
		int p1 = list1[global_index];
		int p2 = list2[global_index];

		int fluid_particle;
		int mt_particle;

		if(p1<p2) {
			fluid_particle = p1;
			mt_particle = p2;
		}
		else{
			fluid_particle = p2;
			mt_particle = p1;
		}
		//printf("global_index: %d, mt_particle: %d, fluid_particle %d\n",global_index,mt_particle,fluid_particle);

		if(bound[fluid_particle]==0&&changestate[fluid_particle]==0) {
			float2 h1 = particles[mt_particle];
			float2 h2 = particles[fluid_particle];
			int init_bind = bound[fluid_particle];
			int fb = bound[fluid_particle];
			double ba = bound_along[fluid_particle];			
		
			float2 uv;
			double dis;
			distance_vector2D(h1,h2,uv,dis, l,periodic);
		// printf("mt_particle %d, fluid_particle %d, (x1,y1) %d,%d (x2,y2) %d,%d\n",mt_particle,fluid_particle,xtemp1,ytemp1,xtemp2,ytemp2);

			BindGPU(mt_particle,fluid_particle,h1,h2,na,nb,L,l,periodic,disless,init_bind,fb,ba, probbind); //only if unbound
			//printf("mt_particle %d, fluid_particle %d, fb %d and ba %f and dis %f\n",mt_particle,fluid_particle,fb,ba, dis);
			if(init_bind!=fb) {
			bound[fluid_particle] =  fb;
			bound_along[fluid_particle] = ba;
			}
		}

			

	}
	else if(global_index >= nt1 && global_index < nt1+nt2) {

		int p1 = list3[global_index-nt1];
		int p2 = list4[global_index-nt1];

		int fluid_particle;
		int mt_particle;

		if(p1<p2) {
			fluid_particle = p1;
			mt_particle = p2;
		}
		else{
			fluid_particle = p2;
			mt_particle = p1;
		}
		//printf("global_index: %d, mt_particle: %d, fluid_particle %d\n",global_index,mt_particle,fluid_particle);

		if(bound[fluid_particle]==0&&changestate[fluid_particle]==0) {
			float2 h1 = particles[mt_particle];
			float2 h2 = particles[fluid_particle];
			int init_bind = bound[fluid_particle];
			int fb = bound[fluid_particle];
			double ba = bound_along[fluid_particle];

			float2 uv;
			double dis;
			distance_vector2D(h1,h2,uv,dis, l,periodic);

			BindGPU(mt_particle,fluid_particle,h1,h2,na,nb,L,l,periodic,disless,init_bind,fb,ba, probbind); //only if unbound
			//printf("mt_particle %d, fluid_particle %d, fb %d and ba %f and dis %f\n",mt_particle,fluid_particle,fb,ba, dis);

			if(init_bind!=fb) {
			bound[fluid_particle] =  fb;
			bound_along[fluid_particle] = ba;
			}
		}	
	}
	else{

	}

	//cout << "calc done" << endl;

}

void Microtubule::callCalculateBindingsGPU(int *list1, int *list2, int *list3, int *list4, float2 *particles, int *bound, double *bound_along, int *changestate, int nt1, int nt2) {
	int blcks = nt1+nt2;
	CalculateBindingsGPU<<<blcks,1>>>(list1,list2,list3,list4,particles,bound,bound_along,changestate,nt1,nt2,Ld,l,is_periodic,disless,(double)na, (double)nb, probbind);
}



template <typename Q>
__global__ void BindingForcesGPUx(float2 *particles, int *bound, double *bound_along, int *con, int *indexf, int *indexp1, int *indexp2, double *forcefx, double *forcefy, double *forcep1x, double *forcep1y,double *forcep2x, double *forcep2y, Q bindp, double L, int na, int nb, double l, bool periodic, int N) {

	//output we want is a list of indices with forces in x ansd y
	int indx1 = threadIdx.x + blockIdx.x * blockDim.x;
	if(indx1> 0 && indx1 < N ) {
		int global_index = con[indx1];
		int mt = bound[global_index];

		if(mt > 0) {
			double tt  =  bound_along[global_index];
			double tt2 = tt*(double)(L-1);
			int k2 = ceil(tt2);
			int k1 = k2 - 1;	

			double t = tt2-k1;

			//na+nb+mt
			int p1a = na+nb+(mt-1)*L+k1;
			int p1b = na+nb+(mt-1)*L+k2;

			//vector1<double> a0(dimension),a1(dimension),a2(dimension);
			float2 a0,a1,a2;

			a0 = particles[global_index];
			a1 = particles[p1a];
			a2 = particles[p1b];

			double dis3=0.0;
			double dis4=0.0;
			float2 uv3,uv4;

			distance_vector2D(a2,a1,uv3,dis3,l,periodic);

			float2 tempvector;
			tempvector.x = a1.x + t*uv3.x;
			tempvector.y = a1.y + t*uv3.y;

			correct_position2D(tempvector,l,periodic);



			distance_vector2D(a0,tempvector,uv4,dis4,l,periodic);


			double f1 = 0;
			
			f1 = bindp(dis4);


			//printf("global index %d and index %d and p1a %d and p1b %d and ba: %f dis3 : %f and dis4 %f and f: %f\n ",indx1,global_index,p1a,p1b,tt,dis3,dis4,f1);

			// for(int j = 0 ; j < dimension ; j++) {
			// forces(p2,j)+=f1*uv4[j]/sqrt(dis4);
			// forces(p1a,j)+=-(1-t)*f1*uv4[j]/sqrt(dis4);
			// forces(p1b,j)+=-(t)*f1*uv4[j]/sqrt(dis4);
			// }
			//int indx1 = con[global_index];
			indexf[indx1-1] = global_index;
			indexp1[indx1-1] = p1a;
			indexp2[indx1-1] = p1b; 

			forcefx[indx1-1] = f1*uv4.x/dis4;

			forcefy[indx1-1] = f1*uv4.y/dis4;

			forcep1x[indx1-1] = -(1-t)*f1*uv4.x/dis4;

			forcep1y[indx1-1] = -(1-t)*f1*uv4.y/dis4;

			forcep2x[indx1-1] = -(t)*f1*uv4.x/dis4;

			forcep2y[indx1-1] = -(t)*f1*uv4.y/dis4;								

		}
	}

}

template<typename Q>
void Microtubule::BindingForcesGPU(float2 *particles, int *bound, double *bound_along, int *&indexf, int *&indexp1, int *&indexp2, double *&forcefx, double *&forcefy, double *&forcep1x, double *&forcep1y,double *&forcep2x, double *&forcep2y, Q bindp, int &nn) {


	int *d_bound;
	int NaNb = na+nb;
	hipMalloc((void**)&d_bound,(NaNb+1)*sizeof(int));

	setstatefrom<<<NaNb+1,1>>>(d_bound,bound,1,NaNb+1);

	thrust::device_ptr<int> t_bound(d_bound);
	thrust::inclusive_scan(t_bound,t_bound+NaNb+1,t_bound);



	int *d_index;
	hipMalloc((void**)&d_index,(NaNb+1)*sizeof(int));
	//this->resetindices(d_index,NaNb);
	setstateincr<<<NaNb+1,1>>>(d_index,NaNb+1,-1);

	thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<int> > new_end;

	thrust::device_ptr<int> t_index(d_index);
	new_end = thrust::unique_by_key(t_bound,t_bound+(NaNb+1),t_index);


	//int num_bound = new_end.first;

	int ih1 = thrust::raw_pointer_cast(&new_end.first[0])-thrust::raw_pointer_cast(&t_bound[0])-1;

	hipMalloc((void**)&indexf,(ih1)*sizeof(int));
	hipMalloc((void**)&indexp1,(ih1)*sizeof(int));
	hipMalloc((void**)&indexp2,(ih1)*sizeof(int));

	hipMalloc((void**)&forcefx,(ih1)*sizeof(double));
	hipMalloc((void**)&forcefy,(ih1)*sizeof(double));
	hipMalloc((void**)&forcep1x,(ih1)*sizeof(double));
	hipMalloc((void**)&forcep1y,(ih1)*sizeof(double));
	hipMalloc((void**)&forcep2x,(ih1)*sizeof(double));
	hipMalloc((void**)&forcep2y,(ih1)*sizeof(double));

	// cout << "state set 4" << endl;
	// pausel();
	BindingForcesGPUx<<<ih1+1,1>>>(particles, bound, bound_along, d_index, indexf, indexp1,indexp2,forcefx, forcefy, forcep1x, forcep1y,forcep2x, forcep2y,bindp,  Ld, na, nb, l, is_periodic, ih1+1);


	hipFree(d_bound);
	hipFree(d_index);

	nn = ih1;


	// cout << "state set 5" << endl;
	// pausel();	
}

template <typename Q>
void Microtubule::BendingForcesGPU(float2 *particles, int *list1, int *list2, int *list3,double *forcep1x, double *forcep1y, double *forcep2x, double *forcep2y,double *forcep3x, double *forcep3y,Q iny, int &n) {
	calculateforces_threebodyGPU<<<n,1>>>(particles, list1, list2,list3,forcep1x,forcep1y, forcep2x,forcep2y, forcep3x, forcep3y,iny,l, is_periodic, n);
}


__global__ void PositionForcesDueToAnglesGPUx(float2 *particles, int *bound_indices, double *pol, double *v0, int *bound, double *bound_along, int L, double l, bool periodic, double gamma, double dt, int *list1, double *forcep1x, double *forcep1y,int na, int nb, int n) {


	int global_index = threadIdx.x + blockIdx.x * blockDim.x;
	if(global_index > 0 && global_index < n) {
		int indx1 = bound_indices[global_index];

		double tt  =  bound_along[indx1];

		int mt =  bound[indx1];
		double tt2 = tt*(double)(L-1);
		int k2 = ceil(tt2);
		int k1 = k2-1;

		int p1a = na+nb+(mt-1)*L+k1;
		int p1b = na+nb+(mt-1)*L+k2;

		float2 a1,a2;

		a1 = particles[p1a];
		a2 = particles[p1b];

		double dis3;
		float2 uv3;

		distance_vector2D(a1,a2,uv3,dis3,l,periodic);

		uv3.x = uv3.x/dis3;

		uv3.y = uv3.y/dis3;

		double v0_a = v0[global_index];

		double incr = dt*v0_a/gamma;

		double polarity_a = pol[global_index];

		bound_along[indx1] = bound_along[indx1] - polarity_a*incr/((double)(L-1));

	//	printf("global_index: %d and index %d and mt: %d, v0  %d and polarity %d\n",global_index,indx1,mt,v0_a,polarity_a);

		if(bound_along[indx1]<0) bound_along[indx1]=0;

		for(int j = 0 ; j < L ; j++) {
			list1[(global_index-1)*L+j]=na+nb+(mt-1)*L+j;
			forcep1x[(global_index-1)*L+j]=-polarity_a*0.5*v0_a*uv3.x; 
			forcep1y[(global_index-1)*L+j]=-polarity_a*0.5*v0_a*uv3.y;
		}		

		
	}



}

struct polarityfunction {
	int n;
	double pa;
	double pb;

	__host__ __device__ polarityfunction(int nn, double paa, double pbb) : n(nn),pa(paa),pb(pbb) {}
	__host__ __device__ double operator()(int i) {
		if(i<n ) return pa;
		else return pb;
	}
};

void Microtubule::PositionForcesDueToAnglesGPU(float2 *particles, int *bound, double *bound_along, int *&list1, double *&forcep1x, double *&forcep1y, int &n) {
	int *d_bound;
	int NaNb = na+nb;
	hipMalloc((void**)&d_bound,(NaNb+1)*sizeof(int));

	setstatefrom<<<NaNb+1,1>>>(d_bound,bound,1,NaNb+1);

	thrust::device_ptr<int> t_bound(d_bound);
	thrust::inclusive_scan(t_bound,t_bound+NaNb+1,t_bound);



	int *d_index;
	hipMalloc((void**)&d_index,(NaNb+1)*sizeof(int));
	//this->resetindices(d_index,NaNb);
	setstateincr<<<NaNb+1,1>>>(d_index,NaNb+1,-1);

	thrust::pair<thrust::device_ptr<int>,thrust::device_ptr<int> > new_end;

	thrust::device_ptr<int> t_index(d_index);
	new_end = thrust::unique_by_key(t_bound,t_bound+(NaNb+1),t_index);


	//int num_bound = new_end.first;

	int ih1 = thrust::raw_pointer_cast(&new_end.first[0])-thrust::raw_pointer_cast(&t_bound[0])-1;


	int ih2 = L*ih1;
	n = ih2;
	if(ih1 > 0) {

	hipMalloc((void**)&list1,(ih2)*sizeof(int));
	hipMalloc((void**)&forcep1x,(ih2)*sizeof(double));
	hipMalloc((void**)&forcep1y,(ih2)*sizeof(double));


	polarityfunction fg(na,polarity_a,polarity_b);
	//setpolarity fg(na,polarity_a,polarity_b);
	// fg.n = na;
	// fg.pa = polarity_a;
	// fg.pb = polarity_b;

	double *d_pol;
	hipMalloc((void**)&d_pol,(ih1+1)*sizeof(double));
	setstatefunc<<<ih1+1,1>>>(d_pol,d_index,fg,ih1+1);


	polarityfunction fg2(na,v0_a,v0_b);
	// fg2.n = na;
	// fg2.pa = v0_a;
	// fg2.pb = v0_b;	
	
	double *d_v0;
	hipMalloc((void**)&d_v0,(ih1+1)*sizeof(double));
	setstatefunc<<<ih1+1,1>>>(d_v0,d_index,fg2,ih1+1);





	PositionForcesDueToAnglesGPUx<<<ih1+1,1>>>(particles, d_index, d_pol, d_v0,bound, bound_along, L, l, is_periodic, gamma, dt, list1, forcep1x, forcep1y,na,nb,ih1+1);


	hipFree(d_v0);
	hipFree(d_pol);
	}
	hipFree(d_bound);



}

void prdshft(int &i, int max) {
if(i >= max) i=i-max;
else if(i<0) i=i+max;
else {}
}

void Microtubule::resetchangestate(int *state) {
	int NaNb = na + nb;
	setstate<<<NaNb,1>>>(state,0,na+nb);
}

void Microtubule::resetforce(double *state) {
	double initforce = 0.0;
	setstate<<<totalN,1>>>(state,initforce,totalN);
}


void Microtubule::resetindices(int *index,int n) {
	setstateincr<<<n,1>>>(index,n);
}
