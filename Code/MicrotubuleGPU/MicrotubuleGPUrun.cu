#include "hip/hip_runtime.h"

void Microtubule::runGPU(int runtime, int every)
{
//	pausel();
	//int ccc;
	int totalN = obj->getN();

	//num is the number of boxes per length

	int ncells = num*num;

	WCApotentialGPU faa_gpu(2.,1.,2.);
	WCApotentialGPU fab_gpu(1.,1.,0.);
	WCApotentialGPU fac_gpu(1.,1.,0.);
	WCApotentialGPU fbb_gpu(2.,1.,2.);
	WCApotentialGPU fbc_gpu(1.,1.,0.);
	WCApotentialGPU fcc_gpu(1.,1.,0.);
	HarmonicPotentialGPU bindp_gpu(100.,1.); 
	FENEPotentialGPU bindm_gpu(50.,1.5); 
	BendingPotentialGPU bendp_gpu(100.,0.);

//we now have the count of each cell list
	int nbpairs = 5*ncells;
	int nperl = num;

	int *cells1 = new int [nbpairs];
	int *cells2 = new int [nbpairs];


	int itery = 0;
	for(int i1 = 0 ; i1 < num ; i1++) {
		for(int i2 = 0 ; i2 < num ; i2++ ) {


			int b1 =  i1*nperl+i2;

			int i3 = i1+0;
			int j3 = i2+0;

			int i4 = i1+1;
			int j4 = i2+0;

			int i5 = i1-1;
			int j5 = i2+1;

			int i6 = i1+0;
			int j6 = i2+1;

			int i7 = i1+1;
			int j7 = i2+1;

			prdshft(i3,nperl);
			prdshft(j3,nperl);

			prdshft(i4,nperl);
			prdshft(j4,nperl);

			prdshft(i5,nperl);
			prdshft(j5,nperl);
			
			prdshft(i6,nperl);
			prdshft(j6,nperl);
			
			prdshft(i7,nperl);
			prdshft(j7,nperl);		

			cells1[itery] =  b1;
			cells2[itery] =  i3*nperl+j3;

			itery++;

			cells1[itery] =  b1;
			cells2[itery] =  i4*nperl+j4;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i5*nperl+j5;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i6*nperl+j6;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i7*nperl+j7;

			itery++;


		}
	}
	int size4 = nbpairs*sizeof(int);

	int *d_cells1;
	int *d_cells2;

	hipMalloc((void**)&d_cells1,size4);

	hipMalloc((void**)&d_cells2,size4);

	hipMemcpy(d_cells1,cells1,size4,hipMemcpyHostToDevice);
	hipMemcpy(d_cells2,cells2,size4,hipMemcpyHostToDevice);
	//int sibdiv = floor(ll/4.0);
	// print_device_array(d_cells1,nbpairs);
	// print_device_array(d_cells2,nbpairs);



	// matrix<int> boxes = (obj)->getgeo().generate_boxes_relationships(num,ccc);
	
	float2 *particles = new float2 [totalN];
	float2 *momenta = new float2 [totalN];
	int *p_indices = new int [totalN];

	for(int i = 0 ; i < totalN ; i++)
	p_indices[i]=i;

	float2 *d_particles;
	float2 *d_momenta;
	int *d_p_indices;

	int *d_bound;
	double *d_boundalong;
	int *d_changestate;

	hipMalloc((void**)&d_bound,(na+nb)*sizeof(int));
	hipMalloc((void**)&d_boundalong,(na+nb)*sizeof(double));
	hipMalloc((void**)&d_changestate,(na+nb)*sizeof(int));

	hipMemset(d_bound,0,(na+nb)*sizeof(int));
	hipMemset(d_boundalong,0.,(na+nb)*sizeof(double));
	hipMemset(d_changestate,0,(na+nb)*sizeof(int));

	double *d_totalforcex;
	double *d_totalforcey;

	hipMalloc((void**)&d_totalforcex,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey,totalN*sizeof(double));


	matrix<double> state(obj->getdat());


	for(int i = 0  ; i < totalN ; i++) {

	float2 c;
	c.x=state(i,0);
	c.y=state(i,1);

	(particles)[i]=c;

	float2 d;

	d.x = 0.;
	d.y = 0.;

	(momenta)[i]=d;
	}


	int size =  totalN*sizeof(float2);
	int size2 = totalN*sizeof(int);


	hipMalloc((void**)&d_particles,size);
	hipMalloc((void**)&d_momenta,size);
	hipMalloc((void**)&d_p_indices,size2);

	hipMemcpy(d_particles,particles,size,hipMemcpyHostToDevice);
	hipMemcpy(d_momenta,momenta,size,hipMemcpyHostToDevice);
	hipMemcpy(d_p_indices,p_indices,size2,hipMemcpyHostToDevice);


	// matrix<int> *froyo1 = obj->calculatepairs(boxes,pai,3.5);
	// matrix<int> *froyo2 = obj->calculatepairs(boxes,pbi,3.5);
	// matrix<int> *froyo3 = obj->calculatepairs(boxes,pci,3.5);
	// matrix<int> *froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
	// matrix<int> *froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
	// matrix<int> *froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);
	
	int *d_indices1;
	int *d_indices2;
	double *d_close;


	int tpp;
	


	construct_possible_pair_list(d_particles,d_p_indices,totalN,l,d_cells1,d_cells2,num,is_periodic,d_indices1,d_indices2,d_close,tpp);



	less_than_condition_AND cond1(SQR(3.5),0,na);
	less_than_condition_AND cond2(SQR(3.5),na,na+nb);
	less_than_condition_AND cond3(SQR(3.5),na+nb,na+nb+nc);
	less_than_condition_NAND cond4(SQR(3.5),0,na,na,na+nb);
	less_than_condition_NAND cond5(SQR(3.5),0,na,na+nb,na+nb+nc);
	less_than_condition_NAND cond6(SQR(3.5),na,na+nb,na+nb,na+nb+nc);


	int th1;
	int *d1_list1,*d1_list2,*d1_list3,*d1_list4;
	pairlist(d_indices1,d_indices2,d_close,cond1,d1_list1,d1_list2,d1_list3,d1_list4,tpp, th1); //faa

	int th2;
	int *d2_list1,*d2_list2,*d2_list3,*d2_list4;
	pairlist(d_indices1,d_indices2,d_close,cond2,d2_list1,d2_list2,d2_list3,d2_list4,tpp, th2);	//fbb

	int th3;
	int *d3_list1,*d3_list2,*d3_list3,*d3_list4;
	pairlist(d_indices1,d_indices2,d_close,cond3,d3_list1,d3_list2,d3_list3,d3_list4,tpp, th3);	//fcc

	int th4;
	int *d4_list1,*d4_list2,*d4_list3,*d4_list4;
	pairlist(d_indices1,d_indices2,d_close,cond4,d4_list1,d4_list2,d4_list3,d4_list4,tpp, th4);	//fab

	int th5;
	int *d5_list1,*d5_list2,*d5_list3,*d5_list4;
	pairlist(d_indices1,d_indices2,d_close,cond5,d5_list1,d5_list2,d5_list3,d5_list4,tpp, th5);	//fac

	int th6;
	int *d6_list1,*d6_list2,*d6_list3,*d6_list4;
	pairlist(d_indices1,d_indices2,d_close,cond6,d6_list1,d6_list2,d6_list3,d6_list4,tpp, th6); //fbc			
	//matrix<double> state(obj->getdat()); //the state of the system

	int th8 = (*bondpairs).getNsafe();
	int th9 = (*bendtriplets).getNsafe();


	int *d8_list1,*d8_list2,*d8_list3,*d8_list4;
	hipMalloc((void**)&d8_list1,th8*sizeof(int));
	hipMalloc((void**)&d8_list2,th8*sizeof(int));
	hipMalloc((void**)&d8_list3,th8*sizeof(int));
	hipMalloc((void**)&d8_list4,th8*sizeof(int));



	int *h8_list1 = new int [th8];
	int *h8_list2 = new int [th8];

	for(int i = 0 ; i < th8 ; i++ ) {
		h8_list1[i] = (*bondpairs)(i,0);
		h8_list2[i] = (*bondpairs)(i,1);
	}

	hipMemcpy(d8_list1,h8_list1,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list2,h8_list2,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list3,h8_list1,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list4,h8_list2,th8*sizeof(int),hipMemcpyHostToDevice);





	int *d9_list1,*d9_list2,*d9_list3,*d9_list4,*d9_list5,*d9_list6;
	hipMalloc((void**)&d9_list1,th9*sizeof(int));
	hipMalloc((void**)&d9_list2,th9*sizeof(int));
	hipMalloc((void**)&d9_list3,th9*sizeof(int));
	hipMalloc((void**)&d9_list4,th9*sizeof(int));
	hipMalloc((void**)&d9_list5,th9*sizeof(int));
	hipMalloc((void**)&d9_list6,th9*sizeof(int));

	int *h9_list1 = new int [th9];
	int *h9_list2 = new int [th9];
	int *h9_list3 = new int [th9];

	for(int i = 0 ; i < th9 ; i++ ) {
		h9_list1[i] = (*bendtriplets)(i,0);
		h9_list2[i] = (*bendtriplets)(i,1);
		h9_list3[i] = (*bendtriplets)(i,2);
	}	

	hipMemcpy(d9_list1,h9_list1,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list2,h9_list2,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list3,h9_list3,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list4,h9_list1,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list5,h9_list2,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list6,h9_list3,th9*sizeof(int),hipMemcpyHostToDevice);


	int i;

	double cons1;
	double cons2;
	double cons3;
	double cons4;

	//(mom)->operator()(i,i1) = c5*c2*((mom)->operator()(i,i1)) + (c5*(c3)+q)*F(i,i1) + (c5*(c4)+r)*R(i,i1);
	cons1 = (*obj).getc5()*(*obj).getc2();
	cons2 = (*obj).getc5()*(*obj).getc3()+(*obj).getq();
	cons3 = (*obj).getc5()*(*obj).getc4()+(*obj).getr();
	cons4 = (*obj).getc1();

	double d_dt = (*obj).getdt();
	double d_m = (*obj).getm();
	double d_kT = (*obj).getkT();


	// vector<matrix<double> > savef1forces;
	// vector<matrix<double> > savef2forces;
	// vector<matrix<double> > savef3forces;
	// vector<matrix<double> > savef4forces;
	// vector<matrix<double> > savef5forces;
	// vector<matrix<double> > savef6forces;
	// vector<matrix<double> > savef7forces;

	// vector<matrix<double> > saveftemp1forces;
	// vector<matrix<double> > saveftemp2forces;
	// vector<matrix<double> > saveftemp3forces;

	// vector<matrix<double> > savepositions;
	// vector<vector1<int> > savebound;
	// vector<vector1<double> > saveboundalongs;

	for(i = 0 ; i < runtime ; i++) {
		//cout << i << endl;

		cout << i << endl;
	
		//cout << (*obj).avmom() << endl;
	if(i%25==0) {
		//delete froyo1,froyo2,froyo3,froyo4,froyo5,froyo6;
		// cout << "updated after: " << i << endl;
		// state = obj->getdat();


		hipFree(d1_list1);hipFree(d1_list2);hipFree(d1_list3);hipFree(d1_list4);
		

		hipFree(d2_list1);hipFree(d2_list2);hipFree(d2_list3);hipFree(d2_list4);
		

		hipFree(d3_list1);hipFree(d3_list2);hipFree(d3_list3);hipFree(d3_list4);
		

		hipFree(d4_list1);hipFree(d4_list2);hipFree(d4_list3);hipFree(d4_list4);
		

		hipFree(d5_list1);hipFree(d5_list2);hipFree(d5_list3);hipFree(d5_list4);
	
		hipFree(d6_list1);hipFree(d6_list2);hipFree(d6_list3);hipFree(d6_list4);

		hipFree(d_indices1);

		hipFree(d_indices2);

		hipFree(d_close);

		this->resetindices(d_p_indices,totalN);


		construct_possible_pair_list(d_particles,d_p_indices,totalN,l,d_cells1,d_cells2,num,is_periodic,d_indices1,d_indices2,d_close,tpp,false);

		// cout << tpp << endl;
		// cout << "pair" << endl;

		pairlist(d_indices1,d_indices2,d_close,cond1,d1_list1,d1_list2,d1_list3,d1_list4,tpp, th1); //faa
		pairlist(d_indices1,d_indices2,d_close,cond2,d2_list1,d2_list2,d2_list3,d2_list4,tpp, th2); //fbb
		pairlist(d_indices1,d_indices2,d_close,cond3,d3_list1,d3_list2,d3_list3,d3_list4,tpp, th3); //fcc
		pairlist(d_indices1,d_indices2,d_close,cond4,d4_list1,d4_list2,d4_list3,d4_list4,tpp, th4); //fab
		pairlist(d_indices1,d_indices2,d_close,cond5,d5_list1,d5_list2,d5_list3,d5_list4,tpp, th5); //fac
		pairlist(d_indices1,d_indices2,d_close,cond6,d6_list1,d6_list2,d6_list3,d6_list4,tpp, th6); //fbc

	}




	double *d1_forces1x;
	double *d1_forces2x;
	double *d1_forces1y;
	double *d1_forces2y;
	hipMalloc((void**)&d1_forces1x,th1*sizeof(double));
	hipMalloc((void**)&d1_forces1y,th1*sizeof(double));
	hipMalloc((void**)&d1_forces2x,th1*sizeof(double));
	hipMalloc((void**)&d1_forces2y,th1*sizeof(double));
	calculateforces2D(d1_list1,d1_list2,d_particles, d1_forces1x,d1_forces1y,d1_forces2x,d1_forces2y, faa_gpu ,th1, l,true);

	// arracychck(d1_forces1x,th1);
	// arracychck(d1_forces1y,th1);
	// cout << "force1" << endl;

	double *d2_forces1x;
	double *d2_forces2x;
	double *d2_forces1y;
	double *d2_forces2y;
	hipMalloc((void**)&d2_forces1x,th2*sizeof(double));
	hipMalloc((void**)&d2_forces1y,th2*sizeof(double));
	hipMalloc((void**)&d2_forces2x,th2*sizeof(double));
	hipMalloc((void**)&d2_forces2y,th2*sizeof(double));
	calculateforces2D(d2_list1,d2_list2,d_particles, d2_forces1x,d2_forces1y,d2_forces2x,d2_forces2y, fbb_gpu ,th2, l,true);	



	// arracychck(d2_forces1x,th2);
	// arracychck(d2_forces1y,th2);

	// cout << "force2" << endl;
	double *d3_forces1x;
	double *d3_forces2x;
	double *d3_forces1y;
	double *d3_forces2y;
	hipMalloc((void**)&d3_forces1x,th3*sizeof(double));
	hipMalloc((void**)&d3_forces1y,th3*sizeof(double));
	hipMalloc((void**)&d3_forces2x,th3*sizeof(double));
	hipMalloc((void**)&d3_forces2y,th3*sizeof(double));
	calculateforces2D(d3_list1,d3_list2,d_particles, d3_forces1x,d3_forces1y,d3_forces2x,d3_forces2y, fcc_gpu ,th3, l,true);	

	// arracychck(d3_forces1x,th3);
	// arracychck(d3_forces1y,th3);

	// cout << "force3" << endl;
	double *d4_forces1x;
	double *d4_forces2x;
	double *d4_forces1y;
	double *d4_forces2y;
	hipMalloc((void**)&d4_forces1x,th4*sizeof(double));
	hipMalloc((void**)&d4_forces1y,th4*sizeof(double));
	hipMalloc((void**)&d4_forces2x,th4*sizeof(double));
	hipMalloc((void**)&d4_forces2y,th4*sizeof(double));
	calculateforces2D(d4_list1,d4_list2,d_particles, d4_forces1x,d4_forces1y,d4_forces2x,d4_forces2y, fab_gpu ,th4, l,true);
	
	// arracychck(d4_forces1x,th4);
	// arracychck(d4_forces1y,th4);
	// cout << "force4" << endl;

	double *d5_forces1x;
	double *d5_forces2x;
	double *d5_forces1y;
	double *d5_forces2y;
	hipMalloc((void**)&d5_forces1x,th5*sizeof(double));
	hipMalloc((void**)&d5_forces1y,th5*sizeof(double));
	hipMalloc((void**)&d5_forces2x,th5*sizeof(double));
	hipMalloc((void**)&d5_forces2y,th5*sizeof(double));
	calculateforces2D(d5_list1,d5_list2,d_particles, d5_forces1x,d5_forces1y,d5_forces2x,d5_forces2y, fbc_gpu ,th5, l,true);
	// arracychck(d5_forces1x,th5);
	// arracychck(d5_forces1y,th5);
	// cout << "force5" << endl;
	// cout << "d5" << endl;
	// print_device_weave_float2(d5_list1,d5_list2,d5_forces1x,d5_forces1y,d5_forces2x,d5_forces2y,d_particles,th5,totalN);
	// pausel();	

	double *d6_forces1x;
	double *d6_forces2x;
	double *d6_forces1y;
	double *d6_forces2y;
	hipMalloc((void**)&d6_forces1x,th6*sizeof(double));
	hipMalloc((void**)&d6_forces1y,th6*sizeof(double));
	hipMalloc((void**)&d6_forces2x,th6*sizeof(double));
	hipMalloc((void**)&d6_forces2y,th6*sizeof(double));
	calculateforces2D(d6_list1,d6_list2,d_particles, d6_forces1x,d6_forces1y,d6_forces2x,d6_forces2y, fac_gpu ,th6, l,true);	
	// arracychck(d6_forces1x,th6);
	// arracychck(d6_forces1y,th6);
	// cout << "force6" << endl;
	// cout << "d6" << endl;
	// print_device_weave_float2(d6_list1,d6_list2,d6_forces1x,d6_forces1y,d6_forces2x,d6_forces2y,d_particles,th6,totalN);
	// cout << endl;
	// pausel();

	// matrix<double> F6((*obj).calculateforces(*bondpairs,*bindm));
	callCalculateUnbindingsGPU(d_particles,d_bound,d_boundalong,d_changestate);



	// cout << "unbindings calculated" << endl;


	callCalculateBindingsGPU(d5_list1,d5_list2,d6_list1,d6_list2,d_particles, d_bound, d_boundalong,d_changestate,th5 ,th6 );


	// cout << "bindings calculated" << endl;


	int *d7_list1,*d7_list2,*d7_list3;
	double *d7_forces1x;
	double *d7_forces1y;
	double *d7_forces2x;
	double *d7_forces2y;
	double *d7_forces3x;
	double *d7_forces3y;
	int th7;
	BindingForcesGPU(d_particles, d_bound, d_boundalong, d7_list1,d7_list2,d7_list3, d7_forces1x, d7_forces1y, d7_forces2x,d7_forces2y,d7_forces3x, d7_forces3y, bindp_gpu, th7);

	// arracychck(d7_forces1x,th7);
	// arracychck(d7_forces1y,th7);
	// cout << "binding forces calculated" << endl;


	double *d8_forces1x;
	double *d8_forces2x;
	double *d8_forces1y;
	double *d8_forces2y;
	hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	hipMalloc((void**)&d8_forces2y,th8*sizeof(double));
	calculateforces2D(d8_list1,d8_list2,d_particles, d8_forces1x,d8_forces1y,d8_forces2x,d8_forces2y, bindm_gpu ,th8, l,true);
	
	// arracychck(d8_forces1x,th8);
	// arracychck(d8_forces1y,th8);

	// cout << "force 8" << endl;
	resetchangestate(d_changestate);
	// matrix<double> F7((*obj).calculateforces_threebody(*bendtriplets,*bendp));

//	int th9;


	double *d9_forces1x;
	double *d9_forces2x;
	double *d9_forces1y;
	double *d9_forces2y;
	double *d9_forces3x;
	double *d9_forces3y;
	// hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2y,th8*sizeof(double));	
	hipMalloc((void**)&d9_forces1x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces1y,th9*sizeof(double));
	hipMalloc((void**)&d9_forces2x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces2y,th9*sizeof(double));
	hipMalloc((void**)&d9_forces3x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces3y,th9*sizeof(double));
	BendingForcesGPU(d_particles, d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces1y,d9_forces2x,d9_forces2y, d9_forces3x, d9_forces3y,bendp_gpu,th9);

	// print_device_array_weave(d9_forces1x,d9_forces1y,th9);
	// print_device_array_weave(d9_forces2x,d9_forces2y,th9);
	// print_device_array_weave(d9_forces3x,d9_forces3y,th9);
	// cout << "force 9" << endl;	
	// arracychck(d9_forces1x,th9);
	// arracychck(d9_forces1y,th9);

	// print_device_float2(d_particles,totalN);
	// print_device_array(d9_list1,th9);
	// print_device_array(d9_list2,th9);
	// print_device_array(d9_list3,th9);

	// cout << "force9" << endl;

	//	matrix<double> F = ftemp1+ftemp2+ftemp3+F1+F2+F3+F4+F5+F6+F7;//+F4+F5;

		// matrix<double> R(totalN,dimension);
		// for(int i1 = 0 ; i1 < totalN ; i1++) {
		// 	for(int j = 0 ; j < dimension ; j++) {
		// 		R(i1,j) = (3.464101615 * ((double) rand() / (RAND_MAX)) - 1.732050808);
		// 	}
		// }

	int *d10_list1;
	double *d10_forces1x;
	double *d10_forces1y;	
	int th10;

	PositionForcesDueToAnglesGPU(d_particles, d_bound, d_boundalong, d10_list1, d10_forces1x, d10_forces1y,th10);
	

	// arracychck(d10_forces1x,th10);
	// arracychck(d10_forces1y,th10);
	// cout << "force10" << endl;
	// cout << "all forces calculated" << endl;





	resetforce(d_totalforcex);

	resetforce(d_totalforcey);

	//cout << "reset" << endl;

	// print_device_array(d_totalforcex,totalN);


	ReduceForces(d1_list1,d1_list2,d1_list3,d1_list4,d1_forces1x,d1_forces2x,d1_forces1y,d1_forces2y,d_totalforcex,d_totalforcey,th1);
	// cout << "d1" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	
	ReduceForces(d2_list1,d2_list2,d2_list3,d2_list4,d2_forces1x,d2_forces2x,d2_forces1y,d2_forces2y,d_totalforcex,d_totalforcey,th2);
	/// cout << "d2" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	ReduceForces(d3_list1,d3_list2,d3_list3,d3_list4,d3_forces1x,d3_forces2x,d3_forces1y,d3_forces2y,d_totalforcex,d_totalforcey,th3);
	// cout << "d3" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	
	ReduceForces(d4_list1,d4_list2,d4_list3,d4_list4,d4_forces1x,d4_forces2x,d4_forces1y,d4_forces2y,d_totalforcex,d_totalforcey,th4);
	// cout << "d4" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	
	ReduceForces(d5_list1,d5_list2,d5_list3,d5_list4,d5_forces1x,d5_forces2x,d5_forces1y,d5_forces2y,d_totalforcex,d_totalforcey,th5);
	// cout << "d5" << endl;	
	// print_device_weave_float2(d5_list1,d5_list2,d5_forces1x,d5_forces1y,d_particles,th5,totalN);
	// pausel();	
	ReduceForces(d6_list1,d6_list2,d6_list3,d6_list4,d6_forces1x,d6_forces2x,d6_forces1y,d6_forces2y,d_totalforcex,d_totalforcey,th6);	
	// cout << "d6" << endl;
	// print_device_weave_float2(d6_list1,d6_list2,d6_forces1x,d6_forces1y,d_particles,th6,totalN);
	// cout << endl;
	// pausel();
	ReduceForces3(d7_list1,d7_list2,d7_list3,d7_forces1x,d7_forces2x,d7_forces3x,d7_forces1y,d7_forces2y,d7_forces3y,d_totalforcex,d_totalforcey,th7);	
	// cout << "d7" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	ReduceForces(d8_list1,d8_list2,d8_forces1x,d8_forces2x,d8_forces1y,d8_forces2y,d_totalforcex,d_totalforcey,th8);
	// cout << "d8" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	ReduceForces3(d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces2x,d9_forces3x,d9_forces1y,d9_forces2y,d9_forces3y,d_totalforcex,d_totalforcey,th9);		
	
	// cout << "d9" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double ff = (v0_a+v0_b)/2.;
	ReduceForcesAndNormalize(d10_list1,d10_forces1x,d10_forces1y,d_totalforcex,d_totalforcey, max_s, ff, th10);

	cout << "reduction" << endl;
	// cout << "d10" << endl;
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	if(i>0&&i%every==0) { 
		// for(int j = 0 ; j < na+nb ; j++) {
		// if(bound[j]>0){
		// cout << "printed" << endl;
		// cout << j << endl;
		// cout << bound[j] << endl;
		// cout << bound_along[j] << endl;
		// cout << F5(j,'r') << endl;
		// cout << F4(j,'r') << endl;
		// cout << obj->getcoordinate(j,0) << " " << obj->getcoordinate(j,1) << endl;
		// cout << obj->getcoordinate(200,0) <<  " " << obj->getcoordinate(200,1) << endl;
		// cout << obj->getcoordinate(200+L,0) <<  " " << obj->getcoordinate(200+L,1) << endl;
		// }
		// }
		// cout << F6 << endl;
		// cout << F7 << endl;

		// cout << ftemp2 << endl;
		// cout << ftemp3 << endl;

		stringstream ss2;
		// ss2 <<i/every;
		// string pairlist = "list";

		 stringstream kts;
		 kts << (*obj).getkT();

		 // stringstream epi;
		 // epi << eps;

		 // stringstream epieq;
		 // epieq << eqeps;

		 stringstream len;
		 len << l;

		 string extension =  "_kT="+kts.str()+"_l="+len.str()+".csv";

		stringstream ss;
		ss <<(i/every);
		string filename = "x";
		filename += ss.str();
		filename += extension;

		string momname = "bind";
		momname += ss.str();
		momname += extension;

		string baname = "bind_along";
		baname += ss.str();
		baname += extension;



		ofstream myfile;
		myfile.open(filename.c_str());
		//myfile <<= (*obj).getdat();
		file_print_device_float2(d_particles,totalN,myfile);
		myfile.close();



		}	

	double *d_R1;
	double *d_R2;

	// hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2y,th8*sizeof(double));	
	hipMalloc((void**)&d_R1,totalN*sizeof(double));
	hipMalloc((void**)&d_R2,totalN*sizeof(double));

	setstaterandom(d_R1,1.732050808,totalN);
 	setstaterandom(d_R2,1.732050808,totalN);	

 	//advmom2D(d_momenta, d_totalforcex, d_totalforcey, d_R1, d_R2, cons1,cons2,cons3,totalN);
// 	advmom2_spatialdependence(d_momenta,d_particles,d_totalforcex,d_totalforcey,d_R1,d_R2,func, d_dt, d_kT, d_m, totalN);
	advmom2D(d_momenta,d_totalforcex,d_totalforcey,d_R1,d_R2,cons1,cons2,cons3, totalN);
 	advpos2D(d_particles, d_momenta, cons4, totalN);


 	applypbc2D(d_particles,d_momenta,l,is_periodic,totalN);

 	cout << "updated" << endl;

	hipFree(d1_forces1x);
	hipFree(d1_forces2x);
	hipFree(d1_forces1y);
	hipFree(d1_forces2y);
	hipFree(d2_forces1x);
	hipFree(d2_forces2x);
	hipFree(d2_forces1y);
	hipFree(d2_forces2y);
	hipFree(d3_forces1x);
	hipFree(d3_forces2x);
	hipFree(d3_forces1y);
	hipFree(d3_forces2y);
	hipFree(d4_forces1x);
	hipFree(d4_forces2x);
	hipFree(d4_forces1y);
	hipFree(d4_forces2y);
	hipFree(d5_forces1x);
	hipFree(d5_forces2x);
	hipFree(d5_forces1y);
	hipFree(d5_forces2y);
	hipFree(d6_forces1x);
	hipFree(d6_forces2x);
	hipFree(d6_forces1y);
	hipFree(d6_forces2y);
 	hipFree(d7_list1);
 	hipFree(d7_list2);
 	hipFree(d7_list3);
	hipFree(d7_forces1x);
	hipFree(d7_forces2x);
	hipFree(d7_forces3x);
	hipFree(d7_forces1y);
	hipFree(d7_forces2y);
	hipFree(d7_forces3y);
	hipFree(d8_forces1x);
	hipFree(d8_forces2x);
	hipFree(d8_forces1y);
	hipFree(d8_forces2y);
	hipFree(d9_forces1x);
	hipFree(d9_forces2x);
	hipFree(d9_forces3x);
	hipFree(d9_forces1y);
	hipFree(d9_forces2y);
	hipFree(d9_forces3y);
	hipFree(d10_list1);
	hipFree(d10_forces1x);
	hipFree(d10_forces1y);

	cout << "freed" << endl;


	}
}


/*
template <typename Fun>
void Microtubule::runGPUcheck(int runtime, int every, Fun func)
{
//	pausel();
	int ccc;
	int totalN = obj->getN();

	//num is the number of boxes per length

	int ncells = num*num;

	WCApotentialGPU faa_gpu(2.,1.,2.);
	WCApotentialGPU fab_gpu(1.,1.,0.);
	WCApotentialGPU fac_gpu(1.,1.,0.);
	WCApotentialGPU fbb_gpu(2.,1.,2.);
	WCApotentialGPU fbc_gpu(1.,1.,0.);
	WCApotentialGPU fcc_gpu(1.,1.,0.);
	HarmonicPotentialGPU bindp_gpu(100.,0.); 
	FENEPotentialGPU bindm_gpu(50.,1.5); 
	BendingPotentialGPU bendp_gpu(100.,0.);

//we now have the count of each cell list

	matrix<int> boxes = (obj)->getgeo().generate_boxes_relationships(num,ccc);


	matrix<int> *froyo1 = obj->calculatepairs(boxes,pai,3.5);
	matrix<int> *froyo2 = obj->calculatepairs(boxes,pbi,3.5);
	matrix<int> *froyo3 = obj->calculatepairs(boxes,pci,3.5);
	matrix<int> *froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
	matrix<int> *froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
	matrix<int> *froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);


	int nbpairs = 5*ncells;
	int nperl = num;

	int *cells1 = new int [nbpairs];
	int *cells2 = new int [nbpairs];


	int itery = 0;
	for(int i1 = 0 ; i1 < num ; i1++) {
		for(int i2 = 0 ; i2 < num ; i2++ ) {


			int b1 =  i1*nperl+i2;

			int i3 = i1+0;
			int j3 = i2+0;

			int i4 = i1+1;
			int j4 = i2+0;

			int i5 = i1-1;
			int j5 = i2+1;

			int i6 = i1+0;
			int j6 = i2+1;

			int i7 = i1+1;
			int j7 = i2+1;

			prdshft(i3,nperl);
			prdshft(j3,nperl);

			prdshft(i4,nperl);
			prdshft(j4,nperl);

			prdshft(i5,nperl);
			prdshft(j5,nperl);
			
			prdshft(i6,nperl);
			prdshft(j6,nperl);
			
			prdshft(i7,nperl);
			prdshft(j7,nperl);		

			cells1[itery] =  b1;
			cells2[itery] =  i3*nperl+j3;

			itery++;

			cells1[itery] =  b1;
			cells2[itery] =  i4*nperl+j4;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i5*nperl+j5;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i6*nperl+j6;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i7*nperl+j7;

			itery++;


		}
	}
	int size4 = nbpairs*sizeof(int);

	int *d_cells1;
	int *d_cells2;

	hipMalloc((void**)&d_cells1,size4);

	hipMalloc((void**)&d_cells2,size4);

	hipMemcpy(d_cells1,cells1,size4,hipMemcpyHostToDevice);
	hipMemcpy(d_cells2,cells2,size4,hipMemcpyHostToDevice);
	//int sibdiv = floor(ll/4.0);
	// print_device_array(d_cells1,nbpairs);
	// print_device_array(d_cells2,nbpairs);



	// matrix<int> boxes = (obj)->getgeo().generate_boxes_relationships(num,ccc);
	
	float2 *particles = new float2 [totalN];
	float2 *momenta = new float2 [totalN];
	int *p_indices = new int [totalN];

	for(int i = 0 ; i < totalN ; i++)
	p_indices[i]=i;

	float2 *d_particles;
	float2 *d_momenta;
	int *d_p_indices;

	int *d_bound;
	double *d_boundalong;
	int *d_changestate;

	hipMalloc((void**)&d_bound,(na+nb)*sizeof(int));
	hipMalloc((void**)&d_boundalong,(na+nb)*sizeof(double));
	hipMalloc((void**)&d_changestate,(na+nb)*sizeof(int));

	hipMemset(d_bound,0,(na+nb)*sizeof(int));
	hipMemset(d_boundalong,0.,(na+nb)*sizeof(double));
	hipMemset(d_changestate,0,(na+nb)*sizeof(int));

	double *d_totalforcex;
	double *d_totalforcey;

	hipMalloc((void**)&d_totalforcex,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey,totalN*sizeof(double));


	matrix<double> state(obj->getdat());


	for(int i = 0  ; i < totalN ; i++) {

	float2 c;
	c.x=state(i,0);
	c.y=state(i,1);

	(particles)[i]=c;

	float2 d;

	d.x = 0.;
	d.y = 0.;

	(momenta)[i]=d;
	}


	int size =  totalN*sizeof(float2);
	int size2 = totalN*sizeof(int);


	hipMalloc((void**)&d_particles,size);
	hipMalloc((void**)&d_momenta,size);
	hipMalloc((void**)&d_p_indices,size2);

	hipMemcpy(d_particles,particles,size,hipMemcpyHostToDevice);
	hipMemcpy(d_momenta,momenta,size,hipMemcpyHostToDevice);
	hipMemcpy(d_p_indices,p_indices,size2,hipMemcpyHostToDevice);


	// matrix<int> *froyo1 = obj->calculatepairs(boxes,pai,3.5);
	// matrix<int> *froyo2 = obj->calculatepairs(boxes,pbi,3.5);
	// matrix<int> *froyo3 = obj->calculatepairs(boxes,pci,3.5);
	// matrix<int> *froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
	// matrix<int> *froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
	// matrix<int> *froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);
	
	int *d_indices1;
	int *d_indices2;
	double *d_close;


	int tpp;
	


	construct_possible_pair_list(d_particles,d_p_indices,totalN,l,d_cells1,d_cells2,num,is_periodic,d_indices1,d_indices2,d_close,tpp);



	less_than_condition_AND cond1(SQR(3.5),0,na);
	less_than_condition_AND cond2(SQR(3.5),na,na+nb);
	less_than_condition_AND cond3(SQR(3.5),na+nb,na+nb+nc);
	less_than_condition_NAND cond4(SQR(3.5),0,na,na,na+nb);
	less_than_condition_NAND cond5(SQR(3.5),0,na,na+nb,na+nb+nc);
	less_than_condition_NAND cond6(SQR(3.5),na,na+nb,na+nb,na+nb+nc);


	int th1;
	int *d1_list1,*d1_list2,*d1_list3,*d1_list4;
	pairlist(d_indices1,d_indices2,d_close,cond1,d1_list1,d1_list2,d1_list3,d1_list4,tpp, th1); //faa

	int th2;
	int *d2_list1,*d2_list2,*d2_list3,*d2_list4;
	pairlist(d_indices1,d_indices2,d_close,cond2,d2_list1,d2_list2,d2_list3,d2_list4,tpp, th2);	//fbb

	int th3;
	int *d3_list1,*d3_list2,*d3_list3,*d3_list4;
	pairlist(d_indices1,d_indices2,d_close,cond3,d3_list1,d3_list2,d3_list3,d3_list4,tpp, th3);	//fcc

	int th4;
	int *d4_list1,*d4_list2,*d4_list3,*d4_list4;
	pairlist(d_indices1,d_indices2,d_close,cond4,d4_list1,d4_list2,d4_list3,d4_list4,tpp, th4);	//fab

	int th5;
	int *d5_list1,*d5_list2,*d5_list3,*d5_list4;
	pairlist(d_indices1,d_indices2,d_close,cond5,d5_list1,d5_list2,d5_list3,d5_list4,tpp, th5);	//fac

	int th6;
	int *d6_list1,*d6_list2,*d6_list3,*d6_list4;
	pairlist(d_indices1,d_indices2,d_close,cond6,d6_list1,d6_list2,d6_list3,d6_list4,tpp, th6); //fbc			
	//matrix<double> state(obj->getdat()); //the state of the system

	int th8 = (*bondpairs).getNsafe();
	int th9 = (*bendtriplets).getNsafe();


	int *d8_list1,*d8_list2,*d8_list3,*d8_list4;
	hipMalloc((void**)&d8_list1,th8*sizeof(int));
	hipMalloc((void**)&d8_list2,th8*sizeof(int));
	hipMalloc((void**)&d8_list3,th8*sizeof(int));
	hipMalloc((void**)&d8_list4,th8*sizeof(int));



	int *h8_list1 = new int [th8];
	int *h8_list2 = new int [th8];

	for(int i = 0 ; i < th8 ; i++ ) {
		h8_list1[i] = (*bondpairs)(i,0);
		h8_list2[i] = (*bondpairs)(i,1);
	}

	hipMemcpy(d8_list1,h8_list1,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list2,h8_list2,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list3,h8_list1,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list4,h8_list2,th8*sizeof(int),hipMemcpyHostToDevice);





	int *d9_list1,*d9_list2,*d9_list3,*d9_list4,*d9_list5,*d9_list6;
	hipMalloc((void**)&d9_list1,th9*sizeof(int));
	hipMalloc((void**)&d9_list2,th9*sizeof(int));
	hipMalloc((void**)&d9_list3,th9*sizeof(int));
	hipMalloc((void**)&d9_list4,th9*sizeof(int));
	hipMalloc((void**)&d9_list5,th9*sizeof(int));
	hipMalloc((void**)&d9_list6,th9*sizeof(int));

	int *h9_list1 = new int [th9];
	int *h9_list2 = new int [th9];
	int *h9_list3 = new int [th9];

	for(int i = 0 ; i < th9 ; i++ ) {
		h9_list1[i] = (*bendtriplets)(i,0);
		h9_list2[i] = (*bendtriplets)(i,1);
		h9_list3[i] = (*bendtriplets)(i,2);
	}	

	hipMemcpy(d9_list1,h9_list1,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list2,h9_list2,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list3,h9_list3,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list4,h9_list1,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list5,h9_list2,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list6,h9_list3,th9*sizeof(int),hipMemcpyHostToDevice);


	int i;

	double cons1;
	double cons2;
	double cons3;
	double cons4;

	//(mom)->operator()(i,i1) = c5*c2*((mom)->operator()(i,i1)) + (c5*(c3)+q)*F(i,i1) + (c5*(c4)+r)*R(i,i1);
	cons1 = (*obj).getc5()*(*obj).getc2();
	cons2 = (*obj).getc5()*(*obj).getc3()+(*obj).getq();
	cons3 = (*obj).getc5()*(*obj).getc4()+(*obj).getr();
	cons4 = (*obj).getc1();


	// vector<matrix<double> > savef1forces;
	// vector<matrix<double> > savef2forces;
	// vector<matrix<double> > savef3forces;
	// vector<matrix<double> > savef4forces;
	// vector<matrix<double> > savef5forces;
	// vector<matrix<double> > savef6forces;
	// vector<matrix<double> > savef7forces;

	// vector<matrix<double> > saveftemp1forces;
	// vector<matrix<double> > saveftemp2forces;
	// vector<matrix<double> > saveftemp3forces;

	// vector<matrix<double> > savepositions;
	// vector<vector1<int> > savebound;
	// vector<vector1<double> > saveboundalongs;

	for(i = 0 ; i < runtime ; i++) {
		//cout << i << endl;

		cout << i << endl;
		//pausel();
	
		//cout << (*obj).avmom() << endl;
	if(i%25==0) {
		//delete froyo1,froyo2,froyo3,froyo4,froyo5,froyo6;
		// cout << "updated after: " << i << endl;
		// state = obj->getdat();

		delete froyo1,froyo2,froyo3,froyo4,froyo5,froyo6;
		// cout << "updated after: " << i << endl;
		// state = obj->getdat();
		froyo1 = obj->calculatepairs(boxes,pai,3.5);
		froyo2 = obj->calculatepairs(boxes,pbi,3.5);
		froyo3 = obj->calculatepairs(boxes,pci,3.5);
		froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
		froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
		froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);


		hipFree(d1_list1);hipFree(d1_list2);hipFree(d1_list3);hipFree(d1_list4);
		

		hipFree(d2_list1);hipFree(d2_list2);hipFree(d2_list3);hipFree(d2_list4);
		

		hipFree(d3_list1);hipFree(d3_list2);hipFree(d3_list3);hipFree(d3_list4);
		

		hipFree(d4_list1);hipFree(d4_list2);hipFree(d4_list3);hipFree(d4_list4);
		

		hipFree(d5_list1);hipFree(d5_list2);hipFree(d5_list3);hipFree(d5_list4);
	
		hipFree(d6_list1);hipFree(d6_list2);hipFree(d6_list3);hipFree(d6_list4);

		hipFree(d_indices1);

		hipFree(d_indices2);

		hipFree(d_close);



		this->resetindices(d_p_indices,totalN);



		construct_possible_pair_list(d_particles,d_p_indices,totalN,l,d_cells1,d_cells2,num,is_periodic,d_indices1,d_indices2,d_close,tpp,false);


		cout << "pair" << endl;

		pairlist(d_indices1,d_indices2,d_close,cond1,d1_list1,d1_list2,d1_list3,d1_list4,tpp, th1); //faa
		pairlist(d_indices1,d_indices2,d_close,cond2,d2_list1,d2_list2,d2_list3,d2_list4,tpp, th2); //fbb
		pairlist(d_indices1,d_indices2,d_close,cond3,d3_list1,d3_list2,d3_list3,d3_list4,tpp, th3); //fcc
		pairlist(d_indices1,d_indices2,d_close,cond4,d4_list1,d4_list2,d4_list3,d4_list4,tpp, th4); //fab
		pairlist(d_indices1,d_indices2,d_close,cond5,d5_list1,d5_list2,d5_list3,d5_list4,tpp, th5); //fac
		pairlist(d_indices1,d_indices2,d_close,cond6,d6_list1,d6_list2,d6_list3,d6_list4,tpp, th6); //fbc


		
		// froyo1 = obj->calculatepairs(boxes,pai,3.5);
		// froyo2 = obj->calculatepairs(boxes,pbi,3.5);
		// froyo3 = obj->calculatepairs(boxes,pci,3.5);
		// froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
		// froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
		// froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);

	}





	//cout << "pairs" << endl;
	// cout << "pairings" << endl;


	// matrix<double> ftemp2(totalN,dimension),ftemp3(totalN,dimension);
	// //matrix<double> angforces1(nc,dimension-1),angforces2(nc,dimension-1),angforces3(nc,dimension-1);
	// // cout << "matrices initialized" << endl;

	// matrix<double> F1((*obj).calculateforces(*froyo1,*faa)); //calculate the forces using the pairs as an input

	// matrix<double> F2((*obj).calculateforces(*froyo2,*fbb)); //calculate the forces using the pairs as an input

	// matrix<double> ftemp1((*obj).calculateforces(*froyo3,*fcc)); //calculate the forces using the pairs as an input
	
	// matrix<double> F3((*obj).calculateforces(*froyo4,*fab)); //calculate the forces using the pairs as an input

	// this->ForcesDueToPositionPL(*froyo5,ftemp2); //calculate the forces using the pairs as an input

	// this->ForcesDueToPositionPL(*froyo6,ftemp3); //calculate the forces using the pairs as an input

	// this->CalculateBindings(*froyo5,*froyo6);

	// matrix<double> F4 = this->BindingForces();

	// matrix<double> F5 = this->PositionForcesDueToAngles();

	//print_device_float2(d_particles,totalN);

	double *d1_forces1x;
	double *d1_forces2x;
	double *d1_forces1y;
	double *d1_forces2y;
	hipMalloc((void**)&d1_forces1x,th1*sizeof(double));
	hipMalloc((void**)&d1_forces1y,th1*sizeof(double));
	hipMalloc((void**)&d1_forces2x,th1*sizeof(double));
	hipMalloc((void**)&d1_forces2y,th1*sizeof(double));
	calculateforces2D(d1_list1,d1_list2,d_particles, d1_forces1x,d1_forces1y,d1_forces2x,d1_forces2y, faa_gpu ,th1, l,true);

	cout << "force1" << endl;
	arracychck(d1_forces1x,th1);
	arracychck(d1_forces1y,th1); 

	double *d2_forces1x;
	double *d2_forces2x;
	double *d2_forces1y;
	double *d2_forces2y;
	hipMalloc((void**)&d2_forces1x,th2*sizeof(double));
	hipMalloc((void**)&d2_forces1y,th2*sizeof(double));
	hipMalloc((void**)&d2_forces2x,th2*sizeof(double));
	hipMalloc((void**)&d2_forces2y,th2*sizeof(double));
	calculateforces2D(d2_list1,d2_list2,d_particles, d2_forces1x,d2_forces1y,d2_forces2x,d2_forces2y, fbb_gpu ,th2, l,true);	

	cout << "force2" << endl;
	arracychck(d1_forces2x,th2);
	arracychck(d1_forces2y,th2); 

	double *d3_forces1x;
	double *d3_forces2x;
	double *d3_forces1y;
	double *d3_forces2y;
	hipMalloc((void**)&d3_forces1x,th3*sizeof(double));
	hipMalloc((void**)&d3_forces1y,th3*sizeof(double));
	hipMalloc((void**)&d3_forces2x,th3*sizeof(double));
	hipMalloc((void**)&d3_forces2y,th3*sizeof(double));
	calculateforces2D(d3_list1,d3_list2,d_particles, d3_forces1x,d3_forces1y,d3_forces2x,d3_forces2y, fcc_gpu ,th3, l,true);	


	cout << "force3" << endl;
	arracychck(d3_forces1x,th3);
	arracychck(d3_forces1y,th3); 

	double *d4_forces1x;
	double *d4_forces2x;
	double *d4_forces1y;
	double *d4_forces2y;
	hipMalloc((void**)&d4_forces1x,th4*sizeof(double));
	hipMalloc((void**)&d4_forces1y,th4*sizeof(double));
	hipMalloc((void**)&d4_forces2x,th4*sizeof(double));
	hipMalloc((void**)&d4_forces2y,th4*sizeof(double));
	calculateforces2D(d4_list1,d4_list2,d_particles, d4_forces1x,d4_forces1y,d4_forces2x,d4_forces2y, fab_gpu ,th4, l,true);

	cout << "force4" << endl;
	arracychck(d4_forces1x,th4);
	arracychck(d4_forces1y,th4); 

	double *d5_forces1x;
	double *d5_forces2x;
	double *d5_forces1y;
	double *d5_forces2y;
	hipMalloc((void**)&d5_forces1x,th5*sizeof(double));
	hipMalloc((void**)&d5_forces1y,th5*sizeof(double));
	hipMalloc((void**)&d5_forces2x,th5*sizeof(double));
	hipMalloc((void**)&d5_forces2y,th5*sizeof(double));
	calculateforces2D(d5_list1,d5_list2,d_particles, d5_forces1x,d5_forces1y,d5_forces2x,d5_forces2y, fbc_gpu ,th5, l,true);

	cout << "force5" << endl;
	arracychck(d5_forces1x,th5);
	arracychck(d5_forces1y,th5); 	
	// cout << "d5" << endl;
	// print_device_weave_float2(d5_list1,d5_list2,d5_forces1x,d5_forces1y,d5_forces2x,d5_forces2y,d_particles,th5,totalN);
	// pausel();	

	double *d6_forces1x;
	double *d6_forces2x;
	double *d6_forces1y;
	double *d6_forces2y;
	hipMalloc((void**)&d6_forces1x,th6*sizeof(double));
	hipMalloc((void**)&d6_forces1y,th6*sizeof(double));
	hipMalloc((void**)&d6_forces2x,th6*sizeof(double));
	hipMalloc((void**)&d6_forces2y,th6*sizeof(double));
	calculateforces2D(d6_list1,d6_list2,d_particles, d6_forces1x,d6_forces1y,d6_forces2x,d6_forces2y, fac_gpu ,th6, l,true);	

	cout << "force6" << endl;
	arracychck(d6_forces1x,th6);
	arracychck(d6_forces1y,th6); 	
	// cout << "d6" << endl;
	// print_device_weave_float2(d6_list1,d6_list2,d6_forces1x,d6_forces1y,d6_forces2x,d6_forces2y,d_particles,th6,totalN);
	// cout << endl;
	// pausel();

	// matrix<double> F6((*obj).calculateforces(*bondpairs,*bindm));
	callCalculateUnbindingsGPU(d_particles,d_bound,d_boundalong,d_changestate);



	cout << "unbindings calculated" << endl;




	callCalculateBindingsGPU(d5_list1,d5_list2,d6_list1,d6_list2,d_particles, d_bound, d_boundalong,d_changestate,th5 ,th6 );


	cout << "bindings calculated" << endl;



	int *d7_list1,*d7_list2,*d7_list3;
	double *d7_forces1x;
	double *d7_forces1y;
	double *d7_forces2x;
	double *d7_forces2y;
	double *d7_forces3x;
	double *d7_forces3y;
	int th7;
	BindingForcesGPU(d_particles, d_bound, d_boundalong, d7_list1,d7_list2,d7_list3, d7_forces1x, d7_forces1y, d7_forces2x,d7_forces2y,d7_forces3x, d7_forces3y, bindp_gpu, th7);


	cout << "binding forces calculated" << endl;
	arracychck(d7_forces1x,th7);
	arracychck(d7_forces1y,th7); 

	double *d8_forces1x;
	double *d8_forces2x;
	double *d8_forces1y;
	double *d8_forces2y;
	hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	hipMalloc((void**)&d8_forces2y,th8*sizeof(double));
	calculateforces2D(d8_list1,d8_list2,d_particles, d8_forces1x,d8_forces1y,d8_forces2x,d8_forces2y, bindm_gpu ,th8, l,true);

	cout << "forces8" << endl;
	arracychck(d8_forces1x,th8);
	arracychck(d8_forces1y,th8); 

	resetchangestate(d_changestate);
	// matrix<double> F7((*obj).calculateforces_threebody(*bendtriplets,*bendp));

//	int th9;


	double *d9_forces1x;
	double *d9_forces2x;
	double *d9_forces1y;
	double *d9_forces2y;
	double *d9_forces3x;
	double *d9_forces3y;
	hipMalloc((void**)&d9_forces1x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces1y,th9*sizeof(double));
	hipMalloc((void**)&d9_forces2x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces2y,th9*sizeof(double));
	hipMalloc((void**)&d9_forces3x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces3y,th9*sizeof(double));
	BendingForcesGPU(d_particles, d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces1y,d9_forces2x,d9_forces2y, d9_forces3x, d9_forces3y,bendp_gpu,th9);

	cout << "bending forces" << endl;
	arracychck(d9_forces1x,th9);
	arracychck(d9_forces1y,th9); 
	//	matrix<double> F = ftemp1+ftemp2+ftemp3+F1+F2+F3+F4+F5+F6+F7;//+F4+F5;

		// matrix<double> R(totalN,dimension);
		// for(int i1 = 0 ; i1 < totalN ; i1++) {
		// 	for(int j = 0 ; j < dimension ; j++) {
		// 		R(i1,j) = (3.464101615 * ((double) rand() / (RAND_MAX)) - 1.732050808);
		// 	}
		// }

	int *d10_list1;
	double *d10_forces1x;
	double *d10_forces1y;	
	int th10;

	PositionForcesDueToAnglesGPU(d_particles, d_bound, d_boundalong, d10_list1, d10_forces1x, d10_forces1y,th10);
	

	cout << "pos forces" << endl;
	arracychck(d10_forces1x,th9);
	arracychck(d10_forces1y,th9); 

	cout << "all forces calculated" << endl;




	double *d_totalforcex1;
	double *d_totalforcey1;

	hipMalloc((void**)&d_totalforcex1,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey1,totalN*sizeof(double));

	resetforce(d_totalforcex1);
	resetforce(d_totalforcey1);

	cout << "reset" << endl;

	// print_device_array(d_totalforcex,totalN);


	//ReduceForces(d1_list1,d1_list2,d1_list3,d1_list4,d1_forces1x,d1_forces2x,d1_forces1y,d1_forces2y,d_totalforcex,d_totalforcey,th1);
	ReduceForces(d1_list1,d1_list2,d1_list3,d1_list4,d1_forces1x,d1_forces2x,d1_forces1y,d1_forces2y,d_totalforcex1,d_totalforcey1,th1);	
	 cout << "d1" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	

	double *d_totalforcex2;
	double *d_totalforcey2;

	hipMalloc((void**)&d_totalforcex2,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey2,totalN*sizeof(double));

	resetforce(d_totalforcex2);
	resetforce(d_totalforcey2);	 
	//ReduceForces(d2_list1,d2_list2,d2_list3,d2_list4,d2_forces1x,d2_forces2x,d2_forces1y,d2_forces2y,d_totalforcex,d_totalforcey,th2);
	ReduceForces(d2_list1,d2_list2,d2_list3,d2_list4,d2_forces1x,d2_forces2x,d2_forces1y,d2_forces2y,d_totalforcex2,d_totalforcey2,th2);
	cout << "d2" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double *d_totalforcex3;
	double *d_totalforcey3;

	hipMalloc((void**)&d_totalforcex3,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey3,totalN*sizeof(double));

	resetforce(d_totalforcex3);
	resetforce(d_totalforcey3);	 	
	//ReduceForces(d3_list1,d3_list2,d3_list3,d3_list4,d3_forces1x,d3_forces2x,d3_forces1y,d3_forces2y,d_totalforcex,d_totalforcey,th3);	 	
	ReduceForces(d3_list1,d3_list2,d3_list3,d3_list4,d3_forces1x,d3_forces2x,d3_forces1y,d3_forces2y,d_totalforcex3,d_totalforcey3,th3);
	 cout << "d3" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double *d_totalforcex4;
	double *d_totalforcey4;

	hipMalloc((void**)&d_totalforcex4,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey4,totalN*sizeof(double));

	resetforce(d_totalforcex4);
	resetforce(d_totalforcey4);	 		
	//ReduceForces(d4_list1,d4_list2,d4_list3,d4_list4,d4_forces1x,d4_forces2x,d4_forces1y,d4_forces2y,d_totalforcex,d_totalforcey,th4);
	ReduceForces(d4_list1,d4_list2,d4_list3,d4_list4,d4_forces1x,d4_forces2x,d4_forces1y,d4_forces2y,d_totalforcex4,d_totalforcey4,th4);
	 cout << "d4" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double *d_totalforcex5;
	double *d_totalforcey5;

	hipMalloc((void**)&d_totalforcex5,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey5,totalN*sizeof(double));

	resetforce(d_totalforcex5);
	resetforce(d_totalforcey5);	 	 
	//ReduceForces(d5_list1,d5_list2,d5_list3,d5_list4,d5_forces1x,d5_forces2x,d5_forces1y,d5_forces2y,d_totalforcex,d_totalforcey,th5);
	ReduceForces(d5_list1,d5_list2,d5_list3,d5_list4,d5_forces1x,d5_forces2x,d5_forces1y,d5_forces2y,d_totalforcex5,d_totalforcey5,th5);	

	 cout << "d5" << endl;	
	// print_device_weave_float2(d5_list1,d5_list2,d5_forces1x,d5_forces1y,d_particles,th5,totalN);
	// pausel();
	double *d_totalforcex6;
	double *d_totalforcey6;

	hipMalloc((void**)&d_totalforcex6,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey6,totalN*sizeof(double));

	resetforce(d_totalforcex6);
	resetforce(d_totalforcey6);	 	
	//ReduceForces(d6_list1,d6_list2,d6_list3,d6_list4,d6_forces1x,d6_forces2x,d6_forces1y,d6_forces2y,d_totalforcex,d_totalforcey,th6);	 	
	ReduceForces(d6_list1,d6_list2,d6_list3,d6_list4,d6_forces1x,d6_forces2x,d6_forces1y,d6_forces2y,d_totalforcex6,d_totalforcey6,th6);		
	 cout << "d6" << endl;
	// print_device_weave_float2(d6_list1,d6_list2,d6_forces1x,d6_forces1y,d_particles,th6,totalN);
	// cout << endl;
	// pausel();
	double *d_totalforcex7;
	double *d_totalforcey7;

	hipMalloc((void**)&d_totalforcex7,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey7,totalN*sizeof(double));

	resetforce(d_totalforcex7);
	resetforce(d_totalforcey7);	 
	//ReduceForces3(d7_list1,d7_list2,d7_list3,d7_forces1x,d7_forces2x,d7_forces3x,d7_forces1y,d7_forces2y,d7_forces3y,d_totalforcex,d_totalforcey,th7);	 
	ReduceForces3(d7_list1,d7_list2,d7_list3,d7_forces1x,d7_forces2x,d7_forces3x,d7_forces1y,d7_forces2y,d7_forces3y,d_totalforcex7,d_totalforcey7,th7);	
	 cout << "d7" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double *d_totalforcex8;
	double *d_totalforcey8;

	hipMalloc((void**)&d_totalforcex8,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey8,totalN*sizeof(double));
	

	
	resetforce(d_totalforcex8);
	resetforce(d_totalforcey8);	 
	//ReduceForces(d8_list1,d8_list2,d8_forces1x,d8_forces2x,d8_forces1y,d8_forces2y,d_totalforcex,d_totalforcey,th8);	 
	ReduceForces(d8_list1,d8_list2,d8_forces1x,d8_forces2x,d8_forces1y,d8_forces2y,d_totalforcex8,d_totalforcey8,th8);
	 cout << "d8" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double *d_totalforcex9;
	double *d_totalforcey9;

	hipMalloc((void**)&d_totalforcex9,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey9,totalN*sizeof(double));

	resetforce(d_totalforcex9);
	resetforce(d_totalforcey9);	 
	//ReduceForces3(d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces2x,d9_forces3x,d9_forces1y,d9_forces2y,d9_forces3y,d_totalforcex,d_totalforcey,th9);		 
	ReduceForces3(d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces2x,d9_forces3x,d9_forces1y,d9_forces2y,d9_forces3y,d_totalforcex9,d_totalforcey9,th9);		
	
	double *d_totalforcex10;
	double *d_totalforcey10;

	hipMalloc((void**)&d_totalforcex10,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey10,totalN*sizeof(double));

	resetforce(d_totalforcex10);
	resetforce(d_totalforcey10);	 
	 cout << "d9" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double ff = (v0_a+v0_b)/2.;
	//ReduceForcesAndNormalize(d10_list1,d10_forces1x,d10_forces1y,d_totalforcex,d_totalforcey, max_s, ff, th10);
	ReduceForcesAndNormalize(d10_list1,d10_forces1x,d10_forces1y,d_totalforcex10,d_totalforcey10, max_s, ff, th10);

	cout << "reduction" << endl;
	// cout << "d10" << endl;
	// print_device_array(d_totalforcex,totalN);
	// pausel();

	matrix<double> ftemp2(totalN,dimension),ftemp3(totalN,dimension);
	//matrix<double> angforces1(nc,dimension-1),angforces2(nc,dimension-1),angforces3(nc,dimension-1);
	// cout << "matrices initialized" << endl;

	matrix<double> F1((*obj).calculateforces(*froyo1,*faa)); //calculate the forces using the pairs as an input

	matrix<double> F2((*obj).calculateforces(*froyo2,*fbb)); //calculate the forces using the pairs as an input

	matrix<double> ftemp1((*obj).calculateforces(*froyo3,*fcc)); //calculate the forces using the pairs as an input
	
	matrix<double> F3((*obj).calculateforces(*froyo4,*fab)); //calculate the forces using the pairs as an input

	this->ForcesDueToPositionPL(*froyo5,ftemp2); //calculate the forces using the pairs as an input

	this->ForcesDueToPositionPL(*froyo6,ftemp3); //calculate the forces using the pairs as an input

	this->CalculateBindings(*froyo5,*froyo6);

	matrix<double> F4 = this->BindingForces();

	matrix<double> F5 = this->PositionForcesDueToAngles();


//	cout << "active forces" << endl;
	//cout << "pos forces" << endl;
	 matrix<double> F6((*obj).calculateforces(*bondpairs,*bindm));

	//cout << "bond forces" << endl;
	matrix<double> F7((*obj).calculateforces_threebody(*bendtriplets,*bendp));

	//cout << "after check matrix" << endl;

	matrix<double> F = ftemp1+ftemp2+ftemp3+F1+F2+F3+F4+F5+F6+F7;

	cout << l << endl;
	for(int j1 = 0 ; j1 < na+nb ; j1++ ) {
		if(bound[j1]>0) cout << j1 << ",";
	}
	cout << endl;
	print_device_array_indices(d_bound,na+nb);
	pausel();




	cout << F1 << endl;
	print_device_array_weave(d_totalforcex1,d_totalforcey1,totalN);
	cout << th1 << endl;
	cout << "faa" << endl;
	pausel();



	cout << F2 << endl;
	print_device_array_weave(d_totalforcex2,d_totalforcey2,totalN);
	cout << "fbb" << endl;
	pausel();


	cout << ftemp1 << endl;
	print_device_array_weave(d_totalforcex3,d_totalforcey3,totalN);
	cout << "fcc" << endl;
	pausel();		

	cout << F3 << endl;
	print_device_array_weave(d_totalforcex4,d_totalforcey4,totalN);
	cout << "fab" << endl;
	pausel();

	cout << ftemp2 << endl;
	print_device_array_weave(d_totalforcex5,d_totalforcey5,totalN);
	cout << "fac" << endl;
	pausel();		

	cout << ftemp3 << endl;
	print_device_array_weave(d_totalforcex6,d_totalforcey6,totalN);
	cout << "fbc" << endl;
	pausel();

	cout << F4 << endl;
	print_device_array_weave(d_totalforcex7,d_totalforcey7,totalN);
	cout << "bound to mt" << endl;
	pausel();	

	cout << F5 << endl;
	print_device_array_weave(d_totalforcex10,d_totalforcey10,totalN);
	cout << "position force" << endl;
	pausel();

	cout << F6 << endl;
	print_device_array_weave(d_totalforcex8,d_totalforcey8,totalN);
	cout << "bound within mt" << endl;
	pausel();		


	cout << F7 << endl;
	print_device_array_weave(d_totalforcex9,d_totalforcey9,totalN);
	cout << "bending force" << endl;
	pausel();		







	double *d_R1;
	double *d_R2;

	// hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2y,th8*sizeof(double));	
	hipMalloc((void**)&d_R1,totalN*sizeof(double));
	hipMalloc((void**)&d_R2,totalN*sizeof(double));

	setstaterandom(d_R1,1.732050808,totalN);
 	setstaterandom(d_R2,1.732050808,totalN);

 	print_device_array(d_R1,totalN);
 	print_device_array(d_R2,totalN);

 	pausel();

 	advmom2D(d_momenta, d_totalforcex, d_totalforcey, d_R1, d_R2, cons1,cons2,cons3,totalN);
 	advpos2D(d_particles, d_momenta, cons4, totalN);

 	applypbc2D(d_particles,d_momenta,l,is_periodic,totalN);

 	cout << "updated" << endl;

	hipFree(d1_forces1x);
	hipFree(d1_forces2x);
	hipFree(d1_forces1y);
	hipFree(d1_forces2y);
	hipFree(d2_forces1x);
	hipFree(d2_forces2x);
	hipFree(d2_forces1y);
	hipFree(d2_forces2y);
	hipFree(d3_forces1x);
	hipFree(d3_forces2x);
	hipFree(d3_forces1y);
	hipFree(d3_forces2y);
	hipFree(d4_forces1x);
	hipFree(d4_forces2x);
	hipFree(d4_forces1y);
	hipFree(d4_forces2y);
	hipFree(d5_forces1x);
	hipFree(d5_forces2x);
	hipFree(d5_forces1y);
	hipFree(d5_forces2y);
	hipFree(d6_forces1x);
	hipFree(d6_forces2x);
	hipFree(d6_forces1y);
	hipFree(d6_forces2y);
 	hipFree(d7_list1);
 	hipFree(d7_list2);
 	hipFree(d7_list3);
	hipFree(d7_forces1x);
	hipFree(d7_forces2x);
	hipFree(d7_forces3x);
	hipFree(d7_forces1y);
	hipFree(d7_forces2y);
	hipFree(d7_forces3y);
	hipFree(d8_forces1x);
	hipFree(d8_forces2x);
	hipFree(d8_forces1y);
	hipFree(d8_forces2y);
	hipFree(d9_forces1x);
	hipFree(d9_forces2x);
	hipFree(d9_forces3x);
	hipFree(d9_forces1y);
	hipFree(d9_forces2y);
	hipFree(d9_forces3y);
	hipFree(d10_list1);
	hipFree(d10_forces1x);
	hipFree(d10_forces1y);

	cout << "freed" << endl;


	}
}
*/

template <typename Fun>
void Microtubule::runGPUSV(int runtime, int every, Fun func)
{
//	pausel();
	//int ccc;
	int totalN = obj->getN();

	//num is the number of boxes per length

	int ncells = num*num;

	WCApotentialGPU faa_gpu(2.,1.,2.);
	WCApotentialGPU fab_gpu(1.,1.,0.);
	WCApotentialGPU fac_gpu(1.,1.,0.);
	WCApotentialGPU fbb_gpu(2.,1.,2.);
	WCApotentialGPU fbc_gpu(1.,1.,0.);
	WCApotentialGPU fcc_gpu(1.,1.,0.);
	HarmonicPotentialGPU bindp_gpu(100.,1.); 
	FENEPotentialGPU bindm_gpu(50.,1.5); 
	BendingPotentialGPU bendp_gpu(100.,0.);

//we now have the count of each cell list
	int nbpairs = 5*ncells;
	int nperl = num;

	int *cells1 = new int [nbpairs];
	int *cells2 = new int [nbpairs];


	int itery = 0;
	for(int i1 = 0 ; i1 < num ; i1++) {
		for(int i2 = 0 ; i2 < num ; i2++ ) {


			int b1 =  i1*nperl+i2;

			int i3 = i1+0;
			int j3 = i2+0;

			int i4 = i1+1;
			int j4 = i2+0;

			int i5 = i1-1;
			int j5 = i2+1;

			int i6 = i1+0;
			int j6 = i2+1;

			int i7 = i1+1;
			int j7 = i2+1;

			prdshft(i3,nperl);
			prdshft(j3,nperl);

			prdshft(i4,nperl);
			prdshft(j4,nperl);

			prdshft(i5,nperl);
			prdshft(j5,nperl);
			
			prdshft(i6,nperl);
			prdshft(j6,nperl);
			
			prdshft(i7,nperl);
			prdshft(j7,nperl);		

			cells1[itery] =  b1;
			cells2[itery] =  i3*nperl+j3;

			itery++;

			cells1[itery] =  b1;
			cells2[itery] =  i4*nperl+j4;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i5*nperl+j5;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i6*nperl+j6;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i7*nperl+j7;

			itery++;


		}
	}
	int size4 = nbpairs*sizeof(int);

	int *d_cells1;
	int *d_cells2;

	hipMalloc((void**)&d_cells1,size4);

	hipMalloc((void**)&d_cells2,size4);

	hipMemcpy(d_cells1,cells1,size4,hipMemcpyHostToDevice);
	hipMemcpy(d_cells2,cells2,size4,hipMemcpyHostToDevice);
	//int sibdiv = floor(ll/4.0);
	// print_device_array(d_cells1,nbpairs);
	// print_device_array(d_cells2,nbpairs);



	// matrix<int> boxes = (obj)->getgeo().generate_boxes_relationships(num,ccc);
	
	float2 *particles = new float2 [totalN];
	float2 *momenta = new float2 [totalN];
	int *p_indices = new int [totalN];

	for(int i = 0 ; i < totalN ; i++)
	p_indices[i]=i;

	float2 *d_particles;
	float2 *d_momenta;
	int *d_p_indices;

	int *d_bound;
	double *d_boundalong;
	int *d_changestate;

	hipMalloc((void**)&d_bound,(na+nb)*sizeof(int));
	hipMalloc((void**)&d_boundalong,(na+nb)*sizeof(double));
	hipMalloc((void**)&d_changestate,(na+nb)*sizeof(int));

	hipMemset(d_bound,0,(na+nb)*sizeof(int));
	hipMemset(d_boundalong,0.,(na+nb)*sizeof(double));
	hipMemset(d_changestate,0,(na+nb)*sizeof(int));

	double *d_totalforcex;
	double *d_totalforcey;

	hipMalloc((void**)&d_totalforcex,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey,totalN*sizeof(double));


	matrix<double> state(obj->getdat());


	for(int i = 0  ; i < totalN ; i++) {

	float2 c;
	c.x=state(i,0);
	c.y=state(i,1);

	(particles)[i]=c;

	float2 d;

	d.x = 0.;
	d.y = 0.;

	(momenta)[i]=d;
	}


	int size =  totalN*sizeof(float2);
	int size2 = totalN*sizeof(int);


	hipMalloc((void**)&d_particles,size);
	hipMalloc((void**)&d_momenta,size);
	hipMalloc((void**)&d_p_indices,size2);

	hipMemcpy(d_particles,particles,size,hipMemcpyHostToDevice);
	hipMemcpy(d_momenta,momenta,size,hipMemcpyHostToDevice);
	hipMemcpy(d_p_indices,p_indices,size2,hipMemcpyHostToDevice);


	// matrix<int> *froyo1 = obj->calculatepairs(boxes,pai,3.5);
	// matrix<int> *froyo2 = obj->calculatepairs(boxes,pbi,3.5);
	// matrix<int> *froyo3 = obj->calculatepairs(boxes,pci,3.5);
	// matrix<int> *froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
	// matrix<int> *froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
	// matrix<int> *froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);
	
	int *d_indices1;
	int *d_indices2;
	double *d_close;


	int tpp;
	


	construct_possible_pair_list(d_particles,d_p_indices,totalN,l,d_cells1,d_cells2,num,is_periodic,d_indices1,d_indices2,d_close,tpp);



	less_than_condition_AND cond1(SQR(3.5),0,na);
	less_than_condition_AND cond2(SQR(3.5),na,na+nb);
	less_than_condition_AND cond3(SQR(3.5),na+nb,na+nb+nc);
	less_than_condition_NAND cond4(SQR(3.5),0,na,na,na+nb);
	less_than_condition_NAND cond5(SQR(3.5),0,na,na+nb,na+nb+nc);
	less_than_condition_NAND cond6(SQR(3.5),na,na+nb,na+nb,na+nb+nc);


	int th1;
	int *d1_list1,*d1_list2,*d1_list3,*d1_list4;
	pairlist(d_indices1,d_indices2,d_close,cond1,d1_list1,d1_list2,d1_list3,d1_list4,tpp, th1); //faa

	int th2;
	int *d2_list1,*d2_list2,*d2_list3,*d2_list4;
	pairlist(d_indices1,d_indices2,d_close,cond2,d2_list1,d2_list2,d2_list3,d2_list4,tpp, th2);	//fbb

	int th3;
	int *d3_list1,*d3_list2,*d3_list3,*d3_list4;
	pairlist(d_indices1,d_indices2,d_close,cond3,d3_list1,d3_list2,d3_list3,d3_list4,tpp, th3);	//fcc

	int th4;
	int *d4_list1,*d4_list2,*d4_list3,*d4_list4;
	pairlist(d_indices1,d_indices2,d_close,cond4,d4_list1,d4_list2,d4_list3,d4_list4,tpp, th4);	//fab

	int th5;
	int *d5_list1,*d5_list2,*d5_list3,*d5_list4;
	pairlist(d_indices1,d_indices2,d_close,cond5,d5_list1,d5_list2,d5_list3,d5_list4,tpp, th5);	//fac

	int th6;
	int *d6_list1,*d6_list2,*d6_list3,*d6_list4;
	pairlist(d_indices1,d_indices2,d_close,cond6,d6_list1,d6_list2,d6_list3,d6_list4,tpp, th6); //fbc			
	//matrix<double> state(obj->getdat()); //the state of the system

	int th8 = (*bondpairs).getNsafe();
	int th9 = (*bendtriplets).getNsafe();


	int *d8_list1,*d8_list2,*d8_list3,*d8_list4;
	hipMalloc((void**)&d8_list1,th8*sizeof(int));
	hipMalloc((void**)&d8_list2,th8*sizeof(int));
	hipMalloc((void**)&d8_list3,th8*sizeof(int));
	hipMalloc((void**)&d8_list4,th8*sizeof(int));



	int *h8_list1 = new int [th8];
	int *h8_list2 = new int [th8];

	for(int i = 0 ; i < th8 ; i++ ) {
		h8_list1[i] = (*bondpairs)(i,0);
		h8_list2[i] = (*bondpairs)(i,1);
	}

	hipMemcpy(d8_list1,h8_list1,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list2,h8_list2,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list3,h8_list1,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list4,h8_list2,th8*sizeof(int),hipMemcpyHostToDevice);





	int *d9_list1,*d9_list2,*d9_list3,*d9_list4,*d9_list5,*d9_list6;
	hipMalloc((void**)&d9_list1,th9*sizeof(int));
	hipMalloc((void**)&d9_list2,th9*sizeof(int));
	hipMalloc((void**)&d9_list3,th9*sizeof(int));
	hipMalloc((void**)&d9_list4,th9*sizeof(int));
	hipMalloc((void**)&d9_list5,th9*sizeof(int));
	hipMalloc((void**)&d9_list6,th9*sizeof(int));

	int *h9_list1 = new int [th9];
	int *h9_list2 = new int [th9];
	int *h9_list3 = new int [th9];

	for(int i = 0 ; i < th9 ; i++ ) {
		h9_list1[i] = (*bendtriplets)(i,0);
		h9_list2[i] = (*bendtriplets)(i,1);
		h9_list3[i] = (*bendtriplets)(i,2);
	}	

	hipMemcpy(d9_list1,h9_list1,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list2,h9_list2,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list3,h9_list3,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list4,h9_list1,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list5,h9_list2,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list6,h9_list3,th9*sizeof(int),hipMemcpyHostToDevice);


	int i;

	double cons1;
	double cons2;
	double cons3;
	double cons4;

	//(mom)->operator()(i,i1) = c5*c2*((mom)->operator()(i,i1)) + (c5*(c3)+q)*F(i,i1) + (c5*(c4)+r)*R(i,i1);
	cons1 = (*obj).getc5()*(*obj).getc2();
	cons2 = (*obj).getc5()*(*obj).getc3()+(*obj).getq();
	cons3 = (*obj).getc5()*(*obj).getc4()+(*obj).getr();
	cons4 = (*obj).getc1();

	double d_dt = (*obj).getdt();
	double d_m = (*obj).getm();
	double d_kT = (*obj).getkT();


	// vector<matrix<double> > savef1forces;
	// vector<matrix<double> > savef2forces;
	// vector<matrix<double> > savef3forces;
	// vector<matrix<double> > savef4forces;
	// vector<matrix<double> > savef5forces;
	// vector<matrix<double> > savef6forces;
	// vector<matrix<double> > savef7forces;

	// vector<matrix<double> > saveftemp1forces;
	// vector<matrix<double> > saveftemp2forces;
	// vector<matrix<double> > saveftemp3forces;

	// vector<matrix<double> > savepositions;
	// vector<vector1<int> > savebound;
	// vector<vector1<double> > saveboundalongs;

	for(i = 0 ; i < runtime ; i++) {
		//cout << i << endl;

		cout << i << endl;
	
		//cout << (*obj).avmom() << endl;
	if(i%25==0) {
		//delete froyo1,froyo2,froyo3,froyo4,froyo5,froyo6;
		// cout << "updated after: " << i << endl;
		// state = obj->getdat();


		hipFree(d1_list1);hipFree(d1_list2);hipFree(d1_list3);hipFree(d1_list4);
		

		hipFree(d2_list1);hipFree(d2_list2);hipFree(d2_list3);hipFree(d2_list4);
		

		hipFree(d3_list1);hipFree(d3_list2);hipFree(d3_list3);hipFree(d3_list4);
		

		hipFree(d4_list1);hipFree(d4_list2);hipFree(d4_list3);hipFree(d4_list4);
		

		hipFree(d5_list1);hipFree(d5_list2);hipFree(d5_list3);hipFree(d5_list4);
	
		hipFree(d6_list1);hipFree(d6_list2);hipFree(d6_list3);hipFree(d6_list4);

		hipFree(d_indices1);

		hipFree(d_indices2);

		hipFree(d_close);

		this->resetindices(d_p_indices,totalN);


		construct_possible_pair_list(d_particles,d_p_indices,totalN,l,d_cells1,d_cells2,num,is_periodic,d_indices1,d_indices2,d_close,tpp,false);

		// cout << tpp << endl;
		// cout << "pair" << endl;

		pairlist(d_indices1,d_indices2,d_close,cond1,d1_list1,d1_list2,d1_list3,d1_list4,tpp, th1); //faa
		pairlist(d_indices1,d_indices2,d_close,cond2,d2_list1,d2_list2,d2_list3,d2_list4,tpp, th2); //fbb
		pairlist(d_indices1,d_indices2,d_close,cond3,d3_list1,d3_list2,d3_list3,d3_list4,tpp, th3); //fcc
		pairlist(d_indices1,d_indices2,d_close,cond4,d4_list1,d4_list2,d4_list3,d4_list4,tpp, th4); //fab
		pairlist(d_indices1,d_indices2,d_close,cond5,d5_list1,d5_list2,d5_list3,d5_list4,tpp, th5); //fac
		pairlist(d_indices1,d_indices2,d_close,cond6,d6_list1,d6_list2,d6_list3,d6_list4,tpp, th6); //fbc

	}




	double *d1_forces1x;
	double *d1_forces2x;
	double *d1_forces1y;
	double *d1_forces2y;
	hipMalloc((void**)&d1_forces1x,th1*sizeof(double));
	hipMalloc((void**)&d1_forces1y,th1*sizeof(double));
	hipMalloc((void**)&d1_forces2x,th1*sizeof(double));
	hipMalloc((void**)&d1_forces2y,th1*sizeof(double));
	calculateforces2D(d1_list1,d1_list2,d_particles, d1_forces1x,d1_forces1y,d1_forces2x,d1_forces2y, faa_gpu ,th1, l,true);

	// arracychck(d1_forces1x,th1);
	// arracychck(d1_forces1y,th1);
	// cout << "force1" << endl;

	double *d2_forces1x;
	double *d2_forces2x;
	double *d2_forces1y;
	double *d2_forces2y;
	hipMalloc((void**)&d2_forces1x,th2*sizeof(double));
	hipMalloc((void**)&d2_forces1y,th2*sizeof(double));
	hipMalloc((void**)&d2_forces2x,th2*sizeof(double));
	hipMalloc((void**)&d2_forces2y,th2*sizeof(double));
	calculateforces2D(d2_list1,d2_list2,d_particles, d2_forces1x,d2_forces1y,d2_forces2x,d2_forces2y, fbb_gpu ,th2, l,true);	



	// arracychck(d2_forces1x,th2);
	// arracychck(d2_forces1y,th2);

	// cout << "force2" << endl;
	double *d3_forces1x;
	double *d3_forces2x;
	double *d3_forces1y;
	double *d3_forces2y;
	hipMalloc((void**)&d3_forces1x,th3*sizeof(double));
	hipMalloc((void**)&d3_forces1y,th3*sizeof(double));
	hipMalloc((void**)&d3_forces2x,th3*sizeof(double));
	hipMalloc((void**)&d3_forces2y,th3*sizeof(double));
	calculateforces2D(d3_list1,d3_list2,d_particles, d3_forces1x,d3_forces1y,d3_forces2x,d3_forces2y, fcc_gpu ,th3, l,true);	

	// arracychck(d3_forces1x,th3);
	// arracychck(d3_forces1y,th3);

	// cout << "force3" << endl;
	double *d4_forces1x;
	double *d4_forces2x;
	double *d4_forces1y;
	double *d4_forces2y;
	hipMalloc((void**)&d4_forces1x,th4*sizeof(double));
	hipMalloc((void**)&d4_forces1y,th4*sizeof(double));
	hipMalloc((void**)&d4_forces2x,th4*sizeof(double));
	hipMalloc((void**)&d4_forces2y,th4*sizeof(double));
	calculateforces2D(d4_list1,d4_list2,d_particles, d4_forces1x,d4_forces1y,d4_forces2x,d4_forces2y, fab_gpu ,th4, l,true);
	
	// arracychck(d4_forces1x,th4);
	// arracychck(d4_forces1y,th4);
	// cout << "force4" << endl;

	double *d5_forces1x;
	double *d5_forces2x;
	double *d5_forces1y;
	double *d5_forces2y;
	hipMalloc((void**)&d5_forces1x,th5*sizeof(double));
	hipMalloc((void**)&d5_forces1y,th5*sizeof(double));
	hipMalloc((void**)&d5_forces2x,th5*sizeof(double));
	hipMalloc((void**)&d5_forces2y,th5*sizeof(double));
	calculateforces2D(d5_list1,d5_list2,d_particles, d5_forces1x,d5_forces1y,d5_forces2x,d5_forces2y, fbc_gpu ,th5, l,true);
	// arracychck(d5_forces1x,th5);
	// arracychck(d5_forces1y,th5);
	// cout << "force5" << endl;
	// cout << "d5" << endl;
	// print_device_weave_float2(d5_list1,d5_list2,d5_forces1x,d5_forces1y,d5_forces2x,d5_forces2y,d_particles,th5,totalN);
	// pausel();	

	double *d6_forces1x;
	double *d6_forces2x;
	double *d6_forces1y;
	double *d6_forces2y;
	hipMalloc((void**)&d6_forces1x,th6*sizeof(double));
	hipMalloc((void**)&d6_forces1y,th6*sizeof(double));
	hipMalloc((void**)&d6_forces2x,th6*sizeof(double));
	hipMalloc((void**)&d6_forces2y,th6*sizeof(double));
	calculateforces2D(d6_list1,d6_list2,d_particles, d6_forces1x,d6_forces1y,d6_forces2x,d6_forces2y, fac_gpu ,th6, l,true);	
	// arracychck(d6_forces1x,th6);
	// arracychck(d6_forces1y,th6);
	// cout << "force6" << endl;
	// cout << "d6" << endl;
	// print_device_weave_float2(d6_list1,d6_list2,d6_forces1x,d6_forces1y,d6_forces2x,d6_forces2y,d_particles,th6,totalN);
	// cout << endl;
	// pausel();

	// matrix<double> F6((*obj).calculateforces(*bondpairs,*bindm));
	callCalculateUnbindingsGPU(d_particles,d_bound,d_boundalong,d_changestate);



	// cout << "unbindings calculated" << endl;


	callCalculateBindingsGPU(d5_list1,d5_list2,d6_list1,d6_list2,d_particles, d_bound, d_boundalong,d_changestate,th5 ,th6 );


	// cout << "bindings calculated" << endl;


	int *d7_list1,*d7_list2,*d7_list3;
	double *d7_forces1x;
	double *d7_forces1y;
	double *d7_forces2x;
	double *d7_forces2y;
	double *d7_forces3x;
	double *d7_forces3y;
	int th7;
	BindingForcesGPU(d_particles, d_bound, d_boundalong, d7_list1,d7_list2,d7_list3, d7_forces1x, d7_forces1y, d7_forces2x,d7_forces2y,d7_forces3x, d7_forces3y, bindp_gpu, th7);

	// arracychck(d7_forces1x,th7);
	// arracychck(d7_forces1y,th7);
	// cout << "binding forces calculated" << endl;


	double *d8_forces1x;
	double *d8_forces2x;
	double *d8_forces1y;
	double *d8_forces2y;
	hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	hipMalloc((void**)&d8_forces2y,th8*sizeof(double));
	calculateforces2D(d8_list1,d8_list2,d_particles, d8_forces1x,d8_forces1y,d8_forces2x,d8_forces2y, bindm_gpu ,th8, l,true);
	
	// arracychck(d8_forces1x,th8);
	// arracychck(d8_forces1y,th8);

	// cout << "force 8" << endl;
	resetchangestate(d_changestate);
	// matrix<double> F7((*obj).calculateforces_threebody(*bendtriplets,*bendp));

//	int th9;


	double *d9_forces1x;
	double *d9_forces2x;
	double *d9_forces1y;
	double *d9_forces2y;
	double *d9_forces3x;
	double *d9_forces3y;
	// hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2y,th8*sizeof(double));	
	hipMalloc((void**)&d9_forces1x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces1y,th9*sizeof(double));
	hipMalloc((void**)&d9_forces2x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces2y,th9*sizeof(double));
	hipMalloc((void**)&d9_forces3x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces3y,th9*sizeof(double));
	BendingForcesGPU(d_particles, d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces1y,d9_forces2x,d9_forces2y, d9_forces3x, d9_forces3y,bendp_gpu,th9);

	// print_device_array_weave(d9_forces1x,d9_forces1y,th9);
	// print_device_array_weave(d9_forces2x,d9_forces2y,th9);
	// print_device_array_weave(d9_forces3x,d9_forces3y,th9);
	// cout << "force 9" << endl;	
	// arracychck(d9_forces1x,th9);
	// arracychck(d9_forces1y,th9);

	// print_device_float2(d_particles,totalN);
	// print_device_array(d9_list1,th9);
	// print_device_array(d9_list2,th9);
	// print_device_array(d9_list3,th9);

	// cout << "force9" << endl;

	//	matrix<double> F = ftemp1+ftemp2+ftemp3+F1+F2+F3+F4+F5+F6+F7;//+F4+F5;

		// matrix<double> R(totalN,dimension);
		// for(int i1 = 0 ; i1 < totalN ; i1++) {
		// 	for(int j = 0 ; j < dimension ; j++) {
		// 		R(i1,j) = (3.464101615 * ((double) rand() / (RAND_MAX)) - 1.732050808);
		// 	}
		// }

	int *d10_list1;
	double *d10_forces1x;
	double *d10_forces1y;	
	int th10;

	PositionForcesDueToAnglesGPU(d_particles, d_bound, d_boundalong, d10_list1, d10_forces1x, d10_forces1y,th10);
	

	// arracychck(d10_forces1x,th10);
	// arracychck(d10_forces1y,th10);
	// cout << "force10" << endl;
	// cout << "all forces calculated" << endl;





	resetforce(d_totalforcex);

	resetforce(d_totalforcey);

	//cout << "reset" << endl;

	// print_device_array(d_totalforcex,totalN);


	ReduceForces(d1_list1,d1_list2,d1_list3,d1_list4,d1_forces1x,d1_forces2x,d1_forces1y,d1_forces2y,d_totalforcex,d_totalforcey,th1);
	// cout << "d1" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	
	ReduceForces(d2_list1,d2_list2,d2_list3,d2_list4,d2_forces1x,d2_forces2x,d2_forces1y,d2_forces2y,d_totalforcex,d_totalforcey,th2);
	/// cout << "d2" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	ReduceForces(d3_list1,d3_list2,d3_list3,d3_list4,d3_forces1x,d3_forces2x,d3_forces1y,d3_forces2y,d_totalforcex,d_totalforcey,th3);
	// cout << "d3" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	
	ReduceForces(d4_list1,d4_list2,d4_list3,d4_list4,d4_forces1x,d4_forces2x,d4_forces1y,d4_forces2y,d_totalforcex,d_totalforcey,th4);
	// cout << "d4" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	
	ReduceForces(d5_list1,d5_list2,d5_list3,d5_list4,d5_forces1x,d5_forces2x,d5_forces1y,d5_forces2y,d_totalforcex,d_totalforcey,th5);
	// cout << "d5" << endl;	
	// print_device_weave_float2(d5_list1,d5_list2,d5_forces1x,d5_forces1y,d_particles,th5,totalN);
	// pausel();	
	ReduceForces(d6_list1,d6_list2,d6_list3,d6_list4,d6_forces1x,d6_forces2x,d6_forces1y,d6_forces2y,d_totalforcex,d_totalforcey,th6);	
	// cout << "d6" << endl;
	// print_device_weave_float2(d6_list1,d6_list2,d6_forces1x,d6_forces1y,d_particles,th6,totalN);
	// cout << endl;
	// pausel();
	ReduceForces3(d7_list1,d7_list2,d7_list3,d7_forces1x,d7_forces2x,d7_forces3x,d7_forces1y,d7_forces2y,d7_forces3y,d_totalforcex,d_totalforcey,th7);	
	// cout << "d7" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	ReduceForces(d8_list1,d8_list2,d8_forces1x,d8_forces2x,d8_forces1y,d8_forces2y,d_totalforcex,d_totalforcey,th8);
	// cout << "d8" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	ReduceForces3(d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces2x,d9_forces3x,d9_forces1y,d9_forces2y,d9_forces3y,d_totalforcex,d_totalforcey,th9);		
	
	// cout << "d9" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double ff = (v0_a+v0_b)/2.;
	ReduceForcesAndNormalize(d10_list1,d10_forces1x,d10_forces1y,d_totalforcex,d_totalforcey, max_s, ff, th10);

	cout << "reduction" << endl;
	// cout << "d10" << endl;
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	if(i>0&&i%every==0) { 
		// for(int j = 0 ; j < na+nb ; j++) {
		// if(bound[j]>0){
		// cout << "printed" << endl;
		// cout << j << endl;
		// cout << bound[j] << endl;
		// cout << bound_along[j] << endl;
		// cout << F5(j,'r') << endl;
		// cout << F4(j,'r') << endl;
		// cout << obj->getcoordinate(j,0) << " " << obj->getcoordinate(j,1) << endl;
		// cout << obj->getcoordinate(200,0) <<  " " << obj->getcoordinate(200,1) << endl;
		// cout << obj->getcoordinate(200+L,0) <<  " " << obj->getcoordinate(200+L,1) << endl;
		// }
		// }
		// cout << F6 << endl;
		// cout << F7 << endl;

		// cout << ftemp2 << endl;
		// cout << ftemp3 << endl;

		stringstream ss2;
		// ss2 <<i/every;
		// string pairlist = "list";

		 stringstream kts;
		 kts << (*obj).getkT();

		 // stringstream epi;
		 // epi << eps;

		 // stringstream epieq;
		 // epieq << eqeps;

		 stringstream len;
		 len << l;

		 string extension =  "_kT="+kts.str()+"_l="+len.str()+".csv";

		stringstream ss;
		ss <<(i/every);
		string filename = "x";
		filename += ss.str();
		filename += extension;

		string momname = "bind";
		momname += ss.str();
		momname += extension;

		string baname = "bind_along";
		baname += ss.str();
		baname += extension;



		ofstream myfile;
		myfile.open(filename.c_str());
		//myfile <<= (*obj).getdat();
		file_print_device_float2(d_particles,totalN,myfile);
		myfile.close();



		}	

	double *d_R1;
	double *d_R2;

	// hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2y,th8*sizeof(double));	
	hipMalloc((void**)&d_R1,totalN*sizeof(double));
	hipMalloc((void**)&d_R2,totalN*sizeof(double));

	setstaterandom(d_R1,1.732050808,totalN);
 	setstaterandom(d_R2,1.732050808,totalN);	

 	//advmom2D(d_momenta, d_totalforcex, d_totalforcey, d_R1, d_R2, cons1,cons2,cons3,totalN);
 	advmom2D_spatialdependence(d_momenta,d_particles,d_totalforcex,d_totalforcey,d_R1,d_R2,func, d_dt, d_kT, d_m, totalN);
 	advpos2D(d_particles, d_momenta, cons4, totalN);


 	applypbc2D(d_particles,d_momenta,l,is_periodic,totalN);

 	cout << "updated" << endl;

	hipFree(d1_forces1x);
	hipFree(d1_forces2x);
	hipFree(d1_forces1y);
	hipFree(d1_forces2y);
	hipFree(d2_forces1x);
	hipFree(d2_forces2x);
	hipFree(d2_forces1y);
	hipFree(d2_forces2y);
	hipFree(d3_forces1x);
	hipFree(d3_forces2x);
	hipFree(d3_forces1y);
	hipFree(d3_forces2y);
	hipFree(d4_forces1x);
	hipFree(d4_forces2x);
	hipFree(d4_forces1y);
	hipFree(d4_forces2y);
	hipFree(d5_forces1x);
	hipFree(d5_forces2x);
	hipFree(d5_forces1y);
	hipFree(d5_forces2y);
	hipFree(d6_forces1x);
	hipFree(d6_forces2x);
	hipFree(d6_forces1y);
	hipFree(d6_forces2y);
 	hipFree(d7_list1);
 	hipFree(d7_list2);
 	hipFree(d7_list3);
	hipFree(d7_forces1x);
	hipFree(d7_forces2x);
	hipFree(d7_forces3x);
	hipFree(d7_forces1y);
	hipFree(d7_forces2y);
	hipFree(d7_forces3y);
	hipFree(d8_forces1x);
	hipFree(d8_forces2x);
	hipFree(d8_forces1y);
	hipFree(d8_forces2y);
	hipFree(d9_forces1x);
	hipFree(d9_forces2x);
	hipFree(d9_forces3x);
	hipFree(d9_forces1y);
	hipFree(d9_forces2y);
	hipFree(d9_forces3y);
	hipFree(d10_list1);
	hipFree(d10_forces1x);
	hipFree(d10_forces1y);

	cout << "freed" << endl;


	}
}


template <typename Fun>
void Microtubule::runGPUPV(int runtime, int every, Fun func)
{
//	pausel();
	//int ccc;
	int totalN = obj->getN();

	//num is the number of boxes per length

	int ncells = num*num;

	WCApotentialGPU faa_gpu(2.,1.,2.);
	WCApotentialGPU fab_gpu(1.,1.,0.);
	WCApotentialGPU fac_gpu(1.,1.,0.);
	WCApotentialGPU fbb_gpu(2.,1.,2.);
	WCApotentialGPU fbc_gpu(1.,1.,0.);
	WCApotentialGPU fcc_gpu(1.,1.,0.);
	HarmonicPotentialGPU bindp_gpu(100.,1.); 
	FENEPotentialGPU bindm_gpu(50.,1.5); 
	BendingPotentialGPU bendp_gpu(100.,0.);

//we now have the count of each cell list
	int nbpairs = 5*ncells;
	int nperl = num;

	int *cells1 = new int [nbpairs];
	int *cells2 = new int [nbpairs];


	int itery = 0;
	for(int i1 = 0 ; i1 < num ; i1++) {
		for(int i2 = 0 ; i2 < num ; i2++ ) {


			int b1 =  i1*nperl+i2;

			int i3 = i1+0;
			int j3 = i2+0;

			int i4 = i1+1;
			int j4 = i2+0;

			int i5 = i1-1;
			int j5 = i2+1;

			int i6 = i1+0;
			int j6 = i2+1;

			int i7 = i1+1;
			int j7 = i2+1;

			prdshft(i3,nperl);
			prdshft(j3,nperl);

			prdshft(i4,nperl);
			prdshft(j4,nperl);

			prdshft(i5,nperl);
			prdshft(j5,nperl);
			
			prdshft(i6,nperl);
			prdshft(j6,nperl);
			
			prdshft(i7,nperl);
			prdshft(j7,nperl);		

			cells1[itery] =  b1;
			cells2[itery] =  i3*nperl+j3;

			itery++;

			cells1[itery] =  b1;
			cells2[itery] =  i4*nperl+j4;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i5*nperl+j5;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i6*nperl+j6;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i7*nperl+j7;

			itery++;


		}
	}
	int size4 = nbpairs*sizeof(int);

	int *d_cells1;
	int *d_cells2;

	hipMalloc((void**)&d_cells1,size4);

	hipMalloc((void**)&d_cells2,size4);

	hipMemcpy(d_cells1,cells1,size4,hipMemcpyHostToDevice);
	hipMemcpy(d_cells2,cells2,size4,hipMemcpyHostToDevice);
	//int sibdiv = floor(ll/4.0);
	// print_device_array(d_cells1,nbpairs);
	// print_device_array(d_cells2,nbpairs);



	// matrix<int> boxes = (obj)->getgeo().generate_boxes_relationships(num,ccc);
	
	float2 *particles = new float2 [totalN];
	float2 *momenta = new float2 [totalN];
	int *p_indices = new int [totalN];

	for(int i = 0 ; i < totalN ; i++)
	p_indices[i]=i;

	float2 *d_particles;
	float2 *d_momenta;
	int *d_p_indices;

	int *d_bound;
	double *d_boundalong;
	int *d_changestate;

	hipMalloc((void**)&d_bound,(na+nb)*sizeof(int));
	hipMalloc((void**)&d_boundalong,(na+nb)*sizeof(double));
	hipMalloc((void**)&d_changestate,(na+nb)*sizeof(int));

	hipMemset(d_bound,0,(na+nb)*sizeof(int));
	hipMemset(d_boundalong,0.,(na+nb)*sizeof(double));
	hipMemset(d_changestate,0,(na+nb)*sizeof(int));

	double *d_totalforcex;
	double *d_totalforcey;

	hipMalloc((void**)&d_totalforcex,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey,totalN*sizeof(double));


	matrix<double> state(obj->getdat());


	for(int i = 0  ; i < totalN ; i++) {

	float2 c;
	c.x=state(i,0);
	c.y=state(i,1);

	(particles)[i]=c;

	float2 d;

	d.x = 0.;
	d.y = 0.;

	(momenta)[i]=d;
	}


	int size =  totalN*sizeof(float2);
	int size2 = totalN*sizeof(int);


	hipMalloc((void**)&d_particles,size);
	hipMalloc((void**)&d_momenta,size);
	hipMalloc((void**)&d_p_indices,size2);

	hipMemcpy(d_particles,particles,size,hipMemcpyHostToDevice);
	hipMemcpy(d_momenta,momenta,size,hipMemcpyHostToDevice);
	hipMemcpy(d_p_indices,p_indices,size2,hipMemcpyHostToDevice);


	// matrix<int> *froyo1 = obj->calculatepairs(boxes,pai,3.5);
	// matrix<int> *froyo2 = obj->calculatepairs(boxes,pbi,3.5);
	// matrix<int> *froyo3 = obj->calculatepairs(boxes,pci,3.5);
	// matrix<int> *froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
	// matrix<int> *froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
	// matrix<int> *froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);
	
	int *d_indices1;
	int *d_indices2;
	double *d_close;


	int tpp;
	


	construct_possible_pair_list(d_particles,d_p_indices,totalN,l,d_cells1,d_cells2,num,is_periodic,d_indices1,d_indices2,d_close,tpp);



	less_than_condition_AND cond1(SQR(3.5),0,na);
	less_than_condition_AND cond2(SQR(3.5),na,na+nb);
	less_than_condition_AND cond3(SQR(3.5),na+nb,na+nb+nc);
	less_than_condition_NAND cond4(SQR(3.5),0,na,na,na+nb);
	less_than_condition_NAND cond5(SQR(3.5),0,na,na+nb,na+nb+nc);
	less_than_condition_NAND cond6(SQR(3.5),na,na+nb,na+nb,na+nb+nc);


	int th1;
	int *d1_list1,*d1_list2,*d1_list3,*d1_list4;
	pairlist(d_indices1,d_indices2,d_close,cond1,d1_list1,d1_list2,d1_list3,d1_list4,tpp, th1); //faa

	int th2;
	int *d2_list1,*d2_list2,*d2_list3,*d2_list4;
	pairlist(d_indices1,d_indices2,d_close,cond2,d2_list1,d2_list2,d2_list3,d2_list4,tpp, th2);	//fbb

	int th3;
	int *d3_list1,*d3_list2,*d3_list3,*d3_list4;
	pairlist(d_indices1,d_indices2,d_close,cond3,d3_list1,d3_list2,d3_list3,d3_list4,tpp, th3);	//fcc

	int th4;
	int *d4_list1,*d4_list2,*d4_list3,*d4_list4;
	pairlist(d_indices1,d_indices2,d_close,cond4,d4_list1,d4_list2,d4_list3,d4_list4,tpp, th4);	//fab

	int th5;
	int *d5_list1,*d5_list2,*d5_list3,*d5_list4;
	pairlist(d_indices1,d_indices2,d_close,cond5,d5_list1,d5_list2,d5_list3,d5_list4,tpp, th5);	//fac

	int th6;
	int *d6_list1,*d6_list2,*d6_list3,*d6_list4;
	pairlist(d_indices1,d_indices2,d_close,cond6,d6_list1,d6_list2,d6_list3,d6_list4,tpp, th6); //fbc			
	//matrix<double> state(obj->getdat()); //the state of the system

	int th8 = (*bondpairs).getNsafe();
	int th9 = (*bendtriplets).getNsafe();


	int *d8_list1,*d8_list2,*d8_list3,*d8_list4;
	hipMalloc((void**)&d8_list1,th8*sizeof(int));
	hipMalloc((void**)&d8_list2,th8*sizeof(int));
	hipMalloc((void**)&d8_list3,th8*sizeof(int));
	hipMalloc((void**)&d8_list4,th8*sizeof(int));



	int *h8_list1 = new int [th8];
	int *h8_list2 = new int [th8];

	for(int i = 0 ; i < th8 ; i++ ) {
		h8_list1[i] = (*bondpairs)(i,0);
		h8_list2[i] = (*bondpairs)(i,1);
	}

	hipMemcpy(d8_list1,h8_list1,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list2,h8_list2,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list3,h8_list1,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list4,h8_list2,th8*sizeof(int),hipMemcpyHostToDevice);





	int *d9_list1,*d9_list2,*d9_list3,*d9_list4,*d9_list5,*d9_list6;
	hipMalloc((void**)&d9_list1,th9*sizeof(int));
	hipMalloc((void**)&d9_list2,th9*sizeof(int));
	hipMalloc((void**)&d9_list3,th9*sizeof(int));
	hipMalloc((void**)&d9_list4,th9*sizeof(int));
	hipMalloc((void**)&d9_list5,th9*sizeof(int));
	hipMalloc((void**)&d9_list6,th9*sizeof(int));

	int *h9_list1 = new int [th9];
	int *h9_list2 = new int [th9];
	int *h9_list3 = new int [th9];

	for(int i = 0 ; i < th9 ; i++ ) {
		h9_list1[i] = (*bendtriplets)(i,0);
		h9_list2[i] = (*bendtriplets)(i,1);
		h9_list3[i] = (*bendtriplets)(i,2);
	}	

	hipMemcpy(d9_list1,h9_list1,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list2,h9_list2,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list3,h9_list3,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list4,h9_list1,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list5,h9_list2,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list6,h9_list3,th9*sizeof(int),hipMemcpyHostToDevice);


	int i;

	double cons1;
	double cons2;
	double cons3;
	double cons4;

	//(mom)->operator()(i,i1) = c5*c2*((mom)->operator()(i,i1)) + (c5*(c3)+q)*F(i,i1) + (c5*(c4)+r)*R(i,i1);
	cons1 = (*obj).getc5()*(*obj).getc2();
	cons2 = (*obj).getc5()*(*obj).getc3()+(*obj).getq();
	cons3 = (*obj).getc5()*(*obj).getc4()+(*obj).getr();
	cons4 = (*obj).getc1();

	double d_dt = (*obj).getdt();
	double d_m = (*obj).getm();
	double d_kT = (*obj).getkT();


	// vector<matrix<double> > savef1forces;
	// vector<matrix<double> > savef2forces;
	// vector<matrix<double> > savef3forces;
	// vector<matrix<double> > savef4forces;
	// vector<matrix<double> > savef5forces;
	// vector<matrix<double> > savef6forces;
	// vector<matrix<double> > savef7forces;

	// vector<matrix<double> > saveftemp1forces;
	// vector<matrix<double> > saveftemp2forces;
	// vector<matrix<double> > saveftemp3forces;

	// vector<matrix<double> > savepositions;
	// vector<vector1<int> > savebound;
	// vector<vector1<double> > saveboundalongs;

	for(i = 0 ; i < runtime ; i++) {
		//cout << i << endl;

		cout << i << endl;
	
		//cout << (*obj).avmom() << endl;
	if(i%25==0) {
		//delete froyo1,froyo2,froyo3,froyo4,froyo5,froyo6;
		// cout << "updated after: " << i << endl;
		// state = obj->getdat();


		hipFree(d1_list1);hipFree(d1_list2);hipFree(d1_list3);hipFree(d1_list4);
		

		hipFree(d2_list1);hipFree(d2_list2);hipFree(d2_list3);hipFree(d2_list4);
		

		hipFree(d3_list1);hipFree(d3_list2);hipFree(d3_list3);hipFree(d3_list4);
		

		hipFree(d4_list1);hipFree(d4_list2);hipFree(d4_list3);hipFree(d4_list4);
		

		hipFree(d5_list1);hipFree(d5_list2);hipFree(d5_list3);hipFree(d5_list4);
	
		hipFree(d6_list1);hipFree(d6_list2);hipFree(d6_list3);hipFree(d6_list4);

		hipFree(d_indices1);

		hipFree(d_indices2);

		hipFree(d_close);

		this->resetindices(d_p_indices,totalN);


		construct_possible_pair_list(d_particles,d_p_indices,totalN,l,d_cells1,d_cells2,num,is_periodic,d_indices1,d_indices2,d_close,tpp,false);

		// cout << tpp << endl;
		// cout << "pair" << endl;

		pairlist(d_indices1,d_indices2,d_close,cond1,d1_list1,d1_list2,d1_list3,d1_list4,tpp, th1); //faa
		pairlist(d_indices1,d_indices2,d_close,cond2,d2_list1,d2_list2,d2_list3,d2_list4,tpp, th2); //fbb
		pairlist(d_indices1,d_indices2,d_close,cond3,d3_list1,d3_list2,d3_list3,d3_list4,tpp, th3); //fcc
		pairlist(d_indices1,d_indices2,d_close,cond4,d4_list1,d4_list2,d4_list3,d4_list4,tpp, th4); //fab
		pairlist(d_indices1,d_indices2,d_close,cond5,d5_list1,d5_list2,d5_list3,d5_list4,tpp, th5); //fac
		pairlist(d_indices1,d_indices2,d_close,cond6,d6_list1,d6_list2,d6_list3,d6_list4,tpp, th6); //fbc

	}




	double *d1_forces1x;
	double *d1_forces2x;
	double *d1_forces1y;
	double *d1_forces2y;
	hipMalloc((void**)&d1_forces1x,th1*sizeof(double));
	hipMalloc((void**)&d1_forces1y,th1*sizeof(double));
	hipMalloc((void**)&d1_forces2x,th1*sizeof(double));
	hipMalloc((void**)&d1_forces2y,th1*sizeof(double));
	calculateforces2D(d1_list1,d1_list2,d_particles, d1_forces1x,d1_forces1y,d1_forces2x,d1_forces2y, faa_gpu ,th1, l,true);

	// arracychck(d1_forces1x,th1);
	// arracychck(d1_forces1y,th1);
	// cout << "force1" << endl;

	double *d2_forces1x;
	double *d2_forces2x;
	double *d2_forces1y;
	double *d2_forces2y;
	hipMalloc((void**)&d2_forces1x,th2*sizeof(double));
	hipMalloc((void**)&d2_forces1y,th2*sizeof(double));
	hipMalloc((void**)&d2_forces2x,th2*sizeof(double));
	hipMalloc((void**)&d2_forces2y,th2*sizeof(double));
	calculateforces2D(d2_list1,d2_list2,d_particles, d2_forces1x,d2_forces1y,d2_forces2x,d2_forces2y, fbb_gpu ,th2, l,true);	



	// arracychck(d2_forces1x,th2);
	// arracychck(d2_forces1y,th2);

	// cout << "force2" << endl;
	double *d3_forces1x;
	double *d3_forces2x;
	double *d3_forces1y;
	double *d3_forces2y;
	hipMalloc((void**)&d3_forces1x,th3*sizeof(double));
	hipMalloc((void**)&d3_forces1y,th3*sizeof(double));
	hipMalloc((void**)&d3_forces2x,th3*sizeof(double));
	hipMalloc((void**)&d3_forces2y,th3*sizeof(double));
	calculateforces2D(d3_list1,d3_list2,d_particles, d3_forces1x,d3_forces1y,d3_forces2x,d3_forces2y, fcc_gpu ,th3, l,true);	

	// arracychck(d3_forces1x,th3);
	// arracychck(d3_forces1y,th3);

	// cout << "force3" << endl;
	double *d4_forces1x;
	double *d4_forces2x;
	double *d4_forces1y;
	double *d4_forces2y;
	hipMalloc((void**)&d4_forces1x,th4*sizeof(double));
	hipMalloc((void**)&d4_forces1y,th4*sizeof(double));
	hipMalloc((void**)&d4_forces2x,th4*sizeof(double));
	hipMalloc((void**)&d4_forces2y,th4*sizeof(double));
	calculateforces2D(d4_list1,d4_list2,d_particles, d4_forces1x,d4_forces1y,d4_forces2x,d4_forces2y, fab_gpu ,th4, l,true);
	
	// arracychck(d4_forces1x,th4);
	// arracychck(d4_forces1y,th4);
	// cout << "force4" << endl;

	double *d5_forces1x;
	double *d5_forces2x;
	double *d5_forces1y;
	double *d5_forces2y;
	hipMalloc((void**)&d5_forces1x,th5*sizeof(double));
	hipMalloc((void**)&d5_forces1y,th5*sizeof(double));
	hipMalloc((void**)&d5_forces2x,th5*sizeof(double));
	hipMalloc((void**)&d5_forces2y,th5*sizeof(double));
	calculateforces2D(d5_list1,d5_list2,d_particles, d5_forces1x,d5_forces1y,d5_forces2x,d5_forces2y, fbc_gpu ,th5, l,true);
	// arracychck(d5_forces1x,th5);
	// arracychck(d5_forces1y,th5);
	// cout << "force5" << endl;
	// cout << "d5" << endl;
	// print_device_weave_float2(d5_list1,d5_list2,d5_forces1x,d5_forces1y,d5_forces2x,d5_forces2y,d_particles,th5,totalN);
	// pausel();	

	double *d6_forces1x;
	double *d6_forces2x;
	double *d6_forces1y;
	double *d6_forces2y;
	hipMalloc((void**)&d6_forces1x,th6*sizeof(double));
	hipMalloc((void**)&d6_forces1y,th6*sizeof(double));
	hipMalloc((void**)&d6_forces2x,th6*sizeof(double));
	hipMalloc((void**)&d6_forces2y,th6*sizeof(double));
	calculateforces2D(d6_list1,d6_list2,d_particles, d6_forces1x,d6_forces1y,d6_forces2x,d6_forces2y, fac_gpu ,th6, l,true);	
	// arracychck(d6_forces1x,th6);
	// arracychck(d6_forces1y,th6);
	// cout << "force6" << endl;
	// cout << "d6" << endl;
	// print_device_weave_float2(d6_list1,d6_list2,d6_forces1x,d6_forces1y,d6_forces2x,d6_forces2y,d_particles,th6,totalN);
	// cout << endl;
	// pausel();

	// matrix<double> F6((*obj).calculateforces(*bondpairs,*bindm));
	callCalculateUnbindingsGPU(d_particles,d_bound,d_boundalong,d_changestate);



	// cout << "unbindings calculated" << endl;


	callCalculateBindingsGPU(d5_list1,d5_list2,d6_list1,d6_list2,d_particles, d_bound, d_boundalong,d_changestate,th5 ,th6 );


	// cout << "bindings calculated" << endl;


	int *d7_list1,*d7_list2,*d7_list3;
	double *d7_forces1x;
	double *d7_forces1y;
	double *d7_forces2x;
	double *d7_forces2y;
	double *d7_forces3x;
	double *d7_forces3y;
	int th7;
	BindingForcesGPU(d_particles, d_bound, d_boundalong, d7_list1,d7_list2,d7_list3, d7_forces1x, d7_forces1y, d7_forces2x,d7_forces2y,d7_forces3x, d7_forces3y, bindp_gpu, th7);

	// arracychck(d7_forces1x,th7);
	// arracychck(d7_forces1y,th7);
	// cout << "binding forces calculated" << endl;


	double *d8_forces1x;
	double *d8_forces2x;
	double *d8_forces1y;
	double *d8_forces2y;
	hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	hipMalloc((void**)&d8_forces2y,th8*sizeof(double));
	calculateforces2D(d8_list1,d8_list2,d_particles, d8_forces1x,d8_forces1y,d8_forces2x,d8_forces2y, bindm_gpu ,th8, l,true);
	
	// arracychck(d8_forces1x,th8);
	// arracychck(d8_forces1y,th8);

	// cout << "force 8" << endl;
	resetchangestate(d_changestate);
	// matrix<double> F7((*obj).calculateforces_threebody(*bendtriplets,*bendp));

//	int th9;


	double *d9_forces1x;
	double *d9_forces2x;
	double *d9_forces1y;
	double *d9_forces2y;
	double *d9_forces3x;
	double *d9_forces3y;
	// hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2y,th8*sizeof(double));	
	hipMalloc((void**)&d9_forces1x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces1y,th9*sizeof(double));
	hipMalloc((void**)&d9_forces2x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces2y,th9*sizeof(double));
	hipMalloc((void**)&d9_forces3x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces3y,th9*sizeof(double));
	BendingForcesGPU(d_particles, d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces1y,d9_forces2x,d9_forces2y, d9_forces3x, d9_forces3y,bendp_gpu,th9);

	// print_device_array_weave(d9_forces1x,d9_forces1y,th9);
	// print_device_array_weave(d9_forces2x,d9_forces2y,th9);
	// print_device_array_weave(d9_forces3x,d9_forces3y,th9);
	// cout << "force 9" << endl;	
	// arracychck(d9_forces1x,th9);
	// arracychck(d9_forces1y,th9);

	// print_device_float2(d_particles,totalN);
	// print_device_array(d9_list1,th9);
	// print_device_array(d9_list2,th9);
	// print_device_array(d9_list3,th9);

	// cout << "force9" << endl;

	//	matrix<double> F = ftemp1+ftemp2+ftemp3+F1+F2+F3+F4+F5+F6+F7;//+F4+F5;

		// matrix<double> R(totalN,dimension);
		// for(int i1 = 0 ; i1 < totalN ; i1++) {
		// 	for(int j = 0 ; j < dimension ; j++) {
		// 		R(i1,j) = (3.464101615 * ((double) rand() / (RAND_MAX)) - 1.732050808);
		// 	}
		// }

	int *d10_list1;
	double *d10_forces1x;
	double *d10_forces1y;	
	int th10;

	PositionForcesDueToAnglesGPU(d_particles, d_bound, d_boundalong, d10_list1, d10_forces1x, d10_forces1y,th10);
	

	// arracychck(d10_forces1x,th10);
	// arracychck(d10_forces1y,th10);
	// cout << "force10" << endl;
	// cout << "all forces calculated" << endl;





	resetforce(d_totalforcex);

	resetforce(d_totalforcey);

	//cout << "reset" << endl;

	// print_device_array(d_totalforcex,totalN);


	ReduceForces(d1_list1,d1_list2,d1_list3,d1_list4,d1_forces1x,d1_forces2x,d1_forces1y,d1_forces2y,d_totalforcex,d_totalforcey,th1);
	// cout << "d1" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	
	ReduceForces(d2_list1,d2_list2,d2_list3,d2_list4,d2_forces1x,d2_forces2x,d2_forces1y,d2_forces2y,d_totalforcex,d_totalforcey,th2);
	/// cout << "d2" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	ReduceForces(d3_list1,d3_list2,d3_list3,d3_list4,d3_forces1x,d3_forces2x,d3_forces1y,d3_forces2y,d_totalforcex,d_totalforcey,th3);
	// cout << "d3" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	
	ReduceForces(d4_list1,d4_list2,d4_list3,d4_list4,d4_forces1x,d4_forces2x,d4_forces1y,d4_forces2y,d_totalforcex,d_totalforcey,th4);
	// cout << "d4" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	
	ReduceForces(d5_list1,d5_list2,d5_list3,d5_list4,d5_forces1x,d5_forces2x,d5_forces1y,d5_forces2y,d_totalforcex,d_totalforcey,th5);
	// cout << "d5" << endl;	
	// print_device_weave_float2(d5_list1,d5_list2,d5_forces1x,d5_forces1y,d_particles,th5,totalN);
	// pausel();	
	ReduceForces(d6_list1,d6_list2,d6_list3,d6_list4,d6_forces1x,d6_forces2x,d6_forces1y,d6_forces2y,d_totalforcex,d_totalforcey,th6);	
	// cout << "d6" << endl;
	// print_device_weave_float2(d6_list1,d6_list2,d6_forces1x,d6_forces1y,d_particles,th6,totalN);
	// cout << endl;
	// pausel();
	ReduceForces3(d7_list1,d7_list2,d7_list3,d7_forces1x,d7_forces2x,d7_forces3x,d7_forces1y,d7_forces2y,d7_forces3y,d_totalforcex,d_totalforcey,th7);	
	// cout << "d7" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	ReduceForces(d8_list1,d8_list2,d8_forces1x,d8_forces2x,d8_forces1y,d8_forces2y,d_totalforcex,d_totalforcey,th8);
	// cout << "d8" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	ReduceForces3(d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces2x,d9_forces3x,d9_forces1y,d9_forces2y,d9_forces3y,d_totalforcex,d_totalforcey,th9);		
	
	// cout << "d9" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double ff = (v0_a+v0_b)/2.;
	ReduceForcesAndNormalize(d10_list1,d10_forces1x,d10_forces1y,d_totalforcex,d_totalforcey, max_s, ff, th10);

	cout << "reduction" << endl;
	// cout << "d10" << endl;
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	if(i>0&&i%every==0) { 
		// for(int j = 0 ; j < na+nb ; j++) {
		// if(bound[j]>0){
		// cout << "printed" << endl;
		// cout << j << endl;
		// cout << bound[j] << endl;
		// cout << bound_along[j] << endl;
		// cout << F5(j,'r') << endl;
		// cout << F4(j,'r') << endl;
		// cout << obj->getcoordinate(j,0) << " " << obj->getcoordinate(j,1) << endl;
		// cout << obj->getcoordinate(200,0) <<  " " << obj->getcoordinate(200,1) << endl;
		// cout << obj->getcoordinate(200+L,0) <<  " " << obj->getcoordinate(200+L,1) << endl;
		// }
		// }
		// cout << F6 << endl;
		// cout << F7 << endl;

		// cout << ftemp2 << endl;
		// cout << ftemp3 << endl;

		stringstream ss2;
		// ss2 <<i/every;
		// string pairlist = "list";

		 stringstream kts;
		 kts << (*obj).getkT();

		 // stringstream epi;
		 // epi << eps;

		 // stringstream epieq;
		 // epieq << eqeps;

		 stringstream len;
		 len << l;

		 string extension =  "_kT="+kts.str()+"_l="+len.str()+".csv";

		stringstream ss;
		ss <<(i/every);
		string filename = "x";
		filename += ss.str();
		filename += extension;

		string momname = "bind";
		momname += ss.str();
		momname += extension;

		string baname = "bind_along";
		baname += ss.str();
		baname += extension;



		ofstream myfile;
		myfile.open(filename.c_str());
		//myfile <<= (*obj).getdat();
		file_print_device_float2(d_particles,totalN,myfile);
		myfile.close();



		}	

	double *d_R1;
	double *d_R2;

	// hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2y,th8*sizeof(double));	
	hipMalloc((void**)&d_R1,totalN*sizeof(double));
	hipMalloc((void**)&d_R2,totalN*sizeof(double));

	setstaterandom(d_R1,1.732050808,totalN);
 	setstaterandom(d_R2,1.732050808,totalN);	

 	//advmom2D(d_momenta, d_totalforcex, d_totalforcey, d_R1, d_R2, cons1,cons2,cons3,totalN);
 	advmom2D_particledependence(d_momenta,d_totalforcex,d_totalforcey,d_R1,d_R2,func, d_dt, d_kT, d_m, totalN);
 	advpos2D(d_particles, d_momenta, cons4, totalN);


 	applypbc2D(d_particles,d_momenta,l,is_periodic,totalN);

 	cout << "updated" << endl;

	hipFree(d1_forces1x);
	hipFree(d1_forces2x);
	hipFree(d1_forces1y);
	hipFree(d1_forces2y);
	hipFree(d2_forces1x);
	hipFree(d2_forces2x);
	hipFree(d2_forces1y);
	hipFree(d2_forces2y);
	hipFree(d3_forces1x);
	hipFree(d3_forces2x);
	hipFree(d3_forces1y);
	hipFree(d3_forces2y);
	hipFree(d4_forces1x);
	hipFree(d4_forces2x);
	hipFree(d4_forces1y);
	hipFree(d4_forces2y);
	hipFree(d5_forces1x);
	hipFree(d5_forces2x);
	hipFree(d5_forces1y);
	hipFree(d5_forces2y);
	hipFree(d6_forces1x);
	hipFree(d6_forces2x);
	hipFree(d6_forces1y);
	hipFree(d6_forces2y);
 	hipFree(d7_list1);
 	hipFree(d7_list2);
 	hipFree(d7_list3);
	hipFree(d7_forces1x);
	hipFree(d7_forces2x);
	hipFree(d7_forces3x);
	hipFree(d7_forces1y);
	hipFree(d7_forces2y);
	hipFree(d7_forces3y);
	hipFree(d8_forces1x);
	hipFree(d8_forces2x);
	hipFree(d8_forces1y);
	hipFree(d8_forces2y);
	hipFree(d9_forces1x);
	hipFree(d9_forces2x);
	hipFree(d9_forces3x);
	hipFree(d9_forces1y);
	hipFree(d9_forces2y);
	hipFree(d9_forces3y);
	hipFree(d10_list1);
	hipFree(d10_forces1x);
	hipFree(d10_forces1y);

	cout << "freed" << endl;


	}
}
/*
template <typename Fun>
void Microtubule::runGPUcheck(int runtime, int every, Fun func)
{
//	pausel();
	int ccc;
	int totalN = obj->getN();

	//num is the number of boxes per length

	int ncells = num*num;

	WCApotentialGPU faa_gpu(2.,1.,2.);
	WCApotentialGPU fab_gpu(1.,1.,0.);
	WCApotentialGPU fac_gpu(1.,1.,0.);
	WCApotentialGPU fbb_gpu(2.,1.,2.);
	WCApotentialGPU fbc_gpu(1.,1.,0.);
	WCApotentialGPU fcc_gpu(1.,1.,0.);
	HarmonicPotentialGPU bindp_gpu(100.,0.); 
	FENEPotentialGPU bindm_gpu(50.,1.5); 
	BendingPotentialGPU bendp_gpu(100.,0.);

//we now have the count of each cell list

	matrix<int> boxes = (obj)->getgeo().generate_boxes_relationships(num,ccc);


	matrix<int> *froyo1 = obj->calculatepairs(boxes,pai,3.5);
	matrix<int> *froyo2 = obj->calculatepairs(boxes,pbi,3.5);
	matrix<int> *froyo3 = obj->calculatepairs(boxes,pci,3.5);
	matrix<int> *froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
	matrix<int> *froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
	matrix<int> *froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);


	int nbpairs = 5*ncells;
	int nperl = num;

	int *cells1 = new int [nbpairs];
	int *cells2 = new int [nbpairs];


	int itery = 0;
	for(int i1 = 0 ; i1 < num ; i1++) {
		for(int i2 = 0 ; i2 < num ; i2++ ) {


			int b1 =  i1*nperl+i2;

			int i3 = i1+0;
			int j3 = i2+0;

			int i4 = i1+1;
			int j4 = i2+0;

			int i5 = i1-1;
			int j5 = i2+1;

			int i6 = i1+0;
			int j6 = i2+1;

			int i7 = i1+1;
			int j7 = i2+1;

			prdshft(i3,nperl);
			prdshft(j3,nperl);

			prdshft(i4,nperl);
			prdshft(j4,nperl);

			prdshft(i5,nperl);
			prdshft(j5,nperl);
			
			prdshft(i6,nperl);
			prdshft(j6,nperl);
			
			prdshft(i7,nperl);
			prdshft(j7,nperl);		

			cells1[itery] =  b1;
			cells2[itery] =  i3*nperl+j3;

			itery++;

			cells1[itery] =  b1;
			cells2[itery] =  i4*nperl+j4;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i5*nperl+j5;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i6*nperl+j6;
			
			itery++;
			
			cells1[itery] =  b1;
			cells2[itery] =  i7*nperl+j7;

			itery++;


		}
	}
	int size4 = nbpairs*sizeof(int);

	int *d_cells1;
	int *d_cells2;

	hipMalloc((void**)&d_cells1,size4);

	hipMalloc((void**)&d_cells2,size4);

	hipMemcpy(d_cells1,cells1,size4,hipMemcpyHostToDevice);
	hipMemcpy(d_cells2,cells2,size4,hipMemcpyHostToDevice);
	//int sibdiv = floor(ll/4.0);
	// print_device_array(d_cells1,nbpairs);
	// print_device_array(d_cells2,nbpairs);



	// matrix<int> boxes = (obj)->getgeo().generate_boxes_relationships(num,ccc);
	
	float2 *particles = new float2 [totalN];
	float2 *momenta = new float2 [totalN];
	int *p_indices = new int [totalN];

	for(int i = 0 ; i < totalN ; i++)
	p_indices[i]=i;

	float2 *d_particles;
	float2 *d_momenta;
	int *d_p_indices;

	int *d_bound;
	double *d_boundalong;
	int *d_changestate;

	hipMalloc((void**)&d_bound,(na+nb)*sizeof(int));
	hipMalloc((void**)&d_boundalong,(na+nb)*sizeof(double));
	hipMalloc((void**)&d_changestate,(na+nb)*sizeof(int));

	hipMemset(d_bound,0,(na+nb)*sizeof(int));
	hipMemset(d_boundalong,0.,(na+nb)*sizeof(double));
	hipMemset(d_changestate,0,(na+nb)*sizeof(int));

	double *d_totalforcex;
	double *d_totalforcey;

	hipMalloc((void**)&d_totalforcex,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey,totalN*sizeof(double));


	matrix<double> state(obj->getdat());


	for(int i = 0  ; i < totalN ; i++) {

	float2 c;
	c.x=state(i,0);
	c.y=state(i,1);

	(particles)[i]=c;

	float2 d;

	d.x = 0.;
	d.y = 0.;

	(momenta)[i]=d;
	}


	int size =  totalN*sizeof(float2);
	int size2 = totalN*sizeof(int);


	hipMalloc((void**)&d_particles,size);
	hipMalloc((void**)&d_momenta,size);
	hipMalloc((void**)&d_p_indices,size2);

	hipMemcpy(d_particles,particles,size,hipMemcpyHostToDevice);
	hipMemcpy(d_momenta,momenta,size,hipMemcpyHostToDevice);
	hipMemcpy(d_p_indices,p_indices,size2,hipMemcpyHostToDevice);


	// matrix<int> *froyo1 = obj->calculatepairs(boxes,pai,3.5);
	// matrix<int> *froyo2 = obj->calculatepairs(boxes,pbi,3.5);
	// matrix<int> *froyo3 = obj->calculatepairs(boxes,pci,3.5);
	// matrix<int> *froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
	// matrix<int> *froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
	// matrix<int> *froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);
	
	int *d_indices1;
	int *d_indices2;
	double *d_close;


	int tpp;
	


	construct_possible_pair_list(d_particles,d_p_indices,totalN,l,d_cells1,d_cells2,num,is_periodic,d_indices1,d_indices2,d_close,tpp);



	less_than_condition_AND cond1(SQR(3.5),0,na);
	less_than_condition_AND cond2(SQR(3.5),na,na+nb);
	less_than_condition_AND cond3(SQR(3.5),na+nb,na+nb+nc);
	less_than_condition_NAND cond4(SQR(3.5),0,na,na,na+nb);
	less_than_condition_NAND cond5(SQR(3.5),0,na,na+nb,na+nb+nc);
	less_than_condition_NAND cond6(SQR(3.5),na,na+nb,na+nb,na+nb+nc);


	int th1;
	int *d1_list1,*d1_list2,*d1_list3,*d1_list4;
	pairlist(d_indices1,d_indices2,d_close,cond1,d1_list1,d1_list2,d1_list3,d1_list4,tpp, th1); //faa

	int th2;
	int *d2_list1,*d2_list2,*d2_list3,*d2_list4;
	pairlist(d_indices1,d_indices2,d_close,cond2,d2_list1,d2_list2,d2_list3,d2_list4,tpp, th2);	//fbb

	int th3;
	int *d3_list1,*d3_list2,*d3_list3,*d3_list4;
	pairlist(d_indices1,d_indices2,d_close,cond3,d3_list1,d3_list2,d3_list3,d3_list4,tpp, th3);	//fcc

	int th4;
	int *d4_list1,*d4_list2,*d4_list3,*d4_list4;
	pairlist(d_indices1,d_indices2,d_close,cond4,d4_list1,d4_list2,d4_list3,d4_list4,tpp, th4);	//fab

	int th5;
	int *d5_list1,*d5_list2,*d5_list3,*d5_list4;
	pairlist(d_indices1,d_indices2,d_close,cond5,d5_list1,d5_list2,d5_list3,d5_list4,tpp, th5);	//fac

	int th6;
	int *d6_list1,*d6_list2,*d6_list3,*d6_list4;
	pairlist(d_indices1,d_indices2,d_close,cond6,d6_list1,d6_list2,d6_list3,d6_list4,tpp, th6); //fbc			
	//matrix<double> state(obj->getdat()); //the state of the system

	int th8 = (*bondpairs).getNsafe();
	int th9 = (*bendtriplets).getNsafe();


	int *d8_list1,*d8_list2,*d8_list3,*d8_list4;
	hipMalloc((void**)&d8_list1,th8*sizeof(int));
	hipMalloc((void**)&d8_list2,th8*sizeof(int));
	hipMalloc((void**)&d8_list3,th8*sizeof(int));
	hipMalloc((void**)&d8_list4,th8*sizeof(int));



	int *h8_list1 = new int [th8];
	int *h8_list2 = new int [th8];

	for(int i = 0 ; i < th8 ; i++ ) {
		h8_list1[i] = (*bondpairs)(i,0);
		h8_list2[i] = (*bondpairs)(i,1);
	}

	hipMemcpy(d8_list1,h8_list1,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list2,h8_list2,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list3,h8_list1,th8*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d8_list4,h8_list2,th8*sizeof(int),hipMemcpyHostToDevice);





	int *d9_list1,*d9_list2,*d9_list3,*d9_list4,*d9_list5,*d9_list6;
	hipMalloc((void**)&d9_list1,th9*sizeof(int));
	hipMalloc((void**)&d9_list2,th9*sizeof(int));
	hipMalloc((void**)&d9_list3,th9*sizeof(int));
	hipMalloc((void**)&d9_list4,th9*sizeof(int));
	hipMalloc((void**)&d9_list5,th9*sizeof(int));
	hipMalloc((void**)&d9_list6,th9*sizeof(int));

	int *h9_list1 = new int [th9];
	int *h9_list2 = new int [th9];
	int *h9_list3 = new int [th9];

	for(int i = 0 ; i < th9 ; i++ ) {
		h9_list1[i] = (*bendtriplets)(i,0);
		h9_list2[i] = (*bendtriplets)(i,1);
		h9_list3[i] = (*bendtriplets)(i,2);
	}	

	hipMemcpy(d9_list1,h9_list1,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list2,h9_list2,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list3,h9_list3,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list4,h9_list1,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list5,h9_list2,th9*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d9_list6,h9_list3,th9*sizeof(int),hipMemcpyHostToDevice);


	int i;

	double cons1;
	double cons2;
	double cons3;
	double cons4;

	//(mom)->operator()(i,i1) = c5*c2*((mom)->operator()(i,i1)) + (c5*(c3)+q)*F(i,i1) + (c5*(c4)+r)*R(i,i1);
	cons1 = (*obj).getc5()*(*obj).getc2();
	cons2 = (*obj).getc5()*(*obj).getc3()+(*obj).getq();
	cons3 = (*obj).getc5()*(*obj).getc4()+(*obj).getr();
	cons4 = (*obj).getc1();


	// vector<matrix<double> > savef1forces;
	// vector<matrix<double> > savef2forces;
	// vector<matrix<double> > savef3forces;
	// vector<matrix<double> > savef4forces;
	// vector<matrix<double> > savef5forces;
	// vector<matrix<double> > savef6forces;
	// vector<matrix<double> > savef7forces;

	// vector<matrix<double> > saveftemp1forces;
	// vector<matrix<double> > saveftemp2forces;
	// vector<matrix<double> > saveftemp3forces;

	// vector<matrix<double> > savepositions;
	// vector<vector1<int> > savebound;
	// vector<vector1<double> > saveboundalongs;

	for(i = 0 ; i < runtime ; i++) {
		//cout << i << endl;

		cout << i << endl;
		//pausel();
	
		//cout << (*obj).avmom() << endl;
	if(i%25==0) {
		//delete froyo1,froyo2,froyo3,froyo4,froyo5,froyo6;
		// cout << "updated after: " << i << endl;
		// state = obj->getdat();

		delete froyo1,froyo2,froyo3,froyo4,froyo5,froyo6;
		// cout << "updated after: " << i << endl;
		// state = obj->getdat();
		froyo1 = obj->calculatepairs(boxes,pai,3.5);
		froyo2 = obj->calculatepairs(boxes,pbi,3.5);
		froyo3 = obj->calculatepairs(boxes,pci,3.5);
		froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
		froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
		froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);


		hipFree(d1_list1);hipFree(d1_list2);hipFree(d1_list3);hipFree(d1_list4);
		

		hipFree(d2_list1);hipFree(d2_list2);hipFree(d2_list3);hipFree(d2_list4);
		

		hipFree(d3_list1);hipFree(d3_list2);hipFree(d3_list3);hipFree(d3_list4);
		

		hipFree(d4_list1);hipFree(d4_list2);hipFree(d4_list3);hipFree(d4_list4);
		

		hipFree(d5_list1);hipFree(d5_list2);hipFree(d5_list3);hipFree(d5_list4);
	
		hipFree(d6_list1);hipFree(d6_list2);hipFree(d6_list3);hipFree(d6_list4);

		hipFree(d_indices1);

		hipFree(d_indices2);

		hipFree(d_close);



		this->resetindices(d_p_indices,totalN);



		construct_possible_pair_list(d_particles,d_p_indices,totalN,l,d_cells1,d_cells2,num,is_periodic,d_indices1,d_indices2,d_close,tpp,false);


		cout << "pair" << endl;

		pairlist(d_indices1,d_indices2,d_close,cond1,d1_list1,d1_list2,d1_list3,d1_list4,tpp, th1); //faa
		pairlist(d_indices1,d_indices2,d_close,cond2,d2_list1,d2_list2,d2_list3,d2_list4,tpp, th2); //fbb
		pairlist(d_indices1,d_indices2,d_close,cond3,d3_list1,d3_list2,d3_list3,d3_list4,tpp, th3); //fcc
		pairlist(d_indices1,d_indices2,d_close,cond4,d4_list1,d4_list2,d4_list3,d4_list4,tpp, th4); //fab
		pairlist(d_indices1,d_indices2,d_close,cond5,d5_list1,d5_list2,d5_list3,d5_list4,tpp, th5); //fac
		pairlist(d_indices1,d_indices2,d_close,cond6,d6_list1,d6_list2,d6_list3,d6_list4,tpp, th6); //fbc


		
		// froyo1 = obj->calculatepairs(boxes,pai,3.5);
		// froyo2 = obj->calculatepairs(boxes,pbi,3.5);
		// froyo3 = obj->calculatepairs(boxes,pci,3.5);
		// froyo4 = obj->calculatepairs(boxes,pai,pbi,3.5);
		// froyo5 = obj->calculatepairs(boxes,pai,pci,3.5);
		// froyo6 = obj->calculatepairs(boxes,pbi,pci,3.5);

	}





	//cout << "pairs" << endl;
	// cout << "pairings" << endl;


	// matrix<double> ftemp2(totalN,dimension),ftemp3(totalN,dimension);
	// //matrix<double> angforces1(nc,dimension-1),angforces2(nc,dimension-1),angforces3(nc,dimension-1);
	// // cout << "matrices initialized" << endl;

	// matrix<double> F1((*obj).calculateforces(*froyo1,*faa)); //calculate the forces using the pairs as an input

	// matrix<double> F2((*obj).calculateforces(*froyo2,*fbb)); //calculate the forces using the pairs as an input

	// matrix<double> ftemp1((*obj).calculateforces(*froyo3,*fcc)); //calculate the forces using the pairs as an input
	
	// matrix<double> F3((*obj).calculateforces(*froyo4,*fab)); //calculate the forces using the pairs as an input

	// this->ForcesDueToPositionPL(*froyo5,ftemp2); //calculate the forces using the pairs as an input

	// this->ForcesDueToPositionPL(*froyo6,ftemp3); //calculate the forces using the pairs as an input

	// this->CalculateBindings(*froyo5,*froyo6);

	// matrix<double> F4 = this->BindingForces();

	// matrix<double> F5 = this->PositionForcesDueToAngles();

	//print_device_float2(d_particles,totalN);

	double *d1_forces1x;
	double *d1_forces2x;
	double *d1_forces1y;
	double *d1_forces2y;
	hipMalloc((void**)&d1_forces1x,th1*sizeof(double));
	hipMalloc((void**)&d1_forces1y,th1*sizeof(double));
	hipMalloc((void**)&d1_forces2x,th1*sizeof(double));
	hipMalloc((void**)&d1_forces2y,th1*sizeof(double));
	calculateforces2D(d1_list1,d1_list2,d_particles, d1_forces1x,d1_forces1y,d1_forces2x,d1_forces2y, faa_gpu ,th1, l,true);

	cout << "force1" << endl;
	arracychck(d1_forces1x,th1);
	arracychck(d1_forces1y,th1); 

	double *d2_forces1x;
	double *d2_forces2x;
	double *d2_forces1y;
	double *d2_forces2y;
	hipMalloc((void**)&d2_forces1x,th2*sizeof(double));
	hipMalloc((void**)&d2_forces1y,th2*sizeof(double));
	hipMalloc((void**)&d2_forces2x,th2*sizeof(double));
	hipMalloc((void**)&d2_forces2y,th2*sizeof(double));
	calculateforces2D(d2_list1,d2_list2,d_particles, d2_forces1x,d2_forces1y,d2_forces2x,d2_forces2y, fbb_gpu ,th2, l,true);	

	cout << "force2" << endl;
	arracychck(d1_forces2x,th2);
	arracychck(d1_forces2y,th2); 

	double *d3_forces1x;
	double *d3_forces2x;
	double *d3_forces1y;
	double *d3_forces2y;
	hipMalloc((void**)&d3_forces1x,th3*sizeof(double));
	hipMalloc((void**)&d3_forces1y,th3*sizeof(double));
	hipMalloc((void**)&d3_forces2x,th3*sizeof(double));
	hipMalloc((void**)&d3_forces2y,th3*sizeof(double));
	calculateforces2D(d3_list1,d3_list2,d_particles, d3_forces1x,d3_forces1y,d3_forces2x,d3_forces2y, fcc_gpu ,th3, l,true);	


	cout << "force3" << endl;
	arracychck(d3_forces1x,th3);
	arracychck(d3_forces1y,th3); 

	double *d4_forces1x;
	double *d4_forces2x;
	double *d4_forces1y;
	double *d4_forces2y;
	hipMalloc((void**)&d4_forces1x,th4*sizeof(double));
	hipMalloc((void**)&d4_forces1y,th4*sizeof(double));
	hipMalloc((void**)&d4_forces2x,th4*sizeof(double));
	hipMalloc((void**)&d4_forces2y,th4*sizeof(double));
	calculateforces2D(d4_list1,d4_list2,d_particles, d4_forces1x,d4_forces1y,d4_forces2x,d4_forces2y, fab_gpu ,th4, l,true);

	cout << "force4" << endl;
	arracychck(d4_forces1x,th4);
	arracychck(d4_forces1y,th4); 

	double *d5_forces1x;
	double *d5_forces2x;
	double *d5_forces1y;
	double *d5_forces2y;
	hipMalloc((void**)&d5_forces1x,th5*sizeof(double));
	hipMalloc((void**)&d5_forces1y,th5*sizeof(double));
	hipMalloc((void**)&d5_forces2x,th5*sizeof(double));
	hipMalloc((void**)&d5_forces2y,th5*sizeof(double));
	calculateforces2D(d5_list1,d5_list2,d_particles, d5_forces1x,d5_forces1y,d5_forces2x,d5_forces2y, fbc_gpu ,th5, l,true);

	cout << "force5" << endl;
	arracychck(d5_forces1x,th5);
	arracychck(d5_forces1y,th5); 	
	// cout << "d5" << endl;
	// print_device_weave_float2(d5_list1,d5_list2,d5_forces1x,d5_forces1y,d5_forces2x,d5_forces2y,d_particles,th5,totalN);
	// pausel();	

	double *d6_forces1x;
	double *d6_forces2x;
	double *d6_forces1y;
	double *d6_forces2y;
	hipMalloc((void**)&d6_forces1x,th6*sizeof(double));
	hipMalloc((void**)&d6_forces1y,th6*sizeof(double));
	hipMalloc((void**)&d6_forces2x,th6*sizeof(double));
	hipMalloc((void**)&d6_forces2y,th6*sizeof(double));
	calculateforces2D(d6_list1,d6_list2,d_particles, d6_forces1x,d6_forces1y,d6_forces2x,d6_forces2y, fac_gpu ,th6, l,true);	

	cout << "force6" << endl;
	arracychck(d6_forces1x,th6);
	arracychck(d6_forces1y,th6); 	
	// cout << "d6" << endl;
	// print_device_weave_float2(d6_list1,d6_list2,d6_forces1x,d6_forces1y,d6_forces2x,d6_forces2y,d_particles,th6,totalN);
	// cout << endl;
	// pausel();

	// matrix<double> F6((*obj).calculateforces(*bondpairs,*bindm));
	callCalculateUnbindingsGPU(d_particles,d_bound,d_boundalong,d_changestate);



	cout << "unbindings calculated" << endl;




	callCalculateBindingsGPU(d5_list1,d5_list2,d6_list1,d6_list2,d_particles, d_bound, d_boundalong,d_changestate,th5 ,th6 );


	cout << "bindings calculated" << endl;



	int *d7_list1,*d7_list2,*d7_list3;
	double *d7_forces1x;
	double *d7_forces1y;
	double *d7_forces2x;
	double *d7_forces2y;
	double *d7_forces3x;
	double *d7_forces3y;
	int th7;
	BindingForcesGPU(d_particles, d_bound, d_boundalong, d7_list1,d7_list2,d7_list3, d7_forces1x, d7_forces1y, d7_forces2x,d7_forces2y,d7_forces3x, d7_forces3y, bindp_gpu, th7);


	cout << "binding forces calculated" << endl;
	arracychck(d7_forces1x,th7);
	arracychck(d7_forces1y,th7); 

	double *d8_forces1x;
	double *d8_forces2x;
	double *d8_forces1y;
	double *d8_forces2y;
	hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	hipMalloc((void**)&d8_forces2y,th8*sizeof(double));
	calculateforces2D(d8_list1,d8_list2,d_particles, d8_forces1x,d8_forces1y,d8_forces2x,d8_forces2y, bindm_gpu ,th8, l,true);

	cout << "forces8" << endl;
	arracychck(d8_forces1x,th8);
	arracychck(d8_forces1y,th8); 

	resetchangestate(d_changestate);
	// matrix<double> F7((*obj).calculateforces_threebody(*bendtriplets,*bendp));

//	int th9;


	double *d9_forces1x;
	double *d9_forces2x;
	double *d9_forces1y;
	double *d9_forces2y;
	double *d9_forces3x;
	double *d9_forces3y;
	hipMalloc((void**)&d9_forces1x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces1y,th9*sizeof(double));
	hipMalloc((void**)&d9_forces2x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces2y,th9*sizeof(double));
	hipMalloc((void**)&d9_forces3x,th9*sizeof(double));
	hipMalloc((void**)&d9_forces3y,th9*sizeof(double));
	BendingForcesGPU(d_particles, d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces1y,d9_forces2x,d9_forces2y, d9_forces3x, d9_forces3y,bendp_gpu,th9);

	cout << "bending forces" << endl;
	arracychck(d9_forces1x,th9);
	arracychck(d9_forces1y,th9); 
	//	matrix<double> F = ftemp1+ftemp2+ftemp3+F1+F2+F3+F4+F5+F6+F7;//+F4+F5;

		// matrix<double> R(totalN,dimension);
		// for(int i1 = 0 ; i1 < totalN ; i1++) {
		// 	for(int j = 0 ; j < dimension ; j++) {
		// 		R(i1,j) = (3.464101615 * ((double) rand() / (RAND_MAX)) - 1.732050808);
		// 	}
		// }

	int *d10_list1;
	double *d10_forces1x;
	double *d10_forces1y;	
	int th10;

	PositionForcesDueToAnglesGPU(d_particles, d_bound, d_boundalong, d10_list1, d10_forces1x, d10_forces1y,th10);
	

	cout << "pos forces" << endl;
	arracychck(d10_forces1x,th9);
	arracychck(d10_forces1y,th9); 

	cout << "all forces calculated" << endl;




	double *d_totalforcex1;
	double *d_totalforcey1;

	hipMalloc((void**)&d_totalforcex1,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey1,totalN*sizeof(double));

	resetforce(d_totalforcex1);
	resetforce(d_totalforcey1);

	cout << "reset" << endl;

	// print_device_array(d_totalforcex,totalN);


	//ReduceForces(d1_list1,d1_list2,d1_list3,d1_list4,d1_forces1x,d1_forces2x,d1_forces1y,d1_forces2y,d_totalforcex,d_totalforcey,th1);
	ReduceForces(d1_list1,d1_list2,d1_list3,d1_list4,d1_forces1x,d1_forces2x,d1_forces1y,d1_forces2y,d_totalforcex1,d_totalforcey1,th1);	
	 cout << "d1" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();	

	double *d_totalforcex2;
	double *d_totalforcey2;

	hipMalloc((void**)&d_totalforcex2,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey2,totalN*sizeof(double));

	resetforce(d_totalforcex2);
	resetforce(d_totalforcey2);	 
	//ReduceForces(d2_list1,d2_list2,d2_list3,d2_list4,d2_forces1x,d2_forces2x,d2_forces1y,d2_forces2y,d_totalforcex,d_totalforcey,th2);
	ReduceForces(d2_list1,d2_list2,d2_list3,d2_list4,d2_forces1x,d2_forces2x,d2_forces1y,d2_forces2y,d_totalforcex2,d_totalforcey2,th2);
	cout << "d2" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double *d_totalforcex3;
	double *d_totalforcey3;

	hipMalloc((void**)&d_totalforcex3,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey3,totalN*sizeof(double));

	resetforce(d_totalforcex3);
	resetforce(d_totalforcey3);	 	
	//ReduceForces(d3_list1,d3_list2,d3_list3,d3_list4,d3_forces1x,d3_forces2x,d3_forces1y,d3_forces2y,d_totalforcex,d_totalforcey,th3);	 	
	ReduceForces(d3_list1,d3_list2,d3_list3,d3_list4,d3_forces1x,d3_forces2x,d3_forces1y,d3_forces2y,d_totalforcex3,d_totalforcey3,th3);
	 cout << "d3" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double *d_totalforcex4;
	double *d_totalforcey4;

	hipMalloc((void**)&d_totalforcex4,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey4,totalN*sizeof(double));

	resetforce(d_totalforcex4);
	resetforce(d_totalforcey4);	 		
	//ReduceForces(d4_list1,d4_list2,d4_list3,d4_list4,d4_forces1x,d4_forces2x,d4_forces1y,d4_forces2y,d_totalforcex,d_totalforcey,th4);
	ReduceForces(d4_list1,d4_list2,d4_list3,d4_list4,d4_forces1x,d4_forces2x,d4_forces1y,d4_forces2y,d_totalforcex4,d_totalforcey4,th4);
	 cout << "d4" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double *d_totalforcex5;
	double *d_totalforcey5;

	hipMalloc((void**)&d_totalforcex5,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey5,totalN*sizeof(double));

	resetforce(d_totalforcex5);
	resetforce(d_totalforcey5);	 	 
	//ReduceForces(d5_list1,d5_list2,d5_list3,d5_list4,d5_forces1x,d5_forces2x,d5_forces1y,d5_forces2y,d_totalforcex,d_totalforcey,th5);
	ReduceForces(d5_list1,d5_list2,d5_list3,d5_list4,d5_forces1x,d5_forces2x,d5_forces1y,d5_forces2y,d_totalforcex5,d_totalforcey5,th5);	

	 cout << "d5" << endl;	
	// print_device_weave_float2(d5_list1,d5_list2,d5_forces1x,d5_forces1y,d_particles,th5,totalN);
	// pausel();
	double *d_totalforcex6;
	double *d_totalforcey6;

	hipMalloc((void**)&d_totalforcex6,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey6,totalN*sizeof(double));

	resetforce(d_totalforcex6);
	resetforce(d_totalforcey6);	 	
	//ReduceForces(d6_list1,d6_list2,d6_list3,d6_list4,d6_forces1x,d6_forces2x,d6_forces1y,d6_forces2y,d_totalforcex,d_totalforcey,th6);	 	
	ReduceForces(d6_list1,d6_list2,d6_list3,d6_list4,d6_forces1x,d6_forces2x,d6_forces1y,d6_forces2y,d_totalforcex6,d_totalforcey6,th6);		
	 cout << "d6" << endl;
	// print_device_weave_float2(d6_list1,d6_list2,d6_forces1x,d6_forces1y,d_particles,th6,totalN);
	// cout << endl;
	// pausel();
	double *d_totalforcex7;
	double *d_totalforcey7;

	hipMalloc((void**)&d_totalforcex7,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey7,totalN*sizeof(double));

	resetforce(d_totalforcex7);
	resetforce(d_totalforcey7);	 
	//ReduceForces3(d7_list1,d7_list2,d7_list3,d7_forces1x,d7_forces2x,d7_forces3x,d7_forces1y,d7_forces2y,d7_forces3y,d_totalforcex,d_totalforcey,th7);	 
	ReduceForces3(d7_list1,d7_list2,d7_list3,d7_forces1x,d7_forces2x,d7_forces3x,d7_forces1y,d7_forces2y,d7_forces3y,d_totalforcex7,d_totalforcey7,th7);	
	 cout << "d7" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double *d_totalforcex8;
	double *d_totalforcey8;

	hipMalloc((void**)&d_totalforcex8,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey8,totalN*sizeof(double));
	

	
	resetforce(d_totalforcex8);
	resetforce(d_totalforcey8);	 
	//ReduceForces(d8_list1,d8_list2,d8_forces1x,d8_forces2x,d8_forces1y,d8_forces2y,d_totalforcex,d_totalforcey,th8);	 
	ReduceForces(d8_list1,d8_list2,d8_forces1x,d8_forces2x,d8_forces1y,d8_forces2y,d_totalforcex8,d_totalforcey8,th8);
	 cout << "d8" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double *d_totalforcex9;
	double *d_totalforcey9;

	hipMalloc((void**)&d_totalforcex9,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey9,totalN*sizeof(double));

	resetforce(d_totalforcex9);
	resetforce(d_totalforcey9);	 
	//ReduceForces3(d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces2x,d9_forces3x,d9_forces1y,d9_forces2y,d9_forces3y,d_totalforcex,d_totalforcey,th9);		 
	ReduceForces3(d9_list1,d9_list2,d9_list3,d9_forces1x,d9_forces2x,d9_forces3x,d9_forces1y,d9_forces2y,d9_forces3y,d_totalforcex9,d_totalforcey9,th9);		
	
	double *d_totalforcex10;
	double *d_totalforcey10;

	hipMalloc((void**)&d_totalforcex10,totalN*sizeof(double));
	hipMalloc((void**)&d_totalforcey10,totalN*sizeof(double));

	resetforce(d_totalforcex10);
	resetforce(d_totalforcey10);	 
	 cout << "d9" << endl;	
	// print_device_array(d_totalforcex,totalN);
	// pausel();
	double ff = (v0_a+v0_b)/2.;
	//ReduceForcesAndNormalize(d10_list1,d10_forces1x,d10_forces1y,d_totalforcex,d_totalforcey, max_s, ff, th10);
	ReduceForcesAndNormalize(d10_list1,d10_forces1x,d10_forces1y,d_totalforcex10,d_totalforcey10, max_s, ff, th10);

	cout << "reduction" << endl;
	// cout << "d10" << endl;
	// print_device_array(d_totalforcex,totalN);
	// pausel();

	matrix<double> ftemp2(totalN,dimension),ftemp3(totalN,dimension);
	//matrix<double> angforces1(nc,dimension-1),angforces2(nc,dimension-1),angforces3(nc,dimension-1);
	// cout << "matrices initialized" << endl;

	matrix<double> F1((*obj).calculateforces(*froyo1,*faa)); //calculate the forces using the pairs as an input

	matrix<double> F2((*obj).calculateforces(*froyo2,*fbb)); //calculate the forces using the pairs as an input

	matrix<double> ftemp1((*obj).calculateforces(*froyo3,*fcc)); //calculate the forces using the pairs as an input
	
	matrix<double> F3((*obj).calculateforces(*froyo4,*fab)); //calculate the forces using the pairs as an input

	this->ForcesDueToPositionPL(*froyo5,ftemp2); //calculate the forces using the pairs as an input

	this->ForcesDueToPositionPL(*froyo6,ftemp3); //calculate the forces using the pairs as an input

	this->CalculateBindings(*froyo5,*froyo6);

	matrix<double> F4 = this->BindingForces();

	matrix<double> F5 = this->PositionForcesDueToAngles();


//	cout << "active forces" << endl;
	//cout << "pos forces" << endl;
	 matrix<double> F6((*obj).calculateforces(*bondpairs,*bindm));

	//cout << "bond forces" << endl;
	matrix<double> F7((*obj).calculateforces_threebody(*bendtriplets,*bendp));

	//cout << "after check matrix" << endl;

	matrix<double> F = ftemp1+ftemp2+ftemp3+F1+F2+F3+F4+F5+F6+F7;

	cout << l << endl;
	for(int j1 = 0 ; j1 < na+nb ; j1++ ) {
		if(bound[j1]>0) cout << j1 << ",";
	}
	cout << endl;
	print_device_array_indices(d_bound,na+nb);
	pausel();




	cout << F1 << endl;
	print_device_array_weave(d_totalforcex1,d_totalforcey1,totalN);
	cout << th1 << endl;
	cout << "faa" << endl;
	pausel();



	cout << F2 << endl;
	print_device_array_weave(d_totalforcex2,d_totalforcey2,totalN);
	cout << "fbb" << endl;
	pausel();


	cout << ftemp1 << endl;
	print_device_array_weave(d_totalforcex3,d_totalforcey3,totalN);
	cout << "fcc" << endl;
	pausel();		

	cout << F3 << endl;
	print_device_array_weave(d_totalforcex4,d_totalforcey4,totalN);
	cout << "fab" << endl;
	pausel();

	cout << ftemp2 << endl;
	print_device_array_weave(d_totalforcex5,d_totalforcey5,totalN);
	cout << "fac" << endl;
	pausel();		

	cout << ftemp3 << endl;
	print_device_array_weave(d_totalforcex6,d_totalforcey6,totalN);
	cout << "fbc" << endl;
	pausel();

	cout << F4 << endl;
	print_device_array_weave(d_totalforcex7,d_totalforcey7,totalN);
	cout << "bound to mt" << endl;
	pausel();	

	cout << F5 << endl;
	print_device_array_weave(d_totalforcex10,d_totalforcey10,totalN);
	cout << "position force" << endl;
	pausel();

	cout << F6 << endl;
	print_device_array_weave(d_totalforcex8,d_totalforcey8,totalN);
	cout << "bound within mt" << endl;
	pausel();		


	cout << F7 << endl;
	print_device_array_weave(d_totalforcex9,d_totalforcey9,totalN);
	cout << "bending force" << endl;
	pausel();		







	double *d_R1;
	double *d_R2;

	// hipMalloc((void**)&d8_forces1x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces1y,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2x,th8*sizeof(double));
	// hipMalloc((void**)&d8_forces2y,th8*sizeof(double));	
	hipMalloc((void**)&d_R1,totalN*sizeof(double));
	hipMalloc((void**)&d_R2,totalN*sizeof(double));

	setstaterandom(d_R1,1.732050808,totalN);
 	setstaterandom(d_R2,1.732050808,totalN);

 	print_device_array(d_R1,totalN);
 	print_device_array(d_R2,totalN);

 	pausel();

 	advmom2D(d_momenta, d_totalforcex, d_totalforcey, d_R1, d_R2, cons1,cons2,cons3,totalN);
 	advpos2D(d_particles, d_momenta, cons4, totalN);

 	applypbc2D(d_particles,d_momenta,l,is_periodic,totalN);

 	cout << "updated" << endl;

	hipFree(d1_forces1x);
	hipFree(d1_forces2x);
	hipFree(d1_forces1y);
	hipFree(d1_forces2y);
	hipFree(d2_forces1x);
	hipFree(d2_forces2x);
	hipFree(d2_forces1y);
	hipFree(d2_forces2y);
	hipFree(d3_forces1x);
	hipFree(d3_forces2x);
	hipFree(d3_forces1y);
	hipFree(d3_forces2y);
	hipFree(d4_forces1x);
	hipFree(d4_forces2x);
	hipFree(d4_forces1y);
	hipFree(d4_forces2y);
	hipFree(d5_forces1x);
	hipFree(d5_forces2x);
	hipFree(d5_forces1y);
	hipFree(d5_forces2y);
	hipFree(d6_forces1x);
	hipFree(d6_forces2x);
	hipFree(d6_forces1y);
	hipFree(d6_forces2y);
 	hipFree(d7_list1);
 	hipFree(d7_list2);
 	hipFree(d7_list3);
	hipFree(d7_forces1x);
	hipFree(d7_forces2x);
	hipFree(d7_forces3x);
	hipFree(d7_forces1y);
	hipFree(d7_forces2y);
	hipFree(d7_forces3y);
	hipFree(d8_forces1x);
	hipFree(d8_forces2x);
	hipFree(d8_forces1y);
	hipFree(d8_forces2y);
	hipFree(d9_forces1x);
	hipFree(d9_forces2x);
	hipFree(d9_forces3x);
	hipFree(d9_forces1y);
	hipFree(d9_forces2y);
	hipFree(d9_forces3y);
	hipFree(d10_list1);
	hipFree(d10_forces1x);
	hipFree(d10_forces1y);

	cout << "freed" << endl;


	}
}
*/